#include "timer.h"            

void Timer::StartTimer() {

   if (UseHostTimer) {
      #ifdef USING_CPP
      start_c = std::chrono::high_resolution_clock::now();
      #else
      gettimeofday(&start_t, NULL);
      #endif
   } else {
      checkCudaErrors(hipEventRecord(start_e, stream));
   }
}

void Timer::StopTimer() {
   if (UseHostTimer) {
      #ifdef USING_CPP
      stop_c = std::chrono::high_resolution_clock::now(); 
      #else
      gettimeofday(&stop_t, NULL);
      #endif 
   } else {
      checkCudaErrors(hipEventRecord(stop_e, stream));   
   }
}

// Returns elasped time in microseconds
float Timer::ElapsedTime() {
   float time = 0.0;

   if (UseHostTimer) {
      #ifdef USING_CPP
      auto total_c = std::chrono::duration_cast<std::chrono::microseconds>(stop_c - start_c);
      time = (float) total_c.count(); 
      #else
      timersub(&stop_t, &start_t, &total_t); 
      time = (float) total_t.tv_usec + (float) total_t.tv_sec * 1.0e6;
      #endif 
   } else {      
      checkCudaErrors(hipEventSynchronize(stop_e)); 
      checkCudaErrors(hipEventElapsedTime(&time, start_e, stop_e)); 
      time *= 1.0e3;  
   }

   return time;
}

//TODO check if cuda events are reusable; if so delete member function
void Timer::ResetTimer() {
   if (!UseHostTimer) {
      checkCudaErrors(hipStreamDestroy(stream));
      checkCudaErrors(hipEventDestroy(start_e));
      checkCudaErrors(hipEventDestroy(stop_e));

      checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));             
      checkCudaErrors(hipEventCreate(&start_e));
      checkCudaErrors(hipEventCreate(&stop_e));        
   } 
}

Timer::~Timer() {
   if (!UseHostTimer) {
      checkCudaErrors(hipStreamDestroy(stream));
      checkCudaErrors(hipEventDestroy(start_e));
      checkCudaErrors(hipEventDestroy(stop_e));
   } 
}

Timer::Timer(bool UseHostTimer = true) : UseHostTimer(UseHostTimer) {
   if (!UseHostTimer) {
      checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));             
      checkCudaErrors(hipEventCreate(&start_e));
      checkCudaErrors(hipEventCreate(&stop_e)); 
   } 
}





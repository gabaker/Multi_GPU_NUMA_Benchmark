#include "timer.h"            

void Timer::StartTimer() {

   if (UseHostTimer) {
      #ifdef USING_CPP
      start_c = std::chrono::high_resolution_clock::now();
      #else
      gettimeofday(&start_t, NULL);
      #endif
   } else {
      checkCudaErrors(hipEventRecord(start_e, stream)); //stream
   }
}

void Timer::StopTimer() {
   if (UseHostTimer) {
      #ifdef USING_CPP
      stop_c = std::chrono::high_resolution_clock::now(); 
      #else
      gettimeofday(&stop_t, NULL);
      #endif 
   } else {
      checkCudaErrors(hipEventRecord(stop_e, stream));   ///stream
   }
}

// Returns elasped time in microseconds
float Timer::ElapsedTime() {
   float time = 0.0;

   if (UseHostTimer) {
      #ifdef USING_CPP
      auto total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
      time = (float) total_c.count() * 1.0e-3; 
      #else
      timersub(&stop_t, &start_t, &total_t); 
      time = (float) total_t.tv_usec + (float) total_t.tv_sec * 1.0e6;
      #endif 
   } else {      
      //checkCudaErrors(hipEventSynchronize(stop_e)); 
      //checkCudaErrors(hipStreamSynchronize(stream)); 
      checkCudaErrors(hipEventSynchronize(stop_e)); 
      checkCudaErrors(hipEventElapsedTime(&time, start_e, stop_e)); 
      time *= 1.0e3;  
   }

   return time;
}

void Timer::SetHostTiming(bool HostTimer) {

   if (!HostTimer && UseHostTimer) {
      checkCudaErrors(hipStreamCreate(&stream));             
      checkCudaErrors(hipEventCreate(&start_e));
      checkCudaErrors(hipEventCreate(&stop_e)); 
   } else if (HostTimer && !UseHostTimer) {
      checkCudaErrors(hipStreamDestroy(stream));
      checkCudaErrors(hipEventDestroy(start_e));
      checkCudaErrors(hipEventDestroy(stop_e));   
   }

   UseHostTimer = HostTimer;
}      

Timer::~Timer() {
   if (!UseHostTimer) {
      checkCudaErrors(hipStreamDestroy(stream));
      checkCudaErrors(hipEventDestroy(start_e));
      checkCudaErrors(hipEventDestroy(stop_e));
   } 
}

Timer::Timer(bool HostTimer) {
   UseHostTimer = HostTimer;
   if (!UseHostTimer) {
      checkCudaErrors(hipStreamCreate(&stream));             
      //checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));             
      checkCudaErrors(hipEventCreate(&start_e));
      checkCudaErrors(hipEventCreate(&stop_e)); 
   } 
}





// CUDA API and includes
#include<hip/hip_runtime.h>

// C/C++ standard includes
#include<memory>
#include<iostream>
#include<stdio.h>
#include<string>
#include<vector>
#include<time.h>
#include<string>
#include<fstream>
#include<iostream>
#include<ios>
#include<vector>

#include<sys/time.h>
//#include<chrono>

#ifdef USING_CPP
#include<chrono>
#include<tuple>
#endif
// OpenMP threading includes
#include<omp.h>

// NUMA Locality includes
//#include<hwloc.h>
#define MILLI_TO_MICRO (1000.0)
#define NANO_TO_MICRO (1000.0)

typedef struct TestParams {
   std::string resultsFile;
   std::string inputFile;
   bool useDefaultParams;

   bool printDevProps;
   std::string devPropFile;

   int nDevices;

   // Overhead memory test for allocation and deallocation of Host and Device memory
   bool runMemoryOverheadTest;
   bool runAllDevices;
   long rangeMemOverhead[3]; //min, max and step size (in bytes)
 
   // Device-Peer PCIe Baseline bandwidth test
   bool runHostDeviceBandwidthTest;
   bool varyBlockSizeHD;
   bool usePinnedHD;
   bool runBurstHD;
   bool runSustainedHD;
   long rangeHostDeviceBW[3]; //min, max and step size (in bytes)

   // Peer-to-peer device memory transfer bandwidth
   bool runP2PBandwidthTest;
   bool varyBlockSizeP2P;
   bool runBurstP2P;
   bool runSustainedP2P;
   long rangeDeviceP2P[3]; //min, max and step size (in bytes)

   // PCIe Congestion tests
   bool runPCIeCongestionTest;

   // CUDA kernel task scalability and load balancing
   bool runTaskScalabilityTest;

} TestParams;

void RunBandwidthTestSuite(TestParams &params);

void PrintDeviceProps(hipDeviceProp_t *props, TestParams &params);
void TestMemoryOverhead(hipDeviceProp_t *props, TestParams &params);
void TestHostDeviceBandwidth(hipDeviceProp_t *props, TestParams &params);
void TestP2PDeviceBandwidth(hipDeviceProp_t *props, TestParams &params);
void TestPCIeCongestion(hipDeviceProp_t *props, TestParams &params);
void TestTaskScalability(hipDeviceProp_t *props, TestParams &params);
void ParseTestParameters(TestParams &params);

void SetDefaultParams(TestParams &params); 
void GetAllDeviceProps(hipDeviceProp_t *props, int dCount);
void ResetDevices(int numToReset);
void SetDefaultParams(TestParams &params); 
void PrintTestParams(TestParams &params);
void getNextLine(std::ifstream &inFile, std::string &lineStr);
void printResults(std::ofstream &outFile, std::vector<long> &steps, std::vector<std::vector<float> > &results, TestParams &params); 

int main (int argc, char **argv) {
   TestParams params;
 
   
   std::cout << "\nStarting Multi-GPU Performance Test Suite...\n" << std::endl; 

   // Determine the number of recognized CUDA enabled devices
   hipGetDeviceCount(&(params.nDevices));

   if (params.nDevices <= 0) {
      std::cout << "No devices found...aborting benchmarks." << std::endl;
      exit(-1);
   }

   // Setup benchmark parameters
   if (argc == 1) { //No input file, use default parameters
   
      SetDefaultParams(params);
   
   } else if (argc == 2) { //Parse input file
   
      params.inputFile = std::string(argv[1]);
      ParseTestParameters(params);
   
   } else { //Unknown input parameter list, abort test
      std::cout << "Aborting test: Incorrect number of input parameters" << std::endl;
      exit(-1);
   }


   PrintTestParams(params);
   RunBandwidthTestSuite(params);

   return 0;
}

void RunBandwidthTestSuite(TestParams &params) {
   hipDeviceProp_t *props = (hipDeviceProp_t *) calloc (sizeof(hipDeviceProp_t), params.nDevices);

   // Aquire device properties for each CUDA enabled GPU
   GetAllDeviceProps(props, params.nDevices);

   if (params.runMemoryOverheadTest != false ) {
      
      TestMemoryOverhead(props, params);
   
   }

   if (params.runHostDeviceBandwidthTest != false) {

      TestHostDeviceBandwidth(props, params);

   }

   if (params.runP2PBandwidthTest != false) {  
      
      TestP2PDeviceBandwidth(props, params);
   
   }

   if (params.runPCIeCongestionTest != false) {

      TestPCIeCongestion(props, params);

   }

   if (params.runTaskScalabilityTest != false) { 

      TestTaskScalability(props, params);

   }

   // Output device properties for each CUDA enabled GPU
   if (params.printDevProps != false) {
      PrintDeviceProps(props, params);
   }

   std::cout << "\n\nBenchmarks complete!\n" << std::endl;

   free(props);
}

void TestMemoryOverhead(hipDeviceProp_t *props, TestParams &params) {
      // Create CUDA runtime events used to time device operations
      hipEvent_t start_e, stop_e; 
      hipEventCreate(&start_e);
      hipEventCreate(&stop_e);

      char *deviceMem = NULL;
      char *hostUnPinnedMem = NULL;
      char *hostPinnedMem = NULL;
      float eTime = 0.0;
      int nDevices = params.nDevices;

      std::vector<long> blockSteps;
//      std::vector<std::vector<float> > devData;
      std::vector<std::vector<float> > overheadData;

   // Only run overhead device cases on a single device
      // default to device 0
      if (!params.runAllDevices)
         nDevices = 1;

      // Memory overhead test will run for each device utilizing the cudaMalloc and cudaFree functions
      // on the first iteration of the look, assuming there is atleast one device, the host will run the 
      // pinned and un-pinned memory tests
      long stepNum = 0;
      long stepSize = (params.rangeMemOverhead[0] / 10 > 0) ? params.rangeMemOverhead[0] : 1; 
      for ( long chunkSize = params.rangeMemOverhead[0]; 
            chunkSize <= params.rangeMemOverhead[1]; 
            chunkSize += stepSize) {
 
         blockSteps.push_back(chunkSize); 
         std::vector<float> chunkData;
//         std::vector<float> chunkDeviceData;

         for (int currDev = 0; currDev < nDevices; currDev++) {
            hipSetDevice(currDev);

            // CASE 1 & 2: Host memory overhead
            // Host test only runs the first time
            if (currDev == 0) {

               // Pinned
               //allocation
               hipEventRecord(start_e);                             //record start call
               hipHostMalloc((void **) &hostPinnedMem, chunkSize, hipHostMallocDefault);  //malloc pinned memory
               hipEventRecord(stop_e);                              //record finish call
               hipEventSynchronize(stop_e);                         //sync all cuda calls before finish event
               hipEventElapsedTime(&eTime, start_e, stop_e);        //calculate function call time
               chunkData.push_back((float) eTime/* * MILLI_TO_MICRO*/);
               //deallocation
               hipEventRecord(start_e);
               hipHostFree((void *) hostPinnedMem);
               hipEventRecord(stop_e);
               hipEventSynchronize(stop_e);
               hipEventElapsedTime(&eTime, start_e, stop_e);
               chunkData.push_back((float) eTime /* * MILLI_TO_MICRO*/);


               // Unpinned
               #ifdef USING_CPP
               auto start_t = std::chrono::high_resolution_clock::now();
               hostUnPinnedMem = (char *) malloc(chunkSize);
               auto stop_t = std::chrono::high_resolution_clock::now();
               auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_t - start_t);

               chunkData.push_back((float) duration.count() * NANO_TO_MICRO);

               start_t = std::chrono::high_resolution_clock::now();
               free(hostUnPinnedMem);
               stop_t = std::chrono::high_resolution_clock::now(); 
               duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_t - start_t);
               chunkData.push_back((float) duration.count() * NANO_TO_MICRO);
               
               #else
               struct timeval stop_t, start_t, total_t;
               
               //allocation
               gettimeofday(&start_t, NULL);
               hostUnPinnedMem = (char *) malloc(chunkSize); 
               gettimeofday(&stop_t, NULL);
               timersub(&stop_t, &start_t, &total_t);
               chunkData.push_back(((float) total_t.tv_usec));
               //deallocation
               gettimeofday(&start_t, NULL);
               free(hostUnPinnedMem); 
               gettimeofday(&stop_t, NULL); 
               timersub(&stop_t, &start_t, &total_t); 
               chunkData.push_back(((float) total_t.tv_usec));
               #endif

            }

            // CASE 3: Allocation of device memory
            hipEventRecord(start_e);
            hipMalloc((void **) &deviceMem, chunkSize); 
            hipEventRecord(stop_e);

            hipEventSynchronize(stop_e);
            hipEventElapsedTime(&eTime, start_e, stop_e);
            chunkData.push_back((float)eTime  /*MILLI_TO_MICRO*/);

            // CASE 4: DeAllocation of device memory
            hipEventRecord(start_e);
            hipFree(deviceMem); 
            hipEventRecord(stop_e);

            hipEventSynchronize(stop_e);   
            hipEventElapsedTime(&eTime, start_e, stop_e); 
            chunkData.push_back((float) eTime /* MILLI_TO_MICRO*/);

         }
         //devData.push_back(chunkDeviceData);
         overheadData.push_back(chunkData);
         //chunkDeviceData.clear();
         chunkData.clear();
        
         //stepNum++; 
         //do 10 steps then check next range
         if (stepNum && (stepNum % (params.rangeMemOverhead[2] - 1)) == 0)
            stepSize *= 10;
         stepNum++;
      }

      // cleanup CUDA runtime events
      hipEventDestroy(start_e);
      hipEventDestroy(stop_e);

      //Print test Data
/*      std::string fileName = param.resultsFile + "_overhead.csv";
      std::ofstream devResultsFile(fileName.c_str()); 
      printResults(devResultsFile, blockSteps, devData, params);
*/
      std::string dataFileName = params.resultsFile + "_overhead.csv";
      std::ofstream overheadResultsFile(dataFileName.c_str());
      printResults(overheadResultsFile, blockSteps, overheadData, params);
}

void printResults(std::ofstream &outFile, std::vector<long> &steps, std::vector<std::vector<float> > &results, TestParams &params) {
   //std::cout.setf(std::ios::showpoint);
   
   if (!outFile.is_open()) {
      std::cout << "Failed to open file to print results" << std::endl;
      return;
   }
   std::vector<std::vector<float> >::iterator iter_o;
   std::vector<float>::iterator iter_i;
   std::vector<long>::iterator iter_l = steps.begin();
   
   for (iter_o = results.begin(); iter_o != results.end(); ++iter_o) {
      outFile << std::fixed << *iter_l++ << ",";
      for (iter_i = (*iter_o).begin(); iter_i != (*iter_o).end(); ++iter_i) {
         outFile << std::fixed << *iter_i;
         if (iter_i + 1 != (*iter_o).end())
            outFile << ",";
      }
      outFile << std::endl;
   }
}

void TestHostDeviceBandwidth(hipDeviceProp_t *props, TestParams &params) {
   std::cout << "Running host-device bandwidth test" << std::endl;
   //printf("\nRunning bandwidth test for %s on bus %d\n", props[0].name, props[0].pciBusID);
}

void TestP2PDeviceBandwidth(hipDeviceProp_t *props, TestParams &params){
   std::cout << "Running P2P device bandwidth test" << std::endl;
}

void TestPCIeCongestion(hipDeviceProp_t *props, TestParams &params) {
   std::cout << "Running PCIe congestion test" << std::endl;
}


void TestTaskScalability(hipDeviceProp_t *props, TestParams &params) {
   std::cout << "Running task scalability test" << std::endl;
}

void getNextLine(std::ifstream &inFile, std::string &lineStr) {
   // get lines of the input file untill the first character of the line is not a dash
   // dashes represent comments
   do { 
      if (inFile) 
         std::getline(inFile, lineStr);
   } while (inFile && lineStr[0] == '-');
}

bool getNextLineBool(std::ifstream &inFile, std::string &lineStr) {
   do { 
      if (inFile) 
         std::getline(inFile, lineStr);
   } while (inFile && lineStr[0] == '-');

   return ((lineStr.find("alse") >= lineStr.length()) ? true : false); 
}

// Function for parsing user provided input file. 
// Users must adhere to input file structure provided 
// in the sample input file to insure correct parsing
void ParseTestParameters(TestParams &params) {
   std::string lineStr;
   std::ifstream inFile(params.inputFile.c_str());

   params.useDefaultParams = false;

   getNextLine(inFile, lineStr); //resultsFile
   params.resultsFile = lineStr.substr(lineStr.find ('=') + 1);

   params.printDevProps = getNextLineBool(inFile, lineStr); //printDeviceProps
   getNextLine(inFile, lineStr);
   params.devPropFile = lineStr.substr(lineStr.find ('=') + 1); //devPropFile
  
   params.runMemoryOverheadTest = getNextLineBool(inFile, lineStr); //runMemoryOverheadTest
   params.runAllDevices = getNextLineBool(inFile, lineStr); //runAllDevices 
   for (int i = 0; i < 3; i++) {
      getNextLine(inFile, lineStr);
      int eqIdx = lineStr.find("=") + 1;
      params.rangeMemOverhead[i] = std::atol(lineStr.substr(eqIdx).c_str());
   }

   params.runHostDeviceBandwidthTest = getNextLineBool(inFile, lineStr); //runHostDeviceBandwidthTest
   params.varyBlockSizeHD = getNextLineBool(inFile, lineStr); //varyBlockSizeHD
   params.usePinnedHD = getNextLineBool(inFile, lineStr); //usePinnedHD
   params.runBurstHD = getNextLineBool(inFile, lineStr); //runBurstHD
   params.runSustainedHD = getNextLineBool(inFile, lineStr); //runSustainedHD
   for (int i = 0; i < 3; i++) {
      getNextLine(inFile, lineStr);
      int eqIdx = lineStr.find("=") + 1;
      params.rangeHostDeviceBW[i] = std::atol(lineStr.substr(eqIdx).c_str());
   }

   params.runP2PBandwidthTest = getNextLineBool(inFile, lineStr); //runP2PBandwidthTest
   params.varyBlockSizeP2P = getNextLineBool(inFile, lineStr); //varyBlockSizeP2P
   params.runBurstP2P = getNextLineBool(inFile, lineStr); //runBurstHD
   params.runSustainedP2P = getNextLineBool(inFile, lineStr); //runSustainedHD
   for (int i = 0; i < 3; i++) {
      getNextLine(inFile, lineStr);
      int eqIdx = lineStr.find("=") + 1;
      params.rangeDeviceP2P[i] = std::atol(lineStr.substr(eqIdx).c_str());
   }
   
   params.runPCIeCongestionTest = getNextLineBool(inFile, lineStr); //runPCIeCongestionTest
   params.runTaskScalabilityTest = getNextLineBool(inFile, lineStr); //runTaskScalabilityTest
   
}

//TODO:hacky print function; fix this
void PrintTestParams(TestParams &params) {

   std::string paramFileName = "benchmark_params.out";
   std::ofstream outParamFile(paramFileName.c_str());

   outParamFile << std::boolalpha;
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "---------------------- Test Parameters ---------------------" << std::endl; 
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "Input File:\t\t\t" << params.inputFile << std::endl;
   outParamFile << "Output file:\t\t\t" << params.resultsFile << std::endl;
   outParamFile << "Using Defaults:\t\t\t" << params.useDefaultParams << std::endl;  
   outParamFile << "Printing Device Props:\t\t" << params.printDevProps << std::endl;
   outParamFile << "Device Property File:\t\t" << params.devPropFile << std::endl;
   outParamFile << "Device Count:\t\t\t" << params.nDevices << std::endl;
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "Run Memory Overhead Test:\t" << params.runMemoryOverheadTest << std::endl;
   outParamFile << "Use all Devices:\t\t" << params.runAllDevices << std::endl;
   outParamFile << "Allocation Range: \t\t";
   outParamFile << params.rangeMemOverhead[0] << "," << params.rangeMemOverhead[1];
   outParamFile << "," << params.rangeMemOverhead[2] << " (min,max,step)" << std::endl;
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "Run Host-Device Bandwidth Test:\t" << params.runHostDeviceBandwidthTest << std::endl;
   outParamFile << "Vary Block Size:\t\t" << params.varyBlockSizeHD << std::endl;
   outParamFile << "Use Pinned Host Mem:\t\t" << params.usePinnedHD << std::endl;
   outParamFile << "Burst Mode:\t\t\t" << params.runBurstHD << std::endl;
   outParamFile << "Sustained Mode:\t\t\t" << params.runSustainedHD << std::endl;
   outParamFile << "Allocation Range:\t\t"; 
   outParamFile << params.rangeHostDeviceBW[0] << "," << params.rangeHostDeviceBW[1] << ","; 
   outParamFile << params.rangeHostDeviceBW[2] << " (min,max,step)" << std::endl;
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "Run P2P Bandwidth Test:\t\t" << params.runP2PBandwidthTest << std::endl;
   outParamFile << "Vary Block Size:\t\t" << params.varyBlockSizeP2P << std::endl;
   outParamFile << "Burst Mode:\t\t\t" << params.runBurstP2P << std::endl;
   outParamFile << "Sustained Mode:\t\t\t" << params.runSustainedP2P << std::endl;
   outParamFile << "Allocation Range:\t\t";
   outParamFile << params.rangeDeviceP2P[0] << "," << params.rangeDeviceP2P[1] << ",";
   outParamFile << params.rangeDeviceP2P[2] << " (min,max,step)" << std::endl;
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "Run PCIe CongestionTest:\t" << params.runPCIeCongestionTest << std::endl;
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "Run Task Scalability Test:\t" << params.runTaskScalabilityTest << std::endl; 
   outParamFile << "------------------------------------------------------------" << std::endl;    
   outParamFile << std::noboolalpha;

   //read params out to command line
   outParamFile.close();
   std::string contents;
   std::ifstream inFile(paramFileName.c_str());
   while (std::getline(inFile,contents)) {
      std::cout << contents << std::endl;
   } 
   inFile.close();

}

// Set default device properties based on an interesting variety of tests 
// in case no input file is provided. These values do necessarily reflect 
// what the developer recommends to demonstrate category performance on any 
// specific system system
void SetDefaultParams(TestParams &params) {

   params.resultsFile = "results";
   params.inputFile = "none";
   params.useDefaultParams = true;

   params.printDevProps = true;
   params.devPropFile = "device_info.out";

   params.runMemoryOverheadTest = true; 
   params.runAllDevices = true;
   params.rangeMemOverhead[0] = 1;
   params.rangeMemOverhead[1] = 1000001;
   params.rangeMemOverhead[2] = 10000;
   
   params.runHostDeviceBandwidthTest = false;
   params.varyBlockSizeHD = true;
   params.usePinnedHD = true;
   params.runBurstHD  = true;
   params.runSustainedHD = true;
   params.rangeHostDeviceBW[0] = 1;
   params.rangeHostDeviceBW[1] = 1024;
   params.rangeHostDeviceBW[2] = 2; 
  
   params.runP2PBandwidthTest = false;
   params.varyBlockSizeP2P = true;
   params.runBurstP2P = true;
   params.runSustainedP2P = true;
   params.rangeDeviceP2P[0] = 1;
   params.rangeDeviceP2P[1] = 2024;
   params.rangeDeviceP2P[2] = 2;
   
   params.runPCIeCongestionTest = false;
   
   params.runTaskScalabilityTest = false;
}

// Prints the device properties out to file based named depending on the 
void PrintDeviceProps(hipDeviceProp_t *props, TestParams &params) {
   std::cout << "\nSee " << params.devPropFile << " for information about your device's properties." << std::endl; 

   std::ofstream deviceProps(params.devPropFile.c_str());

   deviceProps << "Device Properties:" << std::endl;

   deviceProps.close();
}

// Creates an array of cudaDeviceProp structs with populated data
// located in a pre-allocated section of memory
void GetAllDeviceProps(hipDeviceProp_t *props, int dCount) {
   for (int i = 0; i < dCount; ++i) {
      hipGetDeviceProperties(&props[i], i);
   }
}

// function for cleaning up device state including profile data
// to be used before and after any test in benchmark suite.
void ResetDevices(int numToReset) {
   for (int devNum = 0; devNum < numToReset; ++devNum) {
      hipSetDevice(devNum);
      hipDeviceReset();
   }
}


#include "hip/hip_runtime.h"
//CUDA API and includes
#include<hip/hip_runtime.h>

// C/C++ standard includes
#include<memory>
#include<iostream>
#include<stdio.h>
#include<string>
#include<vector>
#include<time.h>
#include<chrono>
//OpenMP threading includes
#include<omp.h>

// NUMA Locality includes
//#include<hwloc.h>

typedef struct TestParams {
   std::string resultsFileName;
   std::string inputFile;

   bool printDevProps;
   std::string devicePropFileName;  

   //Overhead memory test for allocation and deallocation of Host and Device memory
   bool runMemoryOverheadTest;
   bool runAllDevices;
   long rangeMemOverhead[3]; //min, max and step size (in bytes)
 
   //Device-Peer PCIe Baseline bandwidth test
   bool runHostDeviceBandwidthTest;
   bool varyBlockSizeHD;
   bool usePinnedHD;
   bool runBurstHD;
   bool runSustainedHD;
   long rangeHostDeviceBWTest[3]; //min, max and step size (in bytes)

   //Peer-to-peer device memory transfer bandwidth
   bool runP2PBandwidthTest;
   bool varyBlockSizeP2P;
   bool runBurstP2P;
   bool runSustainedP2P;
   long rangeHostDeviceP2P[3]; //min, max and step size (in bytes)

   //PCIe Congestion tests
   bool runPCIeCongestionTest;

   //CUDA kernel task scalability and load balancing
   bool runTaskScalabilityTest;

} TestParams;

void RunBandwidthTestSuite(int argc, char **argv);

void PrintDeviceProps(hipDeviceProp_t *props, int dCount, TestParams &params);
void TestMemoryOverhead(hipDeviceProp_t *props, int dCount, TestParams &params);
void TestHostDeviceBandwidth(hipDeviceProp_t *props, int dCount, TestParams &params);
void TestP2PDeviceBandwidth(hipDeviceProp_t *props, int dCount, TestParams &params);
void TestPCIeCongestion(hipDeviceProp_t *props, int dCount, TestParams &params);
void TestTaskScalability(hipDeviceProp_t *props, int dCount, TestParams &params);

void SetDefaultParams(TestParams &params); 
void GetAllDeviceProps(hipDeviceProp_t *props, int dCount);
void ResetDevices(int numToReset);
void SetDefaultParams(TestParams &params); 

int main (int argc, char **argv) {

   int nDevices = 0;

   hipGetDeviceCount(&nDevices);

   if (nDevices <= 0) {
   
      printf("No devices Found\n");
      return 0;
   
   } else {

      RunBandwidthTestSuite(argc, argv);
   }

   return 0;
}

void RunBandwidthTestSuite(int argc, char **argv) {
   int nDevices = 0;
   TestParams params;

   // If command line parameters provide an input file skip this and do input parsing
   if (1) {
      SetDefaultParams(params);
   } else {
      //TODO: Parse input parameters
      
   }

   //Determine the number of recognized CUDA enabled devices
   hipGetDeviceCount(&nDevices);
   hipDeviceProp_t *props = (hipDeviceProp_t *) calloc (sizeof(hipDeviceProp_t), nDevices);

   //Aquire device properties for each CUDA enabled GPU
   GetAllDeviceProps(props, nDevices);

   //Output device properties for each CUDA enabled GPU
   if (params.printDevProps != false) {
      PrintDeviceProps(props, nDevices, params);
   }

   if (params.runMemoryOverheadTest != false ) {
      
      TestMemoryOverhead(props, nDevices, params);
   
   }

   if (params.runHostDeviceBandwidthTest != false) {

      TestHostDeviceBandwidth(props, nDevices, params);

   }

   if (params.runP2PBandwidthTest != false) {  
      
      TestP2PDeviceBandwidth(props, nDevices, params);
   
   }

   if (params.runPCIeCongestionTest != false) {

      TestPCIeCongestion(props, nDevices, params);

   }

   if (params.runTaskScalabilityTest != false) { 

      TestTaskScalability(props, nDevices, params);

   }

}

void TestMemoryOverhead(hipDeviceProp_t *props, int dCount, TestParams &params) {
      //Create CUDA runtime events used to time device operations
      hipEvent_t start_e, stop_e; 
      hipEventCreate(&start_e);
      hipEventCreate(&stop_e);

      //TODO: There is a problem with this function call on my test system; causes segfault.
      //ResetDevices(dCount);       
      
      char *deviceMem = NULL;
      char *hostUnPinnedMem = NULL;
      char *hostPinnedMem = NULL;
      float eTime = 0.0;

      //Memory overhead test will run for each device utilizing the hipMalloc and hipFree functions
      //on the first iteration of the look, assuming there is atleast one device, the host will run the 
      //pinned and un-pinned memory tests

      //Only run overhead device cases on a single device
      //default to device 0
      if (!params.runAllDevices)
         dCount = 1;

      for (int currDev = 0; currDev < dCount; currDev++) {
         printf("Running device %d (ID) of %d (total)\n", 0, dCount);
         hipSetDevice(currDev);

         for ( long chunkSize = params.rangeMemOverhead[0]; 
               chunkSize <= params.rangeMemOverhead[1]; 
               chunkSize += params.rangeMemOverhead[2]) {
         //for (int chunkSize = 0; chunkSize <= 1000; chunkSize += 100) {

            printf("Blocksize: %ld\n", chunkSize);

            //Host test only runs the first time
            if (currDev == 0) {
               //CASE 1: Allocation of host memory

               //Pinned
               hipEventRecord(start_e);
               hipHostMalloc((void **) &hostPinnedMem, chunkSize);
               hipEventRecord(stop_e);

               hipEventSynchronize(stop_e);
         
               hipEventElapsedTime(&eTime, start_e, stop_e);

               printf("Host Alloc Pinned: %lf\n", eTime);
 
               //Unpinned

               auto start_t = std::chrono::high_resolution_clock::now();
               hostUnPinnedMem = (char *) malloc(chunkSize);
               auto stop_t = std::chrono::high_resolution_clock::now();

               //printf("Host Alloc Unpinned%f\n",std::chrono::duration_cast<std::chrono::microseconds>(start_t, end_t));

               //CASE 2: Deallocation of host Memory
               
               //Pinned
               hipEventRecord(start);
               hipHostFree((void *) hostPinnedMem);
               hipEventRecord(stop);

               hipEventSynchronize(stop);
         
               hipEventElapsedTime(&eTime, start, stop);

               //printf("%lf\n", eTime);
            
               //Unpinned
               
               start_t = std::chrono::high_resolution_clock::now();
               free(hostUnpinnedMem);
               stop_t = std::chrono::high_resolution_clock::now();
               
               //printf("Host Free Unpinned"); 

            }

            //CASE 3: Allocation of device memory
            hipEventRecord(start_e);
            hipFree(deviceMem); 
            hipEventRecord(stop_e);

            hipEventSynchronize(stop);
         
            hipEventElapsedTime(&eTime, start, stop);

            printf("Device malloc: %lf\n", eTime);

            //CASE 4: DeAllocation of device memory
            hipEventRecord(start_e);
            hipFree(deviceMem); 
            hipEventRecord(stop_e);

            hipEventSynchronize(stop);
         
            hipEventElapsedTime(&eTime, start, stop);

            printf("Device free: %lf\n", eTime);

            //CASE 4: DeAllocation of device memory


            //hipDeviceSynchronize();

         }

         printf("\n");      
      }

      // cleanup CUDA runtime events
      hipEventDestroy(start);
      hipEventDestroy(stop);
}



void TestHostDeviceBandwidth(hipDeviceProp_t *props, int dCount, TestParams &params) {

   //printf("\nRunning bandwidth test for %s on bus %d\n", props[0].name, props[0].pciBusID);

}

void TestP2PDeviceBandwidth(hipDeviceProp_t *props, int dCount, TestParams &params){


}

void TestPCIeCongestion(hipDeviceProp_t *props, int dCount, TestParams &params) {


}


void TestTaskScalability(hipDeviceProp_t *props, int dCount, TestParams &params) {


}

// Set default device properties based on an interesting variety of tests 
// in case no input file is provided. These values do necessarily reflect 
// what the developer recommends to demonstrate category performance on any 
// specific system system
void SetDefaultParams(TestParams &params) {

   params.resultsFileName = "Results.csv";
   params.inputFile = "Input.txt";

   params.printDevProps = false;
   params.devicePropFileName = "DeviceInfo.txt";

   params.runMemoryOverheadTest = true; 

   params.runAllDevices = false;
   params.rangeMemOverhead[0] = 1;
   params.rangeMemOverhead[1] = 65535;
   params.rangeMemOverhead[2] = 1024;
   
   params.runHostDeviceBandwidthTest = false;
   params.varyBlockSizeHD = true;
   params.usePinnedHD = true;
   params.runBurstHD  = true;
   params.runSustainedHD = true;
   params.rangeHostDeviceBWTest[0] = 1;
   params.rangeHostDeviceBWTest[1] = 1024;
   params.rangeHostDeviceBWTest[2] = 2; 
  
   params.runP2PBandwidthTest = false;
   params.varyBlockSizeP2P = true;
   params.runBurstP2P = true;
   params.runSustainedP2P = true;
   params.rangeHostDeviceP2P[0] = 1;
   params.rangeHostDeviceP2P[1] = 2024;
   params.rangeHostDeviceP2P[2] = 2;
   
   params.runPCIeCongestionTest = false;
   
   params.runTaskScalabilityTest = false;
}

//Prints the device properties out to file based named depending on the 
void PrintDeviceProps(hipDeviceProp_t *props, int dCount, TestParams &params) {
   printf("See %s for information about your device's properties\n", params.devicePropFileName.c_str());

}

// Creates an array of hipDeviceProp_t structs with populated data
// located in a pre-allocated section of memory
void GetAllDeviceProps(hipDeviceProp_t *props, int dCount) {
   for (int i = 0; i < dCount; ++i) {
      hipGetDeviceProperties(&props[i], i);
   }
}

//function for cleaning up device state including profile data
//to be used before and after any test in benchmark suite.
void ResetDevices(int numToReset) {
   for (int devNum = 0; devNum < numToReset; ++devNum) {
      hipSetDevice(devNum);
      hipDeviceReset();
   }
}


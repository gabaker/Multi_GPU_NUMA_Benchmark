#include "hip/hip_runtime.h"
// CUDA API and includes
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

// C/C++ standard includes
#include<memory>
#include<iostream>
#include<stdio.h>
#include<string>
#include<vector>
#include<time.h>
#include<string>
#include<fstream>
#include<iostream>
#include<ios>
#include<vector>
#include<unistd.h>


#include<sys/time.h>
//#include<chrono>

#ifdef USING_CPP
#include<chrono>
#include<tuple>
#endif
// OpenMP threading includes
#include<omp.h>

// NUMA Locality includes
//#include<hwloc.h>

#define MILLI_TO_MICRO (1.0 / 1000.0)
#define MICRO_TO_MILLI (1000.0)
#define NANO_TO_MILLI (1.0 / 1000000.0)
#define NANO_TO_MICRO (1.0 / 1000.0)

typedef struct TestParams {
   std::string resultsFile;
   std::string inputFile;
   bool useDefaultParams;

   bool printDevProps;
   std::string devPropFile;

   int nDevices;

   // Overhead memory test for allocation and deallocation of Host and Device memory
   bool runMemoryOverheadTest;
   bool runAllDevices;
   long rangeMemOverhead[3]; //min, max and step size (in bytes)
 
   // Device-Peer PCIe Baseline bandwidth test
   bool runHostDeviceBandwidthTest;
   bool varyBlockSizeHD;
   bool usePinnedHD;
   bool runBurstHD;
   bool runSustainedHD;
   long rangeHostDeviceBW[3]; //min, max and step size (in bytes)

   // Peer-to-peer device memory transfer bandwidth
   bool runP2PBandwidthTest;
   bool varyBlockSizeP2P;
   bool runBurstP2P;
   bool runSustainedP2P;
   long rangeDeviceP2P[3]; //min, max and step size (in bytes)

   // PCIe Congestion tests
   bool runPCIeCongestionTest;

   // CUDA kernel task scalability and load balancing
   bool runTaskScalabilityTest;

} TestParams;

typedef enum
{
DEVICE_MALLOC,
HOST_MALLOC,
HOST_PINNED_MALLOC,
DEVICE_FREE,
HOST_FREE,
HOST_PINNED_FREE
} MEM_OP;

void RunBandwidthTestSuite(TestParams &params);

void PrintDeviceProps(hipDeviceProp_t *props, TestParams &params);
void TestMemoryOverhead(hipDeviceProp_t *props, TestParams &params);
void TestHostDeviceBandwidth(hipDeviceProp_t *props, TestParams &params);
void TestP2PDeviceBandwidth(hipDeviceProp_t *props, TestParams &params);
void TestPCIeCongestion(hipDeviceProp_t *props, TestParams &params);
void TestTaskScalability(hipDeviceProp_t *props, TestParams &params);
void ParseTestParameters(TestParams &params);

void SetDefaultParams(TestParams &params); 
void GetAllDeviceProps(hipDeviceProp_t *props, int dCount);
void ResetDevices(int numToReset);
void SetDefaultParams(TestParams &params); 
void PrintTestParams(TestParams &params);
void getNextLine(std::ifstream &inFile, std::string &lineStr);
void printResults(std::ofstream &outFile, std::vector<long> &steps, std::vector<std::vector<float> > &results, TestParams &params); 

int main (int argc, char **argv) {
   TestParams params;
 
   
   std::cout << "\nStarting Multi-GPU Performance Test Suite...\n" << std::endl; 

   // Determine the number of recognized CUDA enabled devices
   hipGetDeviceCount(&(params.nDevices));

   if (params.nDevices <= 0) {
      std::cout << "No devices found...aborting benchmarks." << std::endl;
      exit(-1);
   }

   // Setup benchmark parameters
   if (argc == 1) { //No input file, use default parameters
   
      SetDefaultParams(params);
   
   } else if (argc == 2) { //Parse input file
   
      params.inputFile = std::string(argv[1]);
      ParseTestParameters(params);
   
   } else { //Unknown input parameter list, abort test
      std::cout << "Aborting test: Incorrect number of input parameters" << std::endl;
      exit(-1);
   }


   PrintTestParams(params);
   RunBandwidthTestSuite(params);

   return 0;
}

void RunBandwidthTestSuite(TestParams &params) {
   hipDeviceProp_t *props = (hipDeviceProp_t *) calloc (sizeof(hipDeviceProp_t), params.nDevices);

   // Aquire device properties for each CUDA enabled GPU
   GetAllDeviceProps(props, params.nDevices);

   if (params.runMemoryOverheadTest != false ) {
      
      TestMemoryOverhead(props, params);
   
   }

   if (params.runHostDeviceBandwidthTest != false) {

      TestHostDeviceBandwidth(props, params);

   }

   if (params.runP2PBandwidthTest != false) {  
      
      TestP2PDeviceBandwidth(props, params);
   
   }

   if (params.runPCIeCongestionTest != false) {

      TestPCIeCongestion(props, params);

   }

   if (params.runTaskScalabilityTest != false) { 

      TestTaskScalability(props, params);

   }

   // Output device properties for each CUDA enabled GPU
   if (params.printDevProps != false) {
      PrintDeviceProps(props, params);
   }

   std::cout << "\n\nBenchmarks complete!\n" << std::endl;

   free(props);
}

float TimedMemOp(void **MemBlk, long NumBytes, MEM_OP TimedOp) {
   #ifdef USING_CPP
   std::chrono::high_resolution_clock::time_point start_c, stop_c;
   auto total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
   #else
   struct timeval stop_t, start_t, total_t;
   #endif
   
   hipEvent_t start_e, stop_e; 
   hipEventCreate(&start_e);
   hipEventCreate(&stop_e);
   float OpTime = 0;
  
   switch (TimedOp) {
      case HOST_MALLOC:
         #ifdef USING_CPP
         start_c = std::chrono::high_resolution_clock::now();
         *MemBlk = malloc(NumBytes);
         stop_c = std::chrono::high_resolution_clock::now();
         total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);      
         OpTime = (float) total_c.count() * NANO_TO_MILLI;
         #else
         gettimeofday(&start_t, NULL);
         *MemBlk = malloc(NumBytes); 
         gettimeofday(&stop_t, NULL);
         timersub(&stop_t, &start_t, &total_t);
         OpTime = (float) total_t.tv_usec * MICRO_TO_MILLI;
         #endif
         break;
      case HOST_PINNED_MALLOC:
         hipEventRecord(start_e, 0);      
         hipHostMalloc(MemBlk, NumBytes);
         hipEventRecord(stop_e, 0);
         hipEventSynchronize(stop_e);
         hipEventElapsedTime(&OpTime, start_e, stop_e);
         break;
      case DEVICE_MALLOC:
         checkCudaErrors(hipEventRecord(start_e, 0));
         checkCudaErrors(hipMalloc(MemBlk, NumBytes));
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e)); 
         break;
      case HOST_FREE:
         #ifdef USING_CPP
         start_c = std::chrono::high_resolution_clock::now();
         free(*MemBlk);
         stop_c = std::chrono::high_resolution_clock::now(); 
         total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
         OpTime = (float) total_c.count() * NANO_TO_MILLI;
         #else
         gettimeofday(&start_t, NULL);
         free(*MemBlk); 
         gettimeofday(&stop_t, NULL); 
         timersub(&stop_t, &start_t, &total_t); 
         OpTime = (float) total_t.tv_usec * MICRO_TO_MILLI;
         #endif
         break;
      case HOST_PINNED_FREE:
         hipEventRecord(start_e, 0);
         hipHostFree(*MemBlk);
         hipEventRecord(stop_e, 0);
         hipEventSynchronize(stop_e);
         hipEventElapsedTime(&OpTime, start_e, stop_e);
         break;
      case DEVICE_FREE:
         checkCudaErrors(hipEventRecord(start_e, 0));
         checkCudaErrors(hipFree(*MemBlk)); 
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));   
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e));  
         break;
      default:
         std::cout << "Error: unrecognized timed memory operation type" << std::cout; 
         break;
   }
   hipEventDestroy(start_e);
   hipEventDestroy(stop_e);

   return OpTime;
}

void TestMemoryOverhead(hipDeviceProp_t *props, TestParams &params) {
      char *deviceMem = NULL;
      char *hostMem = NULL;
      char *hostPinnedMem = NULL;
      int nDevices = params.nDevices;

      std::vector<long> blockSteps;
      std::vector<std::vector<float> > overheadData;

      // Only run overhead device cases on a single device
      // default to device 0
      if (!params.runAllDevices)
         nDevices = 1;
      
      // Memory overhead test will run for each device utilizing the hipMalloc and hipFree functions
      // on the first iteration of the look, assuming there is atleast one device, the host will run the 
      // pinned and un-pinned memory tests
      for (int currDev = 0; currDev < nDevices; currDev++) {
         checkCudaErrors(hipSetDevice(currDev));
 
         std::vector<float> chunkData;
         long stepNum = 0;
         long stepSize = params.rangeMemOverhead[0];// / params.rangeMemOverhead[2];
         //stepSize = (stepSize) ? stepSize : params.rangeMemOverhead[0]; 
         for ( long chunkSize = params.rangeMemOverhead[0]; 
               chunkSize <= params.rangeMemOverhead[1]; 
               chunkSize += stepSize) { 

            if (currDev == 0) {
               blockSteps.push_back(chunkSize); 
               //CASE 1: Host Pinned Memory Overhead
               chunkData.push_back(TimedMemOp((void **) &hostPinnedMem, chunkSize, HOST_PINNED_MALLOC));
               chunkData.push_back(TimedMemOp((void **) &hostPinnedMem, 0, HOST_PINNED_FREE)); 
               //CASE 2: Host UnPinned Memory Overhead
               chunkData.push_back(TimedMemOp((void **) &hostMem, 0, HOST_FREE));
               chunkData.push_back(TimedMemOp((void **) &hostMem, chunkSize, HOST_MALLOC));
            }
            // CASE 3: Allocation of device memory  
            chunkData.push_back(TimedMemOp((void **) &deviceMem, chunkSize, DEVICE_MALLOC));
            // CASE 4: DeAllocation of device memory 
            chunkData.push_back(TimedMemOp((void **) &deviceMem, 0, DEVICE_FREE));
            
            //Add device/host run data to correct location of data vector
            if (currDev == 0) {
               overheadData.push_back(chunkData); 
            } else {
               overheadData[stepNum].push_back(chunkData[0]);
               overheadData[stepNum].push_back(chunkData[1]);
            }
            chunkData.clear(); //clear chunkData for next mem step 

            //Move to next stepSize after every numSteps as set by the param file
            long stride = (params.rangeMemOverhead[2] - 1) ? (params.rangeMemOverhead[2] - 1) : 1;
            if (stepNum && (stepNum % stride) == 0) {
               stepSize *= 2;
            }
            stepNum++;
         }
      }

      std::string dataFileName = params.resultsFile + "_overhead.csv";
      std::ofstream overheadResultsFile(dataFileName.c_str());
      printResults(overheadResultsFile, blockSteps, overheadData, params);
}

void printResults(std::ofstream &outFile, std::vector<long> &steps, std::vector<std::vector<float> > &results, TestParams &params) {
   //std::cout.setf(std::ios::showpoint);
   
   if (!outFile.is_open()) {
      std::cout << "Failed to open file to print results" << std::endl;
      return;
   }
   std::vector<std::vector<float> >::iterator iter_o;
   std::vector<float>::iterator iter_i;
   std::vector<long>::iterator iter_l = steps.begin();
   
   for (iter_o = results.begin(); iter_o != results.end(); ++iter_o) {
      outFile << std::fixed << *iter_l++ << ",";
      for (iter_i = (*iter_o).begin(); iter_i != (*iter_o).end(); ++iter_i) {
         outFile << std::fixed << *iter_i;
         if (iter_i + 1 != (*iter_o).end())
            outFile << ",";
      }
      outFile << std::endl;
   }
}

void TestHostDeviceBandwidth(hipDeviceProp_t *props, TestParams &params) {
   std::cout << "Running host-device bandwidth test" << std::endl;
   //printf("\nRunning bandwidth test for %s on bus %d\n", props[0].name, props[0].pciBusID);
}

void TestP2PDeviceBandwidth(hipDeviceProp_t *props, TestParams &params){
   std::cout << "Running P2P device bandwidth test" << std::endl;
}

void TestPCIeCongestion(hipDeviceProp_t *props, TestParams &params) {
   std::cout << "Running PCIe congestion test" << std::endl;
}

void TestTaskScalability(hipDeviceProp_t *props, TestParams &params) {
   std::cout << "Running task scalability test" << std::endl;
}

void getNextLine(std::ifstream &inFile, std::string &lineStr) {
   // get lines of the input file untill the first character of the line is not a dash
   // dashes represent comments
   do { 
      if (inFile) 
         std::getline(inFile, lineStr);
   } while (inFile && lineStr[0] == '-');
}

bool getNextLineBool(std::ifstream &inFile, std::string &lineStr) {
   do { 
      if (inFile) 
         std::getline(inFile, lineStr);
   } while (inFile && lineStr[0] == '-');

   return ((lineStr.find("alse") >= lineStr.length()) ? true : false); 
}

// Function for parsing user provided input file. 
// Users must adhere to input file structure provided 
// in the sample input file to insure correct parsing
void ParseTestParameters(TestParams &params) {
   std::string lineStr;
   std::ifstream inFile(params.inputFile.c_str());

   params.useDefaultParams = false;

   getNextLine(inFile, lineStr); //resultsFile
   params.resultsFile = lineStr.substr(lineStr.find ('=') + 1);

   params.printDevProps = getNextLineBool(inFile, lineStr); //printDeviceProps
   getNextLine(inFile, lineStr);
   params.devPropFile = lineStr.substr(lineStr.find ('=') + 1); //devPropFile
  
   params.runMemoryOverheadTest = getNextLineBool(inFile, lineStr); //runMemoryOverheadTest
   params.runAllDevices = getNextLineBool(inFile, lineStr); //runAllDevices 
   for (int i = 0; i < 3; i++) {
      getNextLine(inFile, lineStr);
      int eqIdx = lineStr.find("=") + 1;
      params.rangeMemOverhead[i] = std::atol(lineStr.substr(eqIdx).c_str());
   }

   params.runHostDeviceBandwidthTest = getNextLineBool(inFile, lineStr); //runHostDeviceBandwidthTest
   params.varyBlockSizeHD = getNextLineBool(inFile, lineStr); //varyBlockSizeHD
   params.usePinnedHD = getNextLineBool(inFile, lineStr); //usePinnedHD
   params.runBurstHD = getNextLineBool(inFile, lineStr); //runBurstHD
   params.runSustainedHD = getNextLineBool(inFile, lineStr); //runSustainedHD
   for (int i = 0; i < 3; i++) {
      getNextLine(inFile, lineStr);
      int eqIdx = lineStr.find("=") + 1;
      params.rangeHostDeviceBW[i] = std::atol(lineStr.substr(eqIdx).c_str());
   }

   params.runP2PBandwidthTest = getNextLineBool(inFile, lineStr); //runP2PBandwidthTest
   params.varyBlockSizeP2P = getNextLineBool(inFile, lineStr); //varyBlockSizeP2P
   params.runBurstP2P = getNextLineBool(inFile, lineStr); //runBurstHD
   params.runSustainedP2P = getNextLineBool(inFile, lineStr); //runSustainedHD
   for (int i = 0; i < 3; i++) {
      getNextLine(inFile, lineStr);
      int eqIdx = lineStr.find("=") + 1;
      params.rangeDeviceP2P[i] = std::atol(lineStr.substr(eqIdx).c_str());
   }
   
   params.runPCIeCongestionTest = getNextLineBool(inFile, lineStr); //runPCIeCongestionTest
   params.runTaskScalabilityTest = getNextLineBool(inFile, lineStr); //runTaskScalabilityTest
   
}

//TODO:hacky print function; fix this
void PrintTestParams(TestParams &params) {

   std::string paramFileName = "benchmark_params.out";
   std::ofstream outParamFile(paramFileName.c_str());

   outParamFile << std::boolalpha;
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "---------------------- Test Parameters ---------------------" << std::endl; 
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "Input File:\t\t\t" << params.inputFile << std::endl;
   outParamFile << "Output file:\t\t\t" << params.resultsFile << std::endl;
   outParamFile << "Using Defaults:\t\t\t" << params.useDefaultParams << std::endl;  
   outParamFile << "Printing Device Props:\t\t" << params.printDevProps << std::endl;
   outParamFile << "Device Property File:\t\t" << params.devPropFile << std::endl;
   outParamFile << "Device Count:\t\t\t" << params.nDevices << std::endl;
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "Run Memory Overhead Test:\t" << params.runMemoryOverheadTest << std::endl;
   outParamFile << "Use all Devices:\t\t" << params.runAllDevices << std::endl;
   outParamFile << "Allocation Range: \t\t";
   outParamFile << params.rangeMemOverhead[0] << "," << params.rangeMemOverhead[1];
   outParamFile << "," << params.rangeMemOverhead[2] << " (min,max,step)" << std::endl;
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "Run Host-Device Bandwidth Test:\t" << params.runHostDeviceBandwidthTest << std::endl;
   outParamFile << "Vary Block Size:\t\t" << params.varyBlockSizeHD << std::endl;
   outParamFile << "Use Pinned Host Mem:\t\t" << params.usePinnedHD << std::endl;
   outParamFile << "Burst Mode:\t\t\t" << params.runBurstHD << std::endl;
   outParamFile << "Sustained Mode:\t\t\t" << params.runSustainedHD << std::endl;
   outParamFile << "Allocation Range:\t\t"; 
   outParamFile << params.rangeHostDeviceBW[0] << "," << params.rangeHostDeviceBW[1] << ","; 
   outParamFile << params.rangeHostDeviceBW[2] << " (min,max,step)" << std::endl;
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "Run P2P Bandwidth Test:\t\t" << params.runP2PBandwidthTest << std::endl;
   outParamFile << "Vary Block Size:\t\t" << params.varyBlockSizeP2P << std::endl;
   outParamFile << "Burst Mode:\t\t\t" << params.runBurstP2P << std::endl;
   outParamFile << "Sustained Mode:\t\t\t" << params.runSustainedP2P << std::endl;
   outParamFile << "Allocation Range:\t\t";
   outParamFile << params.rangeDeviceP2P[0] << "," << params.rangeDeviceP2P[1] << ",";
   outParamFile << params.rangeDeviceP2P[2] << " (min,max,step)" << std::endl;
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "Run PCIe CongestionTest:\t" << params.runPCIeCongestionTest << std::endl;
   outParamFile << "------------------------------------------------------------" << std::endl; 
   outParamFile << "Run Task Scalability Test:\t" << params.runTaskScalabilityTest << std::endl; 
   outParamFile << "------------------------------------------------------------" << std::endl;    
   outParamFile << std::noboolalpha;

   //read params out to command line
   outParamFile.close();
   std::string contents;
   std::ifstream inFile(paramFileName.c_str());
   while (std::getline(inFile,contents)) {
      std::cout << contents << std::endl;
   } 
   inFile.close();

}

// Set default device properties based on an interesting variety of tests 
// in case no input file is provided. These values do necessarily reflect 
// what the developer recommends to demonstrate category performance on any 
// specific system system
void SetDefaultParams(TestParams &params) {

   params.resultsFile = "results";
   params.inputFile = "none";
   params.useDefaultParams = true;

   params.printDevProps = true;
   params.devPropFile = "device_info.out";

   params.runMemoryOverheadTest = true; 
   params.runAllDevices = true;
   params.rangeMemOverhead[0] = 1;
   params.rangeMemOverhead[1] = 1000001;
   params.rangeMemOverhead[2] = 10000;
   
   params.runHostDeviceBandwidthTest = false;
   params.varyBlockSizeHD = true;
   params.usePinnedHD = true;
   params.runBurstHD  = true;
   params.runSustainedHD = true;
   params.rangeHostDeviceBW[0] = 1;
   params.rangeHostDeviceBW[1] = 1024;
   params.rangeHostDeviceBW[2] = 2; 
  
   params.runP2PBandwidthTest = false;
   params.varyBlockSizeP2P = true;
   params.runBurstP2P = true;
   params.runSustainedP2P = true;
   params.rangeDeviceP2P[0] = 1;
   params.rangeDeviceP2P[1] = 2024;
   params.rangeDeviceP2P[2] = 2;
   
   params.runPCIeCongestionTest = false;
   
   params.runTaskScalabilityTest = false;
}

// Prints the device properties out to file based named depending on the 
void PrintDeviceProps(hipDeviceProp_t *props, TestParams &params) {
   std::cout << "\nSee " << params.devPropFile << " for information about your device's properties." << std::endl; 

   std::ofstream deviceProps(params.devPropFile.c_str());

   deviceProps << "Device Properties:" << std::endl;

   deviceProps.close();
}

// Creates an array of hipDeviceProp_t structs with populated data
// located in a pre-allocated section of memory
void GetAllDeviceProps(hipDeviceProp_t *props, int dCount) {
   for (int i = 0; i < dCount; ++i) {
      hipGetDeviceProperties(&props[i], i);
   }
}

// function for cleaning up device state including profile data
// to be used before and after any test in benchmark suite.
void ResetDevices(int numToReset) {
   for (int devNum = 0; devNum < numToReset; ++devNum) {
      hipSetDevice(devNum);
      hipDeviceReset();
   }
}


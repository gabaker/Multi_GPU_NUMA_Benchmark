#include "hip/hip_runtime.h"

// Benchmark includes and defines
#ifndef BENCH_HEADER_INC
#define BENCH_HEADER_INC
#include "benchmark.h"
#endif

// BenchParams class definition
#ifndef PARAM_CLASS_INC
#include "parameters.h"
#define PARAM_CLASS_INC
#endif

// SystemTopo class definition
#ifndef TOPOLOGY_CLASS_INC
#include "topology.h"
#define TOPOLOGY_CLASS_INC
#endif

// Benchmark Tests
void RunBenchmarkSuite(BenchParams &params, SystemTopo &topo);
void TestMemoryOverhead(BenchParams &params, SystemTopo &topo);
void HHBurstTransferTest(BenchParams &params, SystemTopo &topo);
void HDBurstTransferTest(BenchParams &params, SystemTopo &topo);
void P2PBurstTransferTest(BenchParams &params, SystemTopo &topo);
void HHRangeTransferTest(BenchParams &params, SystemTopo &topo);
void HDRangeTransferTest(BenchParams &params, SystemTopo &topo);
void P2PRangeTransferTest(BenchParams &params, SystemTopo &topo);
void TestPCIeCongestion(BenchParams &params, SystemTopo &topo);
void TestTaskScalability(BenchParams &params, SystemTopo &topo);

// Test Subfunctions
void MemCopyRun(SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, MEM_OP copyType, MEM_PATTERN patternType, int destIdx, int srcIdx, int numCopiesPerStep); 
float TimedMemOp(void **MemBlk, long long NumBytes, MEM_OP TimedOp); 
float TimedMemCopyStep(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, MEM_PATTERN patternType, int destIdx = 0, int srcIdx = 0);
float BurstMemCopy(SystemTopo &topo, long long blockSize, MEM_OP copyType, int destIdx, int srcIdx, int numSteps, MEM_PATTERN pattern = REPEATED); 
void MemCopyOp(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, int destIdx = 0, int srcIdx = 0);

void RangeHDBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData); 
void RangeHHBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData); 
void RangeP2PBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData);

void BurstHDBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData); 
void BurstHHBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData); 
void BurstP2PBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData);  

// Support functions
void AllocateMemBlock(SystemTopo &topo, void **destPtr, void **srcPtr, long long numBytes, MEM_OP copyType, int destIdx = 0, int srcIdx = 0);
void FreeMemBlock(SystemTopo &topo, void* destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx = 0, int srcIdx = 0);
void SetMemBlockTransfer(SystemTopo &topo, void *destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx, int srcIdx, long long value); 
int CalcRunSteps(std::vector<long long> &blockSteps, long long startStep, long long stopStep, long long numSteps);

// Results output
void PrintRangedHeader(BenchParams &params, SystemTopo &topo, std::ofstream &fileStream, BW_RANGED_TYPE testType); 
void PrintResults(std::ofstream &outFile, std::vector<long long> &steps, std::vector<std::vector<float> > &results, BenchParams &params);
void PrintHHBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData);
void PrintHDBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData);
void PrintP2PBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData);

std::vector<std::string> PatternNames{"Repeated","Random", "Linear Increasing","Linear Decreasing"};
 
/* Benchmark main()
 * 
 * 
 */
int main (int argc, char **argv) {
   BenchParams benchParams;  
   SystemTopo sysTopo;
   
   std::cout << "\nStarting Multi-GPU, Multi-NUMA Performance Test Suite...\n" << std::endl; 
   
   // Determine the number of recognized CUDA enabled devices
   checkCudaErrors(hipGetDeviceCount(&(benchParams.nDevices)));

   // Exit if system contains no devices
   if (benchParams.nDevices <= 0) {
      std::cout << "No devices found...aborting benchmarks." << std::endl;
      exit(-1);
   }

   // Setup benchmark parameters
   if (argc == 1) { 
      // No input file, use default parameters
      benchParams.SetDefault();
   
   } else if (argc == 2) {       
      // Parse input file and set parameter class local variables
      benchParams.ParseParamFile(std::string(argv[1]));

   } else {
      // Unknown input parameter list, abort test
      std::cout << "Aborting test: Incorrect number of input parameters" << std::endl;
      exit(-1);
   }

   // Print HWLOC topology info
   // Class constructor parses system topology from device files (linux)
   std::string topoFileName ="./results/" + benchParams.topoFile;
   std::ofstream topoFile(topoFileName.c_str());

   // Output system topology to file and screen
   sysTopo.PrintTopology(topoFile);

   // Output device properties for each CUDA enabled GPU to file
   sysTopo.PrintDeviceProps(benchParams);

   // Check parameters and fix parameters associated with boolean flags
   if (benchParams.runSustainedTests == false)
      benchParams.numStepRepeats = 1;

   if (benchParams.runSocketTests)
      benchParams.nSockets = sysTopo.NumSockets();

   if (!benchParams.runAllDevices)
      benchParams.nDevices = 1;

   // Print actual benchmark parameters for user/script parsing
   benchParams.PrintParams();

   // Run the benchmark per parameters defined in params
   RunBenchmarkSuite(benchParams, sysTopo);

   std::cout << "\nBenchmarks complete!\n" << std::endl;
  
   return 0;
}

void RunBenchmarkSuite(BenchParams &params, SystemTopo &topo) {

   // Memory overhead tests (allocation and deallocations)
   if (params.runMemoryOverheadTest)
      TestMemoryOverhead(params, topo);

   //Burst bandwidth tests (matricies printed)
   if (params.runHHBandwidthTest && params.runBurstTests)
      HHBurstTransferTest(params, topo);
   
   if (params.runHDBandwidthTest && params.runBurstTests)
      HDBurstTransferTest(params, topo);

   if (params.runP2PBandwidthTest && params.runBurstTests && params.runAllDevices)
      P2PBurstTransferTest(params, topo);

   //Ranged bandwidth tests (cvs files printed)
   if (params.runHHBandwidthTest && params.runRangeTests)
      HHRangeTransferTest(params, topo);
   
   if (params.runHDBandwidthTest && params.runRangeTests)
      HDRangeTransferTest(params, topo);

   if (params.runP2PBandwidthTest && params.runRangeTests && params.runAllDevices)
      P2PRangeTransferTest(params, topo);

   // Congestion benchmark tests
   if (params.runPCIeCongestionTest)
      TestPCIeCongestion(params, topo);

   if (params.runTaskScalabilityTest) 
      TestTaskScalability(params, topo);

}

void TestMemoryOverhead(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Ranged Memory Overhead Test...\n" << std::endl;
   
   char *deviceMem = NULL, *hostMem = NULL, *hostPinnedMem = NULL;
   std::vector<long long> steps;
   std::vector<std::vector<float> > overheadData;
   int testNum = 0;
  
   CalcRunSteps(steps, params.rangeMemOverhead[0], params.rangeMemOverhead[1], params.rangeMemOverhead[2]);  
   overheadData.resize(steps.size());
   
   // Memory overhead test will run for each device utilizing the hipMalloc and hipFree functions
   // on the first iteration of the look, assuming there is atleast one device, the host will run the 
   // pinned and un-pinned memory tests
   for (int socketIdx = 0; socketIdx < params.nSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int numaIdx = 0; numaIdx < topo.NumNodes(); numaIdx++) { 
         topo.PinNumaNode(numaIdx);
        
         std::cout << "Test " << testNum++ << " Host Alloc/Free, Pinned/Pageable\t" << "NUMA node: " << numaIdx << " CPU " << socketIdx << std::endl;            
         // Host based management for CASE 1 & 2
         for (long stepIdx = 0; stepIdx < steps.size(); stepIdx++) {
            long long chunkSize = steps[stepIdx];
            float pinAllocTime = 0, pinFreeTime = 0, hostAllocTime = 0, hostFreeTime = 0;
            
            // repeat same block run and average times
            for (int reIdx = 0; reIdx < params.numStepRepeats; reIdx++) {
               //CASE 2: Host UnPinned Memory Overhead
               hostFreeTime += TimedMemOp((void **) &hostMem, chunkSize, HOST_MALLOC);
               hostAllocTime += TimedMemOp((void **) &hostMem, 0, HOST_FREE);

               if (params.usePinnedMem) {
                  //CASE 1: Host Pinned Memory Overhead
                  pinAllocTime += TimedMemOp((void **) &hostPinnedMem, chunkSize, HOST_PINNED_MALLOC);
                  pinFreeTime += TimedMemOp((void **) &hostPinnedMem, 0, HOST_PINNED_FREE); 
               }
            }
            overheadData[stepIdx].push_back(hostAllocTime / (float) params.numStepRepeats);
            overheadData[stepIdx].push_back(hostFreeTime / (float) params.numStepRepeats);

            overheadData[stepIdx].push_back(pinAllocTime / (float) params.numStepRepeats);
            overheadData[stepIdx].push_back(pinFreeTime / (float) params.numStepRepeats);
         }   
      }

      topo.PinNumaNode(0);
      // Device based memory management for CASE 3 & 4
      for (int currDev = 0; currDev < params.nDevices; currDev++) {
         checkCudaErrors(hipSetDevice(currDev)); 
         std::cout << "Test " << testNum++ << " Device Alloc/Free \t\t" << " CPU " << socketIdx << " Dev:" << currDev << std::endl;            
         
         for (long stepIdx = 0; stepIdx < steps.size(); stepIdx++) {
            long long chunkSize = steps[stepIdx];
            float devAllocTime = 0, devFreeTime = 0;

            // repeat same block run and average times
            for (int reIdx = 0; reIdx < params.numStepRepeats; reIdx++) {
               // CASE 3: Allocation of device memory  
               devAllocTime += TimedMemOp((void **) &deviceMem, chunkSize, DEVICE_MALLOC);
               // CASE 4: DeAllocation of device memory 
               devFreeTime += TimedMemOp((void **) &deviceMem, 0, DEVICE_FREE);
            }

            overheadData[stepIdx].push_back(devAllocTime / (float) params.numStepRepeats);
            overheadData[stepIdx].push_back(devFreeTime / (float) params.numStepRepeats);
         }
      }
   }
   
   std::string dataFileName = "./results/" + params.resultsFile + "_overhead.csv";
   std::ofstream overheadResultsFile(dataFileName.c_str());
   overheadResultsFile << params.nSockets << ",";
   overheadResultsFile << topo.NumNodes() << ",";
   overheadResultsFile << params.nDevices;
   if (params.usePinnedMem)
      overheadResultsFile << ",t";
   else 
      overheadResultsFile << ",f";

   for (int i = 0; i < params.nDevices; i++)
      overheadResultsFile << "," << topo.GetDeviceName(i);
   overheadResultsFile << std::endl;
   PrintResults(overheadResultsFile, steps, overheadData, params);

   std::cout << "\nMemory Overhead Test Complete!" << std::endl;
   
}

void HHBurstTransferTest(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Host-Host Burst Bandwidth Tests...\n" << std::endl;

   std::vector<std::vector<float> > burstData;

   BurstHHBandwidthRun(params, topo, burstData); 
   PrintHHBurstMatrix(params, topo, burstData);
}

void HDBurstTransferTest(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Host-Device Burst Bandwidth Tests...\n" << std::endl;

   std::vector<std::vector<float> > burstData;

   BurstHDBandwidthRun(params, topo, burstData);  
   
   PrintHDBurstMatrix(params, topo, burstData);
   
}

void P2PBurstTransferTest(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Device-Device Burst Bandwidth Tests...\n" << std::endl;
   
   std::vector<std::vector<float> > burstData;

   BurstP2PBandwidthRun(params, topo, burstData);
 
   PrintP2PBurstMatrix(params, topo, burstData);
}

void HHRangeTransferTest(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Ranged Host-Host Bandwidth Tests...\n" << std::endl;
   
   std::vector<std::vector<float> > rangeData;
   std::vector<long long> steps;
  
   CalcRunSteps(steps, params.rangeHostHostBW[0], params.rangeHostHostBW[1], params.rangeHostHostBW[2]); 
   rangeData.resize(steps.size());
   
   RangeHHBandwidthRun(params, topo, steps, rangeData);

   // tt == Transfer Time
   std::string dataFileName = "./results/" + params.resultsFile + "_ranged_hh_tt.csv";
   std::ofstream ttResultsFileHH(dataFileName.c_str());
   PrintRangedHeader(params, topo, ttResultsFileHH, HH); 
   PrintResults(ttResultsFileHH, steps, rangeData, params);

   // Output throughput (GB/S) and block size
   for (int blkIdx = 0; blkIdx < steps.size(); ++blkIdx) {
      for (int runIdx = 0; runIdx < rangeData[blkIdx].size(); ++runIdx) {
         rangeData[blkIdx][runIdx] = ((double) steps[blkIdx]) / rangeData[blkIdx][runIdx];
         rangeData[blkIdx][runIdx] /= pow(2.0, 30.0);
      }
   }

   dataFileName = "./results/" + params.resultsFile + "_ranged_hh_bw.csv";
   std::ofstream bwResultsFileHH(dataFileName.c_str());
   PrintRangedHeader(params, topo, bwResultsFileHH, HH); 
   PrintResults(bwResultsFileHH, steps, rangeData, params);

   std::cout << "\nRanged Host-Host Bandwidth Tests complete!" << std::endl;
}

void HDRangeTransferTest(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Ranged Host-Device Bandwidth Tests...\n" << std::endl;
   
   std::vector<std::vector<float> > rangeData;
   std::vector<long long> steps;

   CalcRunSteps(steps, params.rangeHostDeviceBW[0], params.rangeHostDeviceBW[1], params.rangeHostDeviceBW[2]); 
   rangeData.resize(steps.size());
   
   RangeHDBandwidthRun(params, topo, steps, rangeData);
   
   // tt == Transfer Time
   std::string dataFileName = "./results/" + params.resultsFile + "_ranged_hd_tt.csv";
   std::ofstream ttResultsFileHD(dataFileName.c_str());
   PrintRangedHeader(params, topo, ttResultsFileHD, HD); 
   PrintResults(ttResultsFileHD, steps, rangeData, params);

   // Output throughput (GB/S) and block size
   for (int blkIdx = 0; blkIdx < steps.size(); ++blkIdx) {
      for (int runIdx = 0; runIdx < rangeData[blkIdx].size(); ++runIdx) {
         rangeData[blkIdx][runIdx] = ((double) steps[blkIdx]) / rangeData[blkIdx][runIdx];
         rangeData[blkIdx][runIdx] /= pow(2.0, 30.0);
      }
   }

   dataFileName = "./results/" + params.resultsFile + "_ranged_hd_bw.csv";
   std::ofstream bwResultsFileHD(dataFileName.c_str());
   PrintRangedHeader(params, topo, bwResultsFileHD, HD); 
   PrintResults(bwResultsFileHD, steps, rangeData, params);

   std::cout << "\nHost-Device Ranged Bandwidth Tests complete!" << std::endl;
}

void P2PRangeTransferTest(BenchParams &params, SystemTopo &topo){
   std::cout << "\nRunning P2P Device Ranged Bandwidth test..." << std::endl;

   std::vector<std::vector<float> > rangeData;
   std::vector<long long> steps;

   CalcRunSteps(steps, params.rangeDeviceBW[0], params.rangeDeviceBW[1], params.rangeDeviceBW[2]); 
   rangeData.resize(steps.size());
   
   RangeP2PBandwidthRun(params, topo, steps, rangeData);

   // tt == Transfer Time
   std::string dataFileName = "./results/" + params.resultsFile + "_ranged_p2p_tt.csv";
   std::ofstream ttResultsFileP2P(dataFileName.c_str());
   PrintRangedHeader(params, topo, ttResultsFileP2P, P2P); 
   PrintResults(ttResultsFileP2P, steps, rangeData, params);

   // Output throughput (GB/S) and block size
   for (int blkIdx = 0; blkIdx < steps.size(); ++blkIdx) {
      for (int runIdx = 0; runIdx < rangeData[blkIdx].size(); ++runIdx) {
         rangeData[blkIdx][runIdx] = ((double) steps[blkIdx]) / rangeData[blkIdx][runIdx];
         rangeData[blkIdx][runIdx] /= pow(2.0, 30.0);
      }
   }

   dataFileName = "./results/" + params.resultsFile + "_ranged_p2p_bw.csv";
   std::ofstream bwResultsFileP2P(dataFileName.c_str());
   PrintRangedHeader(params, topo, bwResultsFileP2P, P2P); 
   PrintResults(bwResultsFileP2P, steps, rangeData, params);

   std::cout << "\nP2P Device Ranged Bandwidth Test Complete!" << std::endl;
}

void TestPCIeCongestion(BenchParams &params, SystemTopo &topo) {
   std::cout << "Running PCIe congestion tests..." << std::endl;


   std::cout << "PCIe congestion tests complete!" << std::endl;
}

void TestTaskScalability(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning task scalability tests..." << std::endl;


   std::cout << "\nTask scalability tests complete!" << std::endl;
}

void BurstHHBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData) { 
   long long blockSize = params.burstBlockSize;
   int numNodes = topo.NumNodes();
   int numSockets = params.nSockets;
   int numPatterns = 1;

   if (params.runPatternsHD)
      numPatterns = NUM_PATTERNS;

   burstData.resize(numPatterns * numSockets);
   double convConst =(double) blockSize / (double) pow(2.0, 30.0); 

   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
      
      for (int patternNum = 0; patternNum < numPatterns; patternNum ++) {
   
         MEM_PATTERN pattern = REPEATED;
         if (patternNum == 1)
            pattern = LINEAR_INC;
         if (patternNum == 2)
            pattern = LINEAR_DEC;
      
         for (int srcIdx = 0; srcIdx < numNodes; srcIdx++) { 

            for (int destIdx = 0; destIdx < numNodes; destIdx++) { 
               // HtoH Ranged Transfer - Pageable Memory
               int rowIdx = socketIdx * numPatterns + patternNum;
               burstData[rowIdx].push_back(convConst / BurstMemCopy(topo, blockSize, HOST_HOST_COPY, destIdx, srcIdx, params.numStepRepeats, pattern));        

               // HtoH Ranged Transfer - Pinned Memory Src Host
               burstData[rowIdx].push_back(convConst / BurstMemCopy(topo, blockSize, HOST_PINNED_HOST_COPY, destIdx, srcIdx, params.numStepRepeats, pattern));        
               
               if (params.usePinnedMem) {
                  // HtoH Ranged Transfer - Pinned Memory Dest
                  burstData[rowIdx].push_back(convConst / BurstMemCopy(topo, blockSize, HOST_HOST_PINNED_COPY, destIdx, srcIdx, params.numStepRepeats, pattern));        

                  // HtoH Ranged Transfer - Pinned Memory Both Hosts
                  burstData[rowIdx].push_back(convConst / BurstMemCopy(topo, blockSize, HOST_HOST_COPY_PINNED, destIdx, srcIdx, params.numStepRepeats, pattern));
               }       
            }
         }
      }
   }
}

void BurstHDBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData) { 
   long long blockSize = params.burstBlockSize;
   double convConst = (double) blockSize / (double) pow(2.0, 30.0); 

   int numSockets = params.nSockets;
   int numPatterns = 1;
   if (params.runPatternsHD)
      numPatterns = NUM_PATTERNS;
   
   burstData.resize(numPatterns * numSockets);
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
      
      for (int patternNum = 0; patternNum < numPatterns; patternNum++) {
      
         MEM_PATTERN pattern = REPEATED;
         if (patternNum == 1)
            pattern = LINEAR_INC;
         if (patternNum == 2)
            pattern = LINEAR_DEC;
    
         for (int srcIdx = 0; srcIdx < topo.NumNodes(); srcIdx++) { 

            //Host-Device Memory Transfers
            for (int destIdx = 0; destIdx < params.nDevices; destIdx++) {
               topo.SetActiveDevice(destIdx); 
               int rowIdx = socketIdx * numPatterns + patternNum; 

               // HtoD Ranged Transfer - Pageable Memory
               burstData[rowIdx].push_back( convConst / BurstMemCopy(topo, blockSize, HOST_DEVICE_COPY, destIdx, srcIdx, params.numStepRepeats, pattern));        
               
               // DtoH Ranged Transfer - Pageable Memory
               burstData[rowIdx].push_back( convConst / BurstMemCopy(topo, blockSize, DEVICE_HOST_COPY, srcIdx, destIdx, params.numStepRepeats, pattern));        
               
               if ( params.usePinnedMem) {      
                  // HtoD Ranged Transfer - Pinned Memory
                  burstData[rowIdx].push_back( convConst / BurstMemCopy(topo, blockSize, HOST_PINNED_DEVICE_COPY, destIdx, srcIdx, params.numStepRepeats, pattern));

                  // DtoH Ranged Transfer - Pinned Memory
                  burstData[rowIdx].push_back( convConst / BurstMemCopy(topo, blockSize, DEVICE_HOST_PINNED_COPY, srcIdx, destIdx, params.numStepRepeats, pattern)); 
               }
            }
         }
      }
   }
}

void BurstP2PBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData) { 
   long long blockSize = params.burstBlockSize;
   double convConst = (double) blockSize / (double) pow(2.0, 30.0); 
   
   burstData.resize(topo.NumGPUs() * params.nSockets);
   for (int socketIdx = 0; socketIdx < params.nSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumGPUs(); srcIdx++) { 
         //topo.SetActiveDevice(srcIdx); 
         for (int destIdx = 0; destIdx < topo.NumGPUs(); destIdx++) { 
            // DtoD Burst Transfer - No Peer, No UVA
            burstData[socketIdx * topo.NumGPUs() + srcIdx].push_back(convConst / BurstMemCopy(topo, blockSize, DEVICE_DEVICE_COPY, destIdx, srcIdx, params.numStepRepeats)); 
            // DtoD Burst Transfer - Peer, No UVA
            if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {
               topo.DeviceGroupSetP2P(srcIdx, destIdx, true);
               burstData[socketIdx * topo.NumGPUs() + srcIdx].push_back(convConst / BurstMemCopy(topo, blockSize, PEER_COPY_NO_UVA, destIdx, srcIdx, params.numStepRepeats)); 
               topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
            }

            if (topo.DeviceGroupUVA(srcIdx, destIdx)) {  
               // DtoD Burst Transfer - No Peer, UVA
               burstData[socketIdx * topo.NumGPUs() + srcIdx].push_back(convConst / BurstMemCopy(topo, blockSize, COPY_UVA, destIdx, srcIdx, params.numStepRepeats)); 
               
               // DtoD Burst Transfer - Peer, UVA
               if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {
                  topo.DeviceGroupSetP2P(srcIdx, destIdx, true);
                  burstData[socketIdx * topo.NumGPUs() + srcIdx].push_back( convConst / BurstMemCopy(topo, blockSize, COPY_UVA, destIdx, srcIdx, params.numStepRepeats));        
                  topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
               }
            }
         }
      }
   }
}

void RangeHHBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData) {
   int testNum = 0;
   long numRepeats = params.numStepRepeats;
   
   for (int socketIdx = 0; socketIdx < params.nSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumNodes(); srcIdx++) { 

         //Host To Host Memory Transfers
         for (int destIdx = 0; destIdx < topo.NumNodes(); destIdx++) { 
            // HtoH Ranged Transfer - Pageable Memory
            std::cout << "Test " << testNum++ << " HtoH, Pageable Memory, Repeated Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY, REPEATED, destIdx, srcIdx, numRepeats); 
            if (params.runPatternsHD) {
               std::cout << "Test " << testNum++ << " HtoH, Pageable Memory, Linear Inc Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
               std::cout << "Test " << testNum++ << " HtoH, Pageable Memory, Linear Dec Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats);
            }

            //HtoH Ranged Transfer - Pinned Memory Src Host
            std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Src, Repeated Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_HOST_COPY, REPEATED, destIdx, srcIdx, numRepeats);
            if (params.runPatternsHD){ 
               std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Src, Linear Inc Addr \tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_HOST_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
               std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Src, Linear Dec Addr \tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_HOST_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats); 
            }

            if (params.usePinnedMem) {
               //HtoH Ranged Transfer - Pinned Memory Dest Host
               std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Dest, Repeated Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_PINNED_COPY, REPEATED, destIdx, srcIdx, numRepeats); 
               if (params.runPatternsHD) {
                  std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Dest, Linear Inc Addr\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_PINNED_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
                  std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Dest, Linear Dec Addr\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_PINNED_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats); 
               }

              //HtoH Ranged Transfer - Pinned Memory Both Hosts
               std::cout << "Test " << testNum++ << " HtoH, Both Pinned Memory, Repeated Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, REPEATED, destIdx, srcIdx, numRepeats); 
               if (params.runPatternsHD) {
                  std::cout << "Test " << testNum++ << " HtoH, Both Pinned Memory, Linear Inc Addr\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, LINEAR_INC, destIdx, srcIdx, numRepeats); 
                  std::cout << "Test " << testNum++ << " HtoH, Both Pinned Memory, Linear Dec Addr\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, LINEAR_DEC, destIdx, srcIdx, numRepeats);
               } 
            }
         }
      }
   }
}

void RangeHDBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData) {
   int testNum = 0;
   long numRepeats = params.numStepRepeats;  
 
   for (int socketIdx = 0; socketIdx < params.nSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumNodes(); srcIdx++) { 

         //Host-Device PCIe Memory Transfers
         for (int destIdx = 0; destIdx < params.nDevices; destIdx++) {
             // HtoD Ranged Transfer - Pageable Memory
            std::cout << "Test " << testNum++ << " HtoD, Pageable Memory, Repeated Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, REPEATED, destIdx, srcIdx, numRepeats); 
            if (params.runPatternsHD) {
               std::cout << "Test " << testNum++ << " HtoD, Pageable Memory, Linear Inc Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
               std::cout << "Test " << testNum++ << " HtoD, Pageable Memory, Linear Dec Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats); 
            }

            // DtoH Ranged Transfer - Pageable Memory
            std::cout << "Test " << testNum++ << " DtoH, Pageable Memory, Repeated Addr\t\tCPU: " << socketIdx << "\t\tDev Src: " << srcIdx << "\tNUMA dest: " << srcIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, REPEATED, srcIdx, destIdx, numRepeats); 
            if (params.runPatternsHD) {
               std::cout << "Test " << testNum++ << " DtoH, Pageable Memory, Linear Inc Addr\t\tCPU: " << socketIdx << "\t\tDev Src: " << destIdx << "\tNUMA dest: " << srcIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, LINEAR_INC, srcIdx, destIdx, numRepeats); 
               std::cout << "Test " << testNum++ << " DtoH, Pageable Memory, Linear Dec Addr\t\tCPU: " << socketIdx << "\t\tDev Src: " << destIdx << "\tNUMA dest: " << srcIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, LINEAR_DEC, srcIdx, destIdx, numRepeats); 
            }
            
            if (params.usePinnedMem) {
               // HtoD Ranged Transfer - Pinned Memory
               std::cout << "Test " << testNum++ << " HtoD, Pinned Memory, Repeated Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_DEVICE_COPY, REPEATED, destIdx, srcIdx, numRepeats); 
               if (params.runPatternsHD) {
                  std::cout << "Test " << testNum++ << " HtoD, Pinned Memory, Linear Inc Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_DEVICE_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
                  std::cout << "Test " << testNum++ << " HtoD, Pinned Memory, Linear Dec Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_DEVICE_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats); 
               } 

               // DtoH Ranged Transfer - Pinned Memory
               std::cout << "Test " << testNum++ << " DtoH, Pinned Memory, Repeated Addr\t\tCPU: " << socketIdx << "\t\tSrc Dev: " << srcIdx << "\tNUMA Dest: " << srcIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_PINNED_COPY, REPEATED, srcIdx, destIdx, numRepeats); 
               if (params.runPatternsHD) {
                  std::cout << "Test " << testNum++ << " DtoH, Pinned Memory, Linear Inc Addr\t\tCPU: " << socketIdx << "\t\tDev Src: " << destIdx << "\tNUMA dest: " << srcIdx << std::endl;
                  MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_PINNED_COPY, LINEAR_INC, srcIdx, destIdx, numRepeats); 
                  std::cout << "Test " << testNum++ << " DtoH, Pinned Memory, Linear Dec Addr\t\tCPU: " << socketIdx << "\t\tDev Src: " << destIdx << "\tNUMA dest: " << srcIdx << std::endl;
                  MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_PINNED_COPY, LINEAR_DEC, srcIdx, destIdx, numRepeats);
               } 
            }               
         }
      }
   }
}

void RangeP2PBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData) {
   int testNum = 0;
   long numRepeats = params.numStepRepeats;  
   
   for (int socketIdx = 0; socketIdx < params.nSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumGPUs(); srcIdx++) { 

         for (int destIdx = 0; destIdx < topo.NumGPUs(); destIdx++) { 
            // DtoD Ranged Transfer - No Peer, No UVA
            std::cout << "Test " << testNum++ << " Device-To-Device, No Peer, No UVA\tCPU: " << socketIdx << "\tSrc Device: " << srcIdx << "\tDest Device: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_DEVICE_COPY, REPEATED, destIdx, srcIdx, numRepeats); 

            // DtoD Ranged Transfer - Peer, No UVA
            if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {
            std::cout << "Test " << testNum++ << " Device-To-Device, Peer Enabled, No UVA\tCPU: " << socketIdx << "\tSrc Device: " << srcIdx << "\tDest Device: " << destIdx << std::endl;
               topo.DeviceGroupSetP2P(srcIdx, destIdx, true);
               MemCopyRun(topo, blockSteps, bandwidthData, PEER_COPY_NO_UVA, REPEATED, destIdx, srcIdx, numRepeats);  
               topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
            }
            
            if (topo.DeviceGroupUVA(srcIdx, destIdx)) {  
               // DtoD Ranged Transfer - No Peer, UVA
               std::cout << "Test " << testNum++ << " Device-To-Device, No Peer, UVA\t\tCPU: " << socketIdx << "\tSrc Device: " << srcIdx << "\tDest Device: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, COPY_UVA, REPEATED, destIdx, srcIdx, numRepeats); 
 
               // DtoD Ranged Transfer - Peer, UVA
               if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {
                  std::cout << "Test " << testNum++ << " Device-To-Device, Peer Enabled, No UVA\tCPU: " << socketIdx << "\tSrc Device: " << srcIdx << "\tDest Device: " << destIdx << std::endl;
                  topo.DeviceGroupSetP2P(srcIdx, destIdx, true);
                  MemCopyRun(topo, blockSteps, bandwidthData, COPY_UVA, REPEATED, destIdx, srcIdx, numRepeats); 
                  topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
               }
            }
         }
      }
   }
}

void MemCopyRun(SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, MEM_OP copyType, MEM_PATTERN patternType, int destIdx, int srcIdx, int numCopiesPerStep) {
   char *destPtr, *srcPtr; 
   long totalSteps = blockSteps.size();
   
   std::vector<float> timedRun(totalSteps, 0.0);
   long long blockSize = blockSteps[totalSteps - 1 ];

   AllocateMemBlock(topo, (void **) &destPtr, (void **) &srcPtr, blockSize, copyType, destIdx, srcIdx);
   SetMemBlockTransfer(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx, -1);
   
   for (long stepNum = 0; stepNum < totalSteps; ++stepNum) { 

      bandwidthData[stepNum].push_back(TimedMemCopyStep((char *) destPtr, (char *) srcPtr, blockSteps[stepNum], blockSize, numCopiesPerStep, copyType, patternType, destIdx, srcIdx));

   }
   
   FreeMemBlock(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx);
}

float BurstMemCopy(SystemTopo &topo, long long blockSize, MEM_OP copyType, int destIdx, int srcIdx, int numSteps, MEM_PATTERN pattern) {  
   float elapsedTime = 0;
   char *destPtr, *srcPtr;

   AllocateMemBlock(topo, (void **) &destPtr, (void **) &srcPtr, blockSize, copyType, destIdx, srcIdx);
   SetMemBlockTransfer(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx, -1); 

   elapsedTime = TimedMemCopyStep((char *) destPtr, (char *) srcPtr, blockSize, blockSize, numSteps, copyType, pattern, destIdx, srcIdx);

   FreeMemBlock(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx);

   return elapsedTime;
}

float TimedMemCopyStep(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, MEM_PATTERN patternType, int destIdx, int srcIdx) {
   long long offset = 0;
   float totalTime = 0; 
   long long maxFrameSize = pow(2, 27);
   long long gap = maxFrameSize - stepSize;

   bool usingPattern = false;
   if (blockSize < maxFrameSize) {
      numCopiesPerStep *= 5;
      switch (patternType) {
         case LINEAR_INC:
            usingPattern = true;
            offset = 0;
            break;
         case LINEAR_DEC:
            usingPattern = true;
            offset = blockSize - stepSize;
            break;
         default:
            usingPattern = false;
            break;
      }
   }

   #ifdef USING_CPP
   std::chrono::high_resolution_clock::time_point start_c, stop_c;
   auto total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
   #else
   struct timeval stop_t, start_t, total_t;
   #endif
   
   hipEvent_t start_e, stop_e; 
   checkCudaErrors(hipEventCreate(&start_e));
   checkCudaErrors(hipEventCreate(&stop_e)); 

   if (copyType == HOST_HOST_COPY) {
      #ifdef USING_CPP
      start_c = std::chrono::high_resolution_clock::now();
      #else
      gettimeofday(&start_t, NULL);
      #endif
   } else{
      checkCudaErrors(hipEventRecord(start_e, 0));
   }

   for (int copyIdx = 0; copyIdx < numCopiesPerStep; copyIdx++) {

      MemCopyOp(destPtr + offset, srcPtr + offset, stepSize, blockSize, numCopiesPerStep, copyType, destIdx, srcIdx); 

      if (usingPattern) {
         switch (patternType) {
       
           case REPEATED:
               offset = 0;
               break;
            case LINEAR_INC:
               offset += gap;
               if (offset > blockSize)
                  offset = 0;
               break;
            case LINEAR_DEC:
               offset -= gap;
               if (offset < 0)
                  offset = blockSize - stepSize;
               break;
            default:
               offset = 0;
               std::cout << "Error: unrecognized memory access pattern during copy operation" << std::endl; 
               break;
         }
      }
   }

   if (copyType == HOST_HOST_COPY) {
      #ifdef USING_CPP
      stop_c = std::chrono::high_resolution_clock::now(); 
      total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
      totalTime = (float) total_c.count() * NANO_TO_MILLI; 
      #else
      gettimeofday(&stop_t, NULL); 
      timersub(&stop_t, &start_t, &total_t); 
      totalTime = (float) total_t.tv_usec * MICRO_TO_MILLI;
      #endif
   } else{
      checkCudaErrors(hipEventRecord(stop_e, 0));
      checkCudaErrors(hipEventSynchronize(stop_e));   
      checkCudaErrors(hipEventElapsedTime(&totalTime, start_e, stop_e));  
   }

   return totalTime / 1000.0 / (double) numCopiesPerStep;
}

void MemCopyOp(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, int destIdx, int srcIdx) {
   switch (copyType) {
      case HOST_HOST_COPY: 
         memcpy((void *) (destPtr), (void *) (srcPtr), stepSize);
         break;
      case HOST_PINNED_HOST_COPY: 
      case HOST_HOST_PINNED_COPY: 
      case HOST_HOST_COPY_PINNED: 
         checkCudaErrors(hipMemcpyAsync((void *)(destPtr), (void *) (srcPtr), stepSize, hipMemcpyHostToHost, 0));
         break;
      case DEVICE_HOST_COPY:
         checkCudaErrors(hipMemcpy((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDeviceToHost));
         break;
      case DEVICE_HOST_PINNED_COPY:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDeviceToHost, 0));
         break;
      case HOST_DEVICE_COPY:
         checkCudaErrors(hipMemcpy((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyHostToDevice));
         break;
      case HOST_PINNED_DEVICE_COPY:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyHostToDevice, 0));
         break;
      case PEER_COPY_NO_UVA:
         checkCudaErrors(hipMemcpyPeerAsync((void *) (destPtr), destIdx, (void *) (srcPtr), srcIdx, 0));
         break;
      case DEVICE_DEVICE_COPY:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDeviceToDevice));
         break;
      case COPY_UVA:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDefault, 0));
         break;
      default:
         std::cout << "Error: unrecognized timed memory copy operation type" << std::endl; 
         break;
   }
}

void FreeMemBlock(SystemTopo &topo, void* destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx, int srcIdx) {
   switch (copyType) {
      case HOST_HOST_COPY: 
         topo.FreeMem((void *) destPtr, numBytes);
         topo.FreeMem((void *) srcPtr, numBytes);
         break;
      case HOST_PINNED_HOST_COPY:  
         checkCudaErrors(hipHostUnregister((void*) srcPtr));
         topo.FreeMem((void *) srcPtr, numBytes);
         topo.FreeMem((void *) destPtr, numBytes);
         break;
     case HOST_HOST_PINNED_COPY:  
         topo.FreeMem((void *) srcPtr, numBytes);
         checkCudaErrors(hipHostUnregister((void*) destPtr));
         topo.FreeMem((void *) destPtr, numBytes);
         break;
     case HOST_HOST_COPY_PINNED:  
         checkCudaErrors(hipHostUnregister((void*) srcPtr));
         topo.FreeMem((void *) srcPtr, numBytes);
         checkCudaErrors(hipHostUnregister((void*) destPtr));
         topo.FreeMem((void *) destPtr, numBytes);
         break;
      case DEVICE_HOST_COPY:
         topo.FreeDeviceMem(srcPtr, srcIdx);
         topo.FreeMem((void *) destPtr, numBytes);
         break;
      case DEVICE_HOST_PINNED_COPY:
         topo.FreeDeviceMem(srcPtr, srcIdx);
         checkCudaErrors(hipHostUnregister((void*) destPtr));
         topo.FreeMem((void *) destPtr, numBytes);
         break;
      case HOST_DEVICE_COPY:
         topo.FreeMem((void *) srcPtr, numBytes);
         topo.FreeDeviceMem(destPtr, destIdx);
         break;
      case HOST_PINNED_DEVICE_COPY:
         checkCudaErrors(hipHostUnregister((void *) srcPtr));
         topo.FreeMem((void *) srcPtr, numBytes);
         topo.FreeDeviceMem(destPtr, destIdx);
         break;
      case PEER_COPY_NO_UVA: 
      case DEVICE_DEVICE_COPY:
      case COPY_UVA:
         topo.FreeDeviceMem(srcPtr, srcIdx);
         topo.FreeDeviceMem(destPtr, destIdx);
         break;
      default:
         std::cout << "Error: unrecognized memory copy operation type for deallocation!" << std::endl; 
         break;
   }
}

void SetMemBlockTransfer(SystemTopo &topo, void *destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx, int srcIdx, long long value) {
   switch (copyType) {
      case HOST_HOST_COPY: 
      case HOST_PINNED_HOST_COPY: 
      case HOST_HOST_PINNED_COPY: 
      case HOST_HOST_COPY_PINNED: 
         topo.SetHostMem(srcPtr, value, numBytes);
         topo.SetHostMem(destPtr, value, numBytes);
         break;
      case DEVICE_HOST_COPY:
      case DEVICE_HOST_PINNED_COPY:
         topo.SetDeviceMem(srcPtr, value, numBytes, srcIdx);
         topo.SetHostMem(destPtr, value, numBytes);
         break;
      case HOST_DEVICE_COPY:
      case HOST_PINNED_DEVICE_COPY:
         topo.SetHostMem(srcPtr, value, numBytes);
         topo.SetDeviceMem(destPtr, value, numBytes, destIdx);
         break;
      case PEER_COPY_NO_UVA: 
      case DEVICE_DEVICE_COPY:
      case COPY_UVA:
         topo.SetDeviceMem(srcPtr, value, numBytes, srcIdx);
         topo.SetDeviceMem(destPtr, value, numBytes, destIdx);
         break;
      default:
         std::cout << "Error: unrecognized memory copy operation type for mem set" << std::endl; 
         break;
   }
}

void AllocateMemBlock(SystemTopo &topo, void **destPtr, void **srcPtr,long  long numBytes, MEM_OP copyType, int destIdx, int srcIdx) {
   switch (copyType) {
      case HOST_HOST_COPY: 
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         *srcPtr =topo.AllocMemByNode(srcIdx, numBytes);
         break;
      case HOST_PINNED_HOST_COPY: 
         *srcPtr =topo.AllocMemByNode(srcIdx, numBytes);
         checkCudaErrors(hipHostRegister(*srcPtr, numBytes, hipHostRegisterPortable));
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         break;
      case HOST_HOST_PINNED_COPY: 
         *srcPtr =topo.AllocMemByNode(srcIdx, numBytes);
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         checkCudaErrors(hipHostRegister(*destPtr, numBytes, hipHostRegisterPortable));
         break;
      case HOST_HOST_COPY_PINNED: 
         *srcPtr = topo.AllocMemByNode(srcIdx, numBytes);
         checkCudaErrors(hipHostRegister(*srcPtr, numBytes, hipHostRegisterPortable));
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         checkCudaErrors(hipHostRegister(*destPtr, numBytes, hipHostRegisterPortable));
         break;
      case DEVICE_HOST_COPY:
         topo.AllocDeviceMem(srcPtr, numBytes, srcIdx);
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         break;
      case DEVICE_HOST_PINNED_COPY:
         topo.AllocDeviceMem(srcPtr, numBytes, srcIdx);
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         checkCudaErrors(hipHostRegister(*destPtr, numBytes, hipHostRegisterPortable));
         break;
      case HOST_DEVICE_COPY:
         *srcPtr = topo.AllocMemByNode(srcIdx, numBytes);
         topo.AllocDeviceMem(destPtr, numBytes, destIdx);
         break;
      case HOST_PINNED_DEVICE_COPY:
         *srcPtr = topo.AllocMemByNode(srcIdx, numBytes);
         checkCudaErrors(hipHostRegister(*srcPtr, numBytes, hipHostRegisterPortable));
         topo.AllocDeviceMem(destPtr, numBytes, destIdx);
         break;
      case PEER_COPY_NO_UVA: 
      case DEVICE_DEVICE_COPY:
      case COPY_UVA:
         topo.AllocDeviceMem(srcPtr, numBytes, srcIdx);
         topo.AllocDeviceMem(destPtr, numBytes, destIdx);
         break;
      default:
         std::cout << "Error: unrecognized memory copy operation type for allocation!" << std::endl; 
         break;
   }
}

float TimedMemOp(void **MemBlk, long long NumBytes, MEM_OP TimedOp) {
   #ifdef USING_CPP
   std::chrono::high_resolution_clock::time_point start_c, stop_c;
   auto total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
   #else
   struct timeval stop_t, start_t, total_t;
   #endif
   
   float OpTime = 0;
   
   #ifdef USING_CPP
   start_c = std::chrono::high_resolution_clock::now();
   #else
   gettimeofday(&start_t, NULL);
   #endif

   switch (TimedOp) {
      case HOST_MALLOC:
         *MemBlk = malloc(NumBytes); 
         break;
      case HOST_PINNED_MALLOC:
         checkCudaErrors(hipHostMalloc(MemBlk, NumBytes));
         break;
      case DEVICE_MALLOC:
         checkCudaErrors(hipMalloc(MemBlk, NumBytes));
         break;
      case HOST_FREE:
         free(*MemBlk);
         break;
      case HOST_PINNED_FREE:
         checkCudaErrors(hipHostFree(*MemBlk));
         break;
      case DEVICE_FREE:
         checkCudaErrors(hipFree(*MemBlk)); 
         break;
      default:
         std::cout << "Error: unrecognized timed memory operation type!" << std::endl; 
         break;
   }

   #ifdef USING_CPP
   stop_c = std::chrono::high_resolution_clock::now();
   total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);      
   OpTime = (float) total_c.count() * NANO_TO_MILLI;
   #else
   gettimeofday(&stop_t, NULL);
   timersub(&stop_t, &start_t, &total_t);
   OpTime = (float) total_t.tv_usec * MICRO_TO_MILLI;
   #endif

   return OpTime;
}
/*float TimedMemOp(void **MemBlk, long long NumBytes, MEM_OP TimedOp) {
   #ifdef USING_CPP
   std::chrono::high_resolution_clock::time_point start_c, stop_c;
   auto total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
   #else
   struct timeval stop_t, start_t, total_t;
   #endif
   
   hipEvent_t start_e, stop_e; 
   checkCudaErrors(hipEventCreate(&start_e));
   checkCudaErrors(hipEventCreate(&stop_e));
   float OpTime = 0;
  
   switch (TimedOp) {
      case HOST_MALLOC:
         #ifdef USING_CPP
         start_c = std::chrono::high_resolution_clock::now();
         *MemBlk = malloc(NumBytes);
         stop_c = std::chrono::high_resolution_clock::now();
         total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);      
         OpTime = (float) total_c.count() * NANO_TO_MILLI;
         #else
         gettimeofday(&start_t, NULL);
         *MemBlk = malloc(NumBytes); 
         gettimeofday(&stop_t, NULL);
         timersub(&stop_t, &start_t, &total_t);
         OpTime = (float) total_t.tv_usec * MICRO_TO_MILLI;
         #endif
         break;
      case HOST_PINNED_MALLOC:
         checkCudaErrors(hipEventRecord(start_e, 0));      
         checkCudaErrors(hipHostMalloc(MemBlk, NumBytes));
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e));
         break;
      case DEVICE_MALLOC:
         checkCudaErrors(hipEventRecord(start_e, 0));
         checkCudaErrors(hipMalloc(MemBlk, NumBytes));
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e)); 
         break;
      case HOST_FREE:
         #ifdef USING_CPP
         start_c = std::chrono::high_resolution_clock::now();
         free(*MemBlk);
         stop_c = std::chrono::high_resolution_clock::now(); 
         total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
         OpTime = (float) total_c.count() * NANO_TO_MILLI;
         #else
         gettimeofday(&start_t, NULL);
         free(*MemBlk); 
         gettimeofday(&stop_t, NULL); 
         timersub(&stop_t, &start_t, &total_t); 
         OpTime = (float) total_t.tv_usec * MICRO_TO_MILLI;
         #endif
         break;
      case HOST_PINNED_FREE:
         checkCudaErrors(hipEventRecord(start_e, 0));
         checkCudaErrors(hipHostFree(*MemBlk));
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e));
         break;
      case DEVICE_FREE:
         checkCudaErrors(hipEventRecord(start_e, 0));
         checkCudaErrors(hipFree(*MemBlk)); 
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));   
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e));  
         break;
      default:
         std::cout << "Error: unrecognized timed memory operation type!" << std::endl; 
         break;
   }
   checkCudaErrors(hipEventDestroy(start_e));
   checkCudaErrors(hipEventDestroy(stop_e));

   return OpTime;
}*/

int CalcRunSteps(std::vector<long long> &blockSteps, long long startStep, long long stopStep, long long numSteps) {
   int magStart = max((int)log10(startStep), 1);
   int magStop = log10(stopStep);

   long long start = pow(10, magStart);
   double stepSize = 10 * start / numSteps;
   int extra = (stopStep - pow(10, magStop)) / pow(10, magStop) * numSteps;
   long long stop = pow(10, magStop - 1) * (10 + extra); 
   int rangeSkip = numSteps / start;
   int totalSteps = (magStop - magStart) * (numSteps - rangeSkip) + extra + 1;  
   double step = start;

   for (long stepNum = 0; stepNum < totalSteps; ++stepNum) { 
      blockSteps.push_back(step);
      
      if ((stepNum) && (stepNum) % (numSteps - rangeSkip) == 0 && (stepSize * numSteps * 10) <= stop) {
         stepSize *= 10.0;
      } 
      
      step += stepSize; 
   }

   return totalSteps;
}

void PrintP2PBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData) {
   long long blockSize = params.burstBlockSize;
   int numSockets = params.nSockets;
   std::vector<int> deviceIdxs;
   deviceIdxs.resize(params.nDevices, 0);
   int dataIdx = 0;
   
   int matrixWidth = params.nDevices;
   int matrixHeight = params.nDevices * 4;
   std::cout << "\nDevice-To-Device Unidirectional Memory Transfers:" << std::endl;
   std::cout << "Transfer Block Size: " << blockSize / BYTES_TO_MEGA << " (MB)"<< std::endl;
  
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      std::cout << "\nInitiating Socket: " << socketIdx << std::endl;
      
      std::cout << "-----------------------------------------"; 
      for (int i = 0; i < matrixWidth; i++)
         std::cout << "----------------";
      std::cout << std::endl;

      std::cout << "|\t\t|-----------------------|"; 
      for (int i = 0; i < matrixWidth * 8 - 7; i++)
         std::cout << "-";

      std::cout << " Destination ";
      for (int i = 0; i < matrixWidth * 8 - 7; i++)
         std::cout << "-";
      std::cout << "|" << std::endl;
      
      std::cout << "|\t\t| GPU   | Transfer\t";
      for (int i = 0; i < matrixWidth; i++)
         std::cout << "|---------------";
      std::cout << "|" << std::endl;

      std::cout << "|\t\t|   #   | Type\t\t|";
      for (int i = 0; i < matrixWidth; i++)
         std::cout << "\t" << i << "\t|";
      std::cout << std::endl;

      std::cout << "|---------------|-----------------------"; 
      for (int i = 0; i < matrixWidth; i++)
         std::cout << "|---------------";
      std::cout << "|" << std::endl;


      std::cout << std::setprecision(2) << std::fixed;          
      
      std::fill(deviceIdxs.begin(), deviceIdxs.end(), 0);
      for (int i = 0; i < matrixHeight; ++i) {

         std::cout << "|\t\t|  " << i  / 4 <<  "\t|";
         if (i % 4 == 0) {
            std::cout << " Standard D2D\t|";
         } else if (i % 4 == 1) {
            std::cout << " Peer, No UVA\t|";
         } else if (i % 4 == 2) {
            std::cout << " No Peer, UVA\t|";
         } else { 
            std::cout << " Peer, UVA\t|";
         }
         
         if (i % 4 == 0) {
            //deviceIdxs.resize(matrixWidth, 0);
            //deviceIdxs.assign(deviceIdxs.begin(), deviceIdxs.end(), 0);
            std::fill(deviceIdxs.begin(), deviceIdxs.end(), 0);
         }
         dataIdx = 0;
         for (int j = 0; j < matrixWidth; ++j) {
            if (i % 4 == 0) {
               std::cout << "      " << burstData[socketIdx * matrixWidth + i / 4][dataIdx + deviceIdxs[j]] << "\t|";
               deviceIdxs[j]++;
            } else if ((i % 4 == 1) && topo.DeviceGroupCanP2P(i / 4, j)) {
               std::cout << "      " << burstData[socketIdx * matrixWidth + i / 4][dataIdx + deviceIdxs[j]] << "\t|";
               deviceIdxs[j]++;
            } else if ((i % 4 == 2) && topo.DeviceGroupUVA(i / 4, j)) {
               std::cout << "      " << burstData[socketIdx * matrixWidth + i / 4][dataIdx + deviceIdxs[j]] << "\t|";
               deviceIdxs[j]++;
            } else if ((i % 4 == 3) && topo.DeviceGroupUVA(i / 4, j) && topo.DeviceGroupCanP2P(i / 4, j)) { 
               std::cout << "      " << burstData[socketIdx * matrixWidth + i / 4][dataIdx + deviceIdxs[j]] << "\t|";
               deviceIdxs[j]++;
            } else { 
               std::cout << "\t-\t|";
            }

            dataIdx++;
            if (topo.DeviceGroupCanP2P(i / 4, j))
               dataIdx++;
            if (topo.DeviceGroupUVA(i / 4, j)) {
               dataIdx++;
               if (topo.DeviceGroupCanP2P(i / 4, j)) 
                  dataIdx++;
            }
         }
         
         std::cout << std::endl;
         
         if (i + 1 < matrixHeight && (i + 1 == ((float) matrixHeight / 2.0))) {
            std::cout << "|   Source\t|-----------------------";
            for (int i = 0; i < matrixWidth; i++)
               std::cout << "|---------------";
            std::cout << "|" << std::endl;
         } else if (i + 1 < matrixHeight && (i + 1) % 4  ==  0) {
            std::cout << "|\t\t|-----------------------";
            for (int i = 0; i < matrixWidth; i++)
               std::cout << "|---------------";
            std::cout << "|" << std::endl;
         }
      }
      std::cout << std::setprecision(4) << std::fixed;          
      
      std::cout << "-----------------------------------------"; 
      for (int i = 0; i < matrixWidth; i++)
         std::cout << "----------------";
      std::cout << std::endl;
   }
}

void PrintHDBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData) {
   long long blockSize = params.burstBlockSize;
   int numSockets = params.nSockets;
   
   int numPatterns = 1;
   if (params.runPatternsHD)
      numPatterns = NUM_PATTERNS;
   
   int matrixWidth = topo.NumNodes();
   int matrixHeight = params.nDevices;
   
   std::cout << "\nHost/Device Unidirectional Memory Transfers:" << std::endl;
   std::cout << "Transfer Block Size: " << blockSize / BYTES_TO_MEGA << " (MB)"<< std::endl;
   std::cout << "Num Patterns: " << numPatterns << std::endl;

   std::cout << std::setprecision(2) << std::fixed;          
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      std::cout << "\nInitiating Socket: " << socketIdx << std::endl;
      
      for (int patternNum = 0; patternNum < numPatterns; patternNum++) {
         std::cout << "Memory Access Pattern: " <<  PatternNames[patternNum] << std::endl;   
   
         std::cout << "-------------------------"; 
         for (int i = 0; i < matrixWidth * 2; i++)
            std::cout << "----------------";
         std::cout << std::endl;

         std::cout << "|\t\t\t|"; 
         for (int i = 0; i < matrixWidth * 16 - 6; i++)
            std::cout << "-";

         std::cout << " Host CPU ";
         for (int i = 0; i < matrixWidth * 16 - 5; i++)
            std::cout << "-";
         std::cout << "|" << std::endl;

         std::cout << "|\t\t\t|";
         for (int i = 0; i < matrixWidth; i++)
            std::cout << "\t\t" << i << "\t\t|";
         std::cout << std::endl;

         std::cout << "|\t\t\t|"; 
         for (int i = 0; i < matrixWidth * 2; i++){
            if (i + 1 < matrixWidth * 2)
               std::cout << "----------------";
            else 
               std::cout << "---------------";
         }
         std::cout << "|" << std::endl;
       
         std::cout << "|\t\t\t";
         for (int i = 0; i < matrixWidth; i++)
            std::cout << "| Host-2-Device | Device-2-Host ";
         std::cout << "|" << std::endl;
           
         std::cout << "|\t       Transfer\t|";
         for (int i = 0; i < matrixWidth * 2; i++){
            if (i + 1 < matrixWidth * 2)
               std::cout << "----------------";
            else 
               std::cout << "---------------";
         }
         std::cout << "|" << std::endl;

         std::cout << "|\t\t  Type\t";
         for (int i = 0; i < matrixWidth * 2; i++) {
            std::cout << "| Page\t";
            std::cout << "|  Pin\t";
         }
         std::cout << "|" << std::endl;

         std::cout << "|-----------------------"; 
         for (int i = 0; i < matrixWidth * 2; i++)
            std::cout << "----------------";
         std::cout << "|" << std::endl;
         
         std::cout << std::setprecision(2) << std::fixed;          
         for (int i = 0; i < matrixHeight; ++i) {

            std::cout << "|\t\t|  " << i <<  "\t|";
            int rowIdx = socketIdx * numPatterns + patternNum;
            for (int j = 0; j < matrixWidth; ++j) {
                  int colIdx = j * topo.NumGPUs() * 4 + i * 4;
                  std::cout << " " << burstData[rowIdx][colIdx + 0] << "\t|";
                  std::cout << " " << burstData[rowIdx][colIdx + 2] << "\t|";
                  std::cout << " " << burstData[rowIdx][colIdx + 1] << "\t|";
                  std::cout << " " << burstData[rowIdx][colIdx + 3] << "\t|";
            }
            std::cout << std::endl;
            
            if (i + 1 < matrixHeight && (i + 1 == ((float) matrixHeight / 2.0))) {
               std::cout << "|     Device\t|-------";
               for (int i = 0; i < matrixWidth * 2; i++)
                  std::cout << "----------------";
               std::cout << "|" << std::endl;
            } else if (i + 1 < matrixHeight) {
               std::cout << "|\t\t|-------";
               for (int i = 0; i < matrixWidth * 2; i++)
                  std::cout << "----------------";
               std::cout << "|" << std::endl;
            }
         }
         std::cout << std::setprecision(4) << std::fixed;          

         std::cout << "-------------------------"; 
         for (int i = 0; i < matrixWidth * 2; i++)
            std::cout << "----------------";
         std::cout << std::endl;
      }
   }
}

void PrintHHBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData) {
   long long blockSize = params.burstBlockSize;
   int numSockets = params.nSockets;

   int numPatterns = 1;
   if (params.runPatternsHH)
      numPatterns = NUM_PATTERNS;
   int nodeWidth = pow(HOST_MEM_TYPES * topo.NumNodes(), 2) / topo.NumNodes();

   int matrixWidth = HOST_MEM_TYPES * topo.NumNodes();
   int matrixHeight = HOST_MEM_TYPES * topo.NumNodes();
   
   std::cout << "\nHost-Host Multi-NUMA Unidirectional Memory Transfers:" << std::endl;
   std::cout << "Transfer Block Size: " << blockSize / BYTES_TO_MEGA << " (MB)"<< std::endl;
   std::cout << "Num Patterns: " << numPatterns << std::endl;

   std::cout << std::setprecision(2) << std::fixed;          
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      std::cout << "\nInitiating Socket: " << socketIdx << std::endl;
      
      for (int patternNum = 0; patternNum < numPatterns; patternNum++) {
         std::cout << "Memory Access Pattern: " <<  PatternNames[patternNum] << std::endl;   
         
         std::cout << "---------------------------------"; 
         for (int i = 0; i < matrixWidth; i++)
            std::cout << "----------------";
         std::cout << std::endl;

         std::cout << "|\t\t|----------------"; 
         for (int i = 0; i < matrixWidth * 8 - 7; i++)
            std::cout << "-";

         std::cout << " Destination ";
         for (int i = 0; i < matrixWidth * 8 - 7; i++)
            std::cout << "-";
         std::cout << "|" << std::endl;

         std::cout << "|   Transfer \t|---------------";// << std::endl;
         for (int i = 0; i < matrixWidth; i++)
            std::cout << "----------------";
         std::cout << "|" << std::endl;

         std::cout << "|   Point\t| NUMA \t\t|";
         for (int i = 0; i < topo.NumNodes(); i++)
            std::cout << "\t\t" << i << "\t\t|";
         std::cout << "" << std::endl;
         
         std::cout << "|\t\t| Node \t\t|";
         for (int i = 0; i < matrixWidth; i++) {
            if (i + 1 < matrixWidth)
               std::cout << "----------------";
            else 
               std::cout << "---------------";
         }
         std::cout << "|" << std::endl;
    
         std::cout << "|\t\t| #     Mem Type";
         for (int i = 0; i < matrixWidth; i++){
            if (i % 2)
               std::cout << "|    Pinned\t";
            else
               std::cout << "|    Pageable\t";
         }
         std::cout << "|"<< std::endl;
    
         std::cout << "|-------------------------------"; 
         for (int i = 0; i < matrixWidth; i++)
            std::cout << "----------------";
         std::cout << "|" << std::endl;
        
         for (int i = 0; i < matrixHeight; ++i) {
            std::cout << "|\t\t|\t|";//<< std::endl;
            for (int j = 0; j < matrixWidth + 1; ++j)
               std::cout << "\t|\t";
            std::cout << std::endl; 

            std::cout << "|\t\t| " << i / (matrixHeight / topo.NumNodes()) <<  "\t|";
            if (i % 2)
               std::cout << " Pin\t|    ";
            else
               std::cout << " Page\t|    ";
       
            int rowIdx = socketIdx * numPatterns + patternNum;
            for (int j = 0; j < matrixWidth; ++j) {
               int colIdx = (i / HOST_MEM_TYPES * nodeWidth) + j * HOST_MEM_TYPES + i % HOST_MEM_TYPES;
               std::cout << burstData[rowIdx][colIdx] << "\t|    ";
            }
                
            std::cout << "\n|\t\t|\t|";
            for (int j = 0; j < matrixWidth + 1; ++j)
               std::cout << "\t|\t";
            std::cout << std::endl;
            
            if (i + 1 < matrixHeight && (i + 1 != ((float) matrixHeight / 2.0))) {
               std::cout << "|\t\t|-------|-------|";
               for (int i = 0; i < matrixWidth; i++) {
                  if (i + 1 < matrixWidth)
                     std::cout << "----------------";
                  else 
                     std::cout << "---------------";
               }
               std::cout << "|" << std::endl; 
            } else if (i + 1 < matrixHeight) {
               std::cout << "|    Source     |-------|-------|";
               for (int i = 0; i < matrixWidth; i++) {
                  if (i + 1 < matrixWidth)
                     std::cout << "----------------";
                  else 
                     std::cout << "---------------";
               }
               std::cout << "|" << std::endl; 
            }
         }

         std::cout << "---------------------------------"; 
         for (int i = 0; i < matrixWidth; i++)
            std::cout << "----------------";
         std::cout << std::endl;
      }
      std::cout << std::setprecision(2) << std::fixed;          
   }
}

void PrintRangedHeader(BenchParams &params, SystemTopo &topo, std::ofstream &fileStream, BW_RANGED_TYPE testType) {

   std::vector<std::vector<int> > peerGroups;// = topo.GetPeerGroups();
   switch (testType) {
      case HH: 
         if (!params.runSocketTests) 
            fileStream << "0,";
         else
            fileStream << topo.NumSockets() << ",";

         fileStream << topo.NumNodes();
         if (params.usePinnedMem)
            fileStream << ",t";
         else 
           fileStream  << ",f";

         if (params.runSocketTests) 
            fileStream << ",t";
         else
            fileStream << ",f";

         fileStream << ",Repeated";
         if (params.runPatternsHD) {
            fileStream << ",Linear Inc";
            fileStream << ",Linear Dec";
         }
         fileStream << std::endl;
         break;
      case HD:
         if (!params.runSocketTests) 
            fileStream << "0,";
         else
            fileStream << topo.NumSockets() << ",";

         fileStream << topo.NumNodes() << ",";
         fileStream << params.nDevices;
         if (params.usePinnedMem)
            fileStream << ",t";
         else 
           fileStream  << ",f";

         if (params.runSocketTests) 
            fileStream << ",t";
         else
            fileStream << ",f";

         for (int i = 0; i < params.nDevices; i++) {
            fileStream << "," << topo.GetDeviceName(i);
         }

         fileStream << ",Repeated";
         if (params.runPatternsHD) {
            fileStream << ",Linear Inc";
            fileStream << ",Linear Dec";
         }

         fileStream << std::endl;
         break;
      case P2P:
         if (!params.runSocketTests) 
            fileStream << "0,";
         else
            fileStream << topo.NumSockets() << ",";

         fileStream << params.nDevices;
         fileStream << "," << topo.NumPeerGroups();
         
         if (params.runSocketTests) 
            fileStream << ",t";
         else
            fileStream << ",f";

         for (int i = 0; i < params.nDevices; i++) {
            fileStream << "," << topo.GetDeviceName(i);
         }
 
         for (int i = 0; i < params.nDevices; i++) {
            fileStream << "," << std::boolalpha << topo.DeviceUVA(i) << std::noboolalpha;
         }
         
         fileStream << std::endl;
         peerGroups = topo.GetPeerGroups();
         for (int i = 0; i < peerGroups.size(); i++) {
            for (int j = 0; j < peerGroups[i].size(); j++) {
               fileStream << peerGroups[i][j];
               if (j + 1 < peerGroups[i].size()) {
                  fileStream << ",";
               }
            }
            fileStream << std::endl;
         }
         break;
      default:
         std::cout << "Error: unrecognized ranged transfer test type!" << std::endl; 
         break;

   }
}

void PrintResults(std::ofstream &outFile, std::vector<long long> &steps, std::vector<std::vector<float> > &results, BenchParams &params) {
   
   if (!outFile.is_open()) {
      std::cout << "Failed to open file to print results" << std::endl;
      return;
   }
   std::vector<std::vector<float> >::iterator iter_o;
   std::vector<float>::iterator iter_i;
   std::vector<long long>::iterator iter_l = steps.begin();
   
   for (iter_o = results.begin(); iter_o != results.end(); ++iter_o) {
      outFile << std::fixed << *iter_l++ << ",";
      for (iter_i = (*iter_o).begin(); iter_i != (*iter_o).end(); ++iter_i) {
         outFile << std::fixed << *iter_i;
         if (iter_i + 1 != (*iter_o).end())
            outFile << ",";
      }
      outFile << std::endl;
   }
}


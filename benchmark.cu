#include "hip/hip_runtime.h"

// Benchmark includes and defines
#ifndef BENCH_HEADER_INC
#define BENCH_HEADER_INC
#include "benchmark.h"
#endif

// BenchParams class definition
#ifndef PARAM_CLASS_INC
#include "parameters.h"
#define PARAM_CLASS_INC
#endif

// SystemTopo class definition
#ifndef TOPOLOGY_CLASS_INC
#include "topology.h"
#define TOPOLOGY_CLASS_INC
#endif

// Benchmark Tests
void RunTestSuite(BenchParams &params, SystemTopo &topo);
void TestMemoryOverhead(BenchParams &params, SystemTopo &topo);
void TestHDBandwidth(BenchParams &params, SystemTopo &topo);
void TestP2PBandwidth(BenchParams &params, SystemTopo &topo);
void TestPCIeCongestion(BenchParams &params, SystemTopo &topo);
void TestTaskScalability(BenchParams &params, SystemTopo &topo);

// Test Subfunctions
void MemCopyRun(SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, MEM_OP copyType, MEM_PATTERN patternType, int destIdx, int srcIdx, int numCopiesPerStep); 
float TimedMemOp(void **MemBlk, long long NumBytes, MEM_OP TimedOp); 
float TimedMemCopyStep(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, MEM_PATTERN patternType, int destIdx = 0, int srcIdx = 0);
float BurstMemCopy(SystemTopo &topo, long long blockSize, MEM_OP copyType, int destIdx, int srcIdx, int numSteps);
void MemCopyOp(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, int destIdx = 0, int srcIdx = 0);

void TestRangeHDBandwidth(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, bool testSockets, int &testNum); 
void TestRangeHHBandwidth(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, bool testSockets, int &testNum); 
void TestRangeP2PBandwidth(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, bool testSockets, int &testNum);

void TestBurstHDBandwidth(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData, bool testSockets, int &testNum); 
void TestBurstHHBandwidth(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData, bool testSockets, int &testNum); 
void TestBurstP2PBandwidth(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData, bool testSockets, int &testNum);  

// Support functions
void AllocateMemBlock(SystemTopo &topo, void **destPtr, void **srcPtr, long long numBytes, MEM_OP copyType, int destIdx = 0, int srcIdx = 0);
void FreeMemBlock(SystemTopo &topo, void* destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx = 0, int srcIdx = 0);
int CalcRunSteps(std::vector<long long> &blockSteps, long long startStep, long long stopStep, long long numSteps);
void SetMemBlockTransfer(SystemTopo &topo, void *destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx, int srcIdx, long long value); 

// Results output
void PrintResults(std::ofstream &outFile, std::vector<long long> &steps, std::vector<std::vector<float> > &results, BenchParams &params);
void PrintHHBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData, bool testSockets);
void PrintHDBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData, bool testSockets);
void PrintP2PBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData, bool testSockets);
 
/* Benchmark main()
 * 
 * 
 */
int main (int argc, char **argv) {
   BenchParams benchParams;  
   SystemTopo sysTopo;
   
   std::cout << "\nStarting Multi-GPU Performance Test Suite...\n" << std::endl; 
   
   // Determine the number of recognized CUDA enabled devices
   checkCudaErrors(hipGetDeviceCount(&(benchParams.nDevices)));

   // Exit if system contains no devices
   if (benchParams.nDevices <= 0) {
      std::cout << "No devices found...aborting benchmarks." << std::endl;
      exit(-1);
   }

   // Setup benchmark parameters
   if (argc == 1) { 
      // No input file, use default parameters
      benchParams.SetDefault();
   
   } else if (argc == 2) {       
      // Parse input file and set parameter class local variables
      benchParams.ParseParamFile(std::string(argv[1]));

   } else {
      // Unknown input parameter list, abort test
      std::cout << "Aborting test: Incorrect number of input parameters" << std::endl;
      exit(-1);
   }

   // Print HWLOC topology info
   // Class constructor parses system topology from device files (linux)
   std::string topoFileName ="./results/topology.out";
   std::ofstream topoFile(topoFileName.c_str());

   // Output system topology to file and screen
   sysTopo.PrintTopology(topoFile);

   // Output device properties for each CUDA enabled GPU to file
   sysTopo.PrintDeviceProps(benchParams);

   // Print actual benchmark parameters for user/script parsing
   benchParams.PrintParams();

   // Run the benchmark per parameters defines in params
   RunTestSuite(benchParams, sysTopo);
  
   return 0;
}

void RunTestSuite(BenchParams &params, SystemTopo &topo) {

   if (params.runMemoryOverheadTest) {
      
      TestMemoryOverhead(params, topo);
   
   }

   if (params.runHDBandwidthTest) {

      TestHDBandwidth(params, topo);

   }

   if (params.runP2PBandwidthTest) {  
      
      TestP2PBandwidth(params, topo);
   
   }

   if (params.runPCIeCongestionTest) {

      TestPCIeCongestion(params, topo);

   }

   if (params.runTaskScalabilityTest) { 

      TestTaskScalability(params, topo);

   }

   std::cout << "\nBenchmarks complete!\n" << std::endl;

}

void TestMemoryOverhead(BenchParams &params, SystemTopo &topo) {
   char *deviceMem = NULL;
   char *hostMem = NULL;
   char *hostPinnedMem = NULL;
   int nDevices = params.nDevices;
   long long chunkSize = 0;
   int testNum = 0;

   std::cout << "\nRunning Memory Overhead Test...\n" << std::endl;

   // Only run overhead device cases on a single device
   // default to device 0
   if (!params.runAllDevices)
      nDevices = 1;

   std::vector<long long> blockSteps;
   CalcRunSteps(blockSteps, params.rangeMemOverhead[0], params.rangeMemOverhead[1], params.rangeMemOverhead[2]);  
   std::vector<std::vector<float> > overheadData;
   overheadData.resize(blockSteps.size());
   
   // Memory overhead test will run for each device utilizing the hipMalloc and hipFree functions
   // on the first iteration of the look, assuming there is atleast one device, the host will run the 
   // pinned and un-pinned memory tests
   for (int numaIdx = 0; numaIdx < topo.NumNodes(); numaIdx++) { 
      topo.PinNumaNode(numaIdx);
      
      for (int socketIdx = 0; socketIdx < topo.NumSockets(); socketIdx++) {
         topo.PinSocket(socketIdx);
         
            std::cout << "Test " << testNum++ << " Host Alloc/Free, Pinned/Pageable\t" << "NUMA node: " << numaIdx << " CPU " << socketIdx << std::endl;            
         // Host based management for CASE 1 & 2
         for (long stepIdx = 0; stepIdx < blockSteps.size(); stepIdx++) {
            chunkSize = blockSteps[stepIdx];
            float pinAllocTime = 0, pinFreeTime = 0, hostAllocTime = 0, hostFreeTime = 0;
            
            // repeat same block run and average times
            for (int reIdx = 0; reIdx < params.numStepRepeatsOH; reIdx++) {
               if (params.usePinnedMem) {
                  //CASE 1: Host Pinned Memory Overhead
                  pinAllocTime += TimedMemOp((void **) &hostPinnedMem, chunkSize, HOST_PINNED_MALLOC);
                  pinFreeTime += TimedMemOp((void **) &hostPinnedMem, 0, HOST_PINNED_FREE); 
               }
               //CASE 2: Host UnPinned Memory Overhead
               hostAllocTime += TimedMemOp((void **) &hostMem, 0, HOST_FREE);
               hostFreeTime += TimedMemOp((void **) &hostMem, chunkSize, HOST_MALLOC);
            }

            overheadData[stepIdx].push_back(pinAllocTime / (float) params.numStepRepeatsOH);
            overheadData[stepIdx].push_back(pinFreeTime / (float) params.numStepRepeatsOH);
            overheadData[stepIdx].push_back(hostAllocTime / (float) params.numStepRepeatsOH);
            overheadData[stepIdx].push_back(hostFreeTime / (float) params.numStepRepeatsOH);
         }
         
         // Device based memory management for CASE 3 & 4
         for (int currDev = 0; currDev < nDevices; currDev++) {
            checkCudaErrors(hipSetDevice(currDev)); 
            std::cout << "Test " << testNum++ << " Device Alloc/Free \t\t" << "NUMA node: " << numaIdx << " CPU " << socketIdx << " Dev:" << currDev << std::endl;            
            
            for (long stepIdx = 0; stepIdx < blockSteps.size(); stepIdx++) {
               chunkSize = blockSteps[stepIdx];
               float devAllocTime = 0, devFreeTime = 0;

               // repeat same block run and average times
               for (int reIdx = 0; reIdx < params.numStepRepeatsOH; reIdx++) {
                  // CASE 3: Allocation of device memory  
                  devAllocTime += TimedMemOp((void **) &deviceMem, chunkSize, DEVICE_MALLOC);
                  // CASE 4: DeAllocation of device memory 
                  devFreeTime += TimedMemOp((void **) &deviceMem, 0, DEVICE_FREE);
               }

               overheadData[stepIdx].push_back(devAllocTime / (float) params.numStepRepeatsOH);
               overheadData[stepIdx].push_back(devFreeTime / (float) params.numStepRepeatsOH);
            }
         }
      }
   }
   std::string dataFileName = "./results/" + params.resultsFile + "_overhead.csv";
   std::ofstream overheadResultsFile(dataFileName.c_str());
   PrintResults(overheadResultsFile, blockSteps, overheadData, params);

   std::cout << "\nMemory Overhead Test Complete!" << std::endl;
   
}

void TestHDBandwidth(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Host-Device and Device-Host Bandwidth Tests..." << std::endl;

   int testNum = 0;

   bool testSockets = false;  

   if (params.runSustainedHD == false)
      params.numCopiesPerStepHD = 1;

   if (params.runBurstHD) {
      std::vector<std::vector<float> > burstData;
      std::cout << "\nRunning Burst Bandwidth test...\n" << std::endl;

      TestBurstHHBandwidth(params, topo, burstData, testSockets, testNum); 
      PrintHHBurstMatrix(params, topo, burstData, testSockets);
      burstData.clear();
      TestBurstHDBandwidth(params, topo, burstData, testSockets, testNum);  
      PrintHDBurstMatrix(params, topo, burstData, testSockets);
      
      std::cout << "\nFinished Burst Bandwidth test!" << std::endl;
   }

   if (params.runRangeTestHD) {
      std::vector<std::vector<float> > rangeData;
      std::vector<long long> blockSteps;
      
      std::cout << "\nRunning Ranged Bandwidth test...\n" << std::endl;

      CalcRunSteps(blockSteps, params.rangeHostDeviceBW[0], params.rangeHostDeviceBW[1], params.rangeHostDeviceBW[2]); 
      rangeData.resize(blockSteps.size());
      TestRangeHHBandwidth(params, topo, blockSteps, rangeData, testSockets, testNum);
      TestRangeHDBandwidth(params, topo, blockSteps, rangeData, testSockets, testNum);

      // tt == Transfer Time
      std::string dataFileName = "./results/" + params.resultsFile + "_ranged_hd_tt.csv";
      std::ofstream ttResultsFile(dataFileName.c_str());
      PrintResults(ttResultsFile, blockSteps, rangeData, params);

      // Output throughput (GB/S) and block size
      for (int blkIdx = 0; blkIdx < blockSteps.size(); ++blkIdx) {
         for (int runIdx = 0; runIdx < rangeData[blkIdx].size(); ++runIdx) {
            rangeData[blkIdx][runIdx] = ((double) blockSteps[blkIdx]) / rangeData[blkIdx][runIdx];
            rangeData[blkIdx][runIdx] /= pow(2.0, 30.0);
            rangeData[blkIdx][runIdx] *= 10e3f;
         }
      }

      dataFileName = "./results/" + params.resultsFile + "_ranged_hd_bw.csv";
      std::ofstream bwResultsFile(dataFileName.c_str());
      PrintResults(bwResultsFile, blockSteps, rangeData, params);
      
      std::cout << "\nRanged Bandwidth Test Complete!" << std::endl;
   }

   std::cout << "\nHost-Device and Host-Host Bandwidth Tests complete!" << std::endl;
}

void TestP2PBandwidth(BenchParams &params, SystemTopo &topo){

   std::cout << "Running P2P Device Bandwidth Test..." << std::endl;

   std::vector<std::vector<float> > rangeData;
   std::vector<std::vector<float> > burstData;
   std::vector<long long> blockSteps;
   int testNum = 0;

   bool testSockets = false;  

   if (params.runSustainedP2P == false)
      params.numCopiesPerStepP2P = 1;

   if (params.runBurstP2P) {
      std::cout << "\nRunning Peer Burst Bandwidth test...\n" << std::endl;
      TestBurstP2PBandwidth(params, topo, burstData, testSockets, testNum); 
      
      PrintP2PBurstMatrix(params, topo, burstData, testSockets);
      std::cout << "\nFinished Peer Burst Bandwidth test!" << std::endl;
   }

   if (params.runRangeTestP2P) {
      std::cout << "\nRunning Peer Ranged Bandwidth test...\n" << std::endl;

      CalcRunSteps(blockSteps, params.rangeDeviceP2P[0], params.rangeDeviceP2P[1], params.rangeDeviceP2P[2]); 
      rangeData.resize(blockSteps.size());
      TestRangeP2PBandwidth(params, topo, blockSteps, rangeData, testSockets, testNum);

      // tt == Transfer Time
      std::string dataFileName = "./results/" + params.resultsFile + "_ranged_p2p_tt.csv";
      std::ofstream ttResultsFile(dataFileName.c_str());
      PrintResults(ttResultsFile, blockSteps, rangeData, params);

      // Output throughput (GB/S) and block size
      for (int blkIdx = 0; blkIdx < blockSteps.size(); ++blkIdx) {
         for (int runIdx = 0; runIdx < rangeData[blkIdx].size(); ++runIdx) {
            rangeData[blkIdx][runIdx] = ((double) blockSteps[blkIdx]) / rangeData[blkIdx][runIdx];
            rangeData[blkIdx][runIdx] /= pow(2.0, 30.0);
            rangeData[blkIdx][runIdx] *= 10e3f;
         }
      }

      dataFileName = "./results/" + params.resultsFile + "_ranged_p2p_bw.csv";
      std::ofstream p2pResultsFile(dataFileName.c_str());
      PrintResults(p2pResultsFile, blockSteps, rangeData, params);
      
      std::cout << "\nRanged Peer Device Bandwidth Test Complete!" << std::endl;
   }

   std::cout << "\nP2P Device Bandwidth Test Complete!" << std::endl;
}

void TestPCIeCongestion(BenchParams &params, SystemTopo &topo) {
   std::cout << "Running PCIe congestion test" << std::endl;
   return;
}

void TestTaskScalability(BenchParams &params, SystemTopo &topo) {
   std::cout << "Running task scalability test" << std::endl;
   return;
}

void TestRangeP2PBandwidth(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, bool testSockets, int &testNum) {
   
   int numSockets = 1;
   if (testSockets)
      numSockets = topo.NumSockets();

   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumGPUs(); srcIdx++) { 

         for (int destIdx = 0; destIdx < topo.NumGPUs(); destIdx++) { 
            // DtoD Ranged Transfer - No Peer, No UVA
            std::cout << "Test " << testNum++ << " Device-To-Device, No Peer, No UVA\tCPU: " << socketIdx << "\tSrc Device: " << srcIdx << "\tDest Device: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_DEVICE_COPY, REPEATED, destIdx, srcIdx, params.numCopiesPerStepHD); 

            // DtoD Ranged Transfer - Peer, No UVA
            if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {
            std::cout << "Test " << testNum++ << " Device-To-Device, Peer Enabled, No UVA\tCPU: " << socketIdx << "\tSrc Device: " << srcIdx << "\tDest Device: " << destIdx << std::endl;
               topo.DeviceGroupSetP2P(srcIdx, destIdx, true);
               MemCopyRun(topo, blockSteps, bandwidthData, PEER_COPY_NO_UVA, REPEATED, destIdx, srcIdx, params.numCopiesPerStepHD);  
               topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
            }
            
            if (topo.DeviceGroupUVA(srcIdx, destIdx)) {  
               // DtoD Ranged Transfer - No Peer, UVA
               std::cout << "Test " << testNum++ << " Device-To-Device, No Peer, UVA\t\tCPU: " << socketIdx << "\tSrc Device: " << srcIdx << "\tDest Device: " << destIdx << std::endl;
               //topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
               MemCopyRun(topo, blockSteps, bandwidthData, COPY_UVA, REPEATED, destIdx, srcIdx, params.numCopiesPerStepHD); 
 
               // DtoD Ranged Transfer - Peer, UVA
               if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {
                  std::cout << "Test " << testNum++ << " Device-To-Device, Peer Enabled, No UVA\tCPU: " << socketIdx << "\tSrc Device: " << srcIdx << "\tDest Device: " << destIdx << std::endl;
                  topo.DeviceGroupSetP2P(srcIdx, destIdx, true);
                  MemCopyRun(topo, blockSteps, bandwidthData, COPY_UVA, REPEATED, destIdx, srcIdx, params.numCopiesPerStepHD); 
                  topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
               }
            }
         }
      }
   }
}

void TestBurstP2PBandwidth(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData, bool testSockets, int &testNum) { 
   long long blockSize = pow(2, 26); //set test size for 16 MB

   double convConst =(double) blockSize * 1e3f / (double) pow(2.0, 30.0); 
   //(double) blockSize * (double) params.numCopiesPerStepHD * 1000 / (double) pow(2.0, 30.0);
   int numSockets = 1;
   if (testSockets)
      numSockets = topo.NumSockets();

   burstData.resize(topo.NumGPUs());
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumGPUs(); srcIdx++) { 

         for (int destIdx = 0; destIdx < topo.NumGPUs(); destIdx++) { 
            // DtoD Burst Transfer - No Peer, No UVA
            burstData[srcIdx].push_back(convConst / BurstMemCopy(topo, blockSize, DEVICE_DEVICE_COPY, destIdx, srcIdx, params.numCopiesPerStepP2P)); 

            // DtoD Burst Transfer - Peer, No UVA
            if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {
               topo.DeviceGroupSetP2P(srcIdx, destIdx, true);
               burstData[srcIdx].push_back(convConst / BurstMemCopy(topo, blockSize, PEER_COPY_NO_UVA, destIdx, srcIdx, params.numCopiesPerStepHD)); 
               topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
            }

            if (topo.DeviceGroupUVA(srcIdx, destIdx)) {  
               // DtoD Burst Transfer - No Peer, UVA
               //topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
               burstData[srcIdx].push_back(convConst / BurstMemCopy(topo, blockSize, COPY_UVA, destIdx, srcIdx, params.numCopiesPerStepHD)); 
               
               // DtoD Burst Transfer - Peer, UVA
               if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {
                  topo.DeviceGroupSetP2P(srcIdx, destIdx, true);
                  burstData[srcIdx].push_back( convConst / BurstMemCopy(topo, blockSize, COPY_UVA, destIdx, srcIdx, params.numCopiesPerStepHD));        
                  topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
               }
            }
         }
      }
   }
}

void TestBurstHDBandwidth(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData, bool testSockets, int &testNum) { 
   long long blockSize = pow(2, 26); //set test size for 16 MB
   double convConst =(double) blockSize * 1e3f / (double) pow(2.0, 30.0); //(double) blockSize * (double) params.numCopiesPerStepHD * 1000 / (double) pow(2.0, 30.0);

   int numSockets = 1;
   if (testSockets)
      numSockets = topo.NumSockets();

   //TODO: fix patterns
   int numPatterns = 1;
   if (false)
      numPatterns = NUM_PATTERNS;

   burstData.resize(topo.NumNodes());
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumNodes(); srcIdx++) { 

         //Host-Device Memory Transfers
         for (int destIdx = 0; destIdx < params.nDevices; destIdx++) {
            // HtoD Ranged Transfer - Pageable Memory
            burstData[srcIdx].push_back( convConst / BurstMemCopy(topo, blockSize, HOST_DEVICE_COPY, destIdx, srcIdx, params.numCopiesPerStepHD));        
            
            // DtoH Ranged Transfer - Pageable Memory
            burstData[srcIdx].push_back( convConst / BurstMemCopy(topo, blockSize, DEVICE_HOST_COPY, srcIdx, destIdx, params.numCopiesPerStepHD));        
            
            // HtoD Ranged Transfer - Pinned Memory
            burstData[srcIdx].push_back( convConst / BurstMemCopy(topo, blockSize, HOST_PINNED_DEVICE_COPY, destIdx, srcIdx, params.numCopiesPerStepHD));

            // DtoH Ranged Transfer - Pinned Memory
            burstData[srcIdx].push_back( convConst / BurstMemCopy(topo, blockSize, DEVICE_HOST_PINNED_COPY, srcIdx, destIdx, params.numCopiesPerStepHD)); 
         }
      }
   }
}



void TestBurstHHBandwidth(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData, bool testSockets, int &testNum) { 
   long long blockSize = pow(2, 26); //set test size for 16 MB

   int numSockets = 1;
   if (testSockets)
      numSockets = topo.NumSockets();

   //TODO: fix patterns
   int numPatterns = 1;
   if (false)
      numPatterns = NUM_PATTERNS;

   int stride = 2;
   int matrixWidth = 2 * topo.NumNodes();
   int matrixHeight = numPatterns * matrixWidth;
   burstData.resize(matrixHeight);

   for (int idx = 0; idx < matrixHeight; ++idx) {
      burstData[idx].resize(matrixWidth);
   }
   
   double convConst =(double) blockSize * 1e3f / (double) pow(2.0, 30.0); //(double) blockSize * (double) params.numCopiesPerStepHD * 1000 / (double) pow(2.0, 30.0);  
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumNodes(); srcIdx++) { 

         //Host-To-Host Memory Transfers
         for (int destIdx = 0; destIdx < topo.NumNodes(); destIdx++) { 
            // HtoH Ranged Transfer - Pageable Memory
            burstData[srcIdx * stride][destIdx * stride] = convConst / BurstMemCopy(topo, blockSize, HOST_HOST_COPY, destIdx, srcIdx, params.numCopiesPerStepHD); 

            // HtoH Ranged Transfer - Pinned Memory Src Host
            burstData[srcIdx * stride + 1][destIdx * stride] = convConst / BurstMemCopy(topo, blockSize, HOST_PINNED_HOST_COPY, destIdx, srcIdx, params.numCopiesPerStepHD); 
            
            // HtoH Ranged Transfer - Pinned Memory Dest
            burstData[srcIdx * stride][destIdx * stride + 1] = convConst / BurstMemCopy(topo, blockSize, HOST_HOST_PINNED_COPY, destIdx, srcIdx, params.numCopiesPerStepHD); 

            // HtoH Ranged Transfer - Pinned Memory Both Hosts
            burstData[srcIdx * stride + 1][destIdx * stride + 1] = convConst / BurstMemCopy(topo, blockSize, HOST_HOST_COPY_PINNED, destIdx, srcIdx, params.numCopiesPerStepHD);        
         }
      }
   }
}

void TestRangeHHBandwidth(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, bool testSockets, int &testNum) {
   int numSockets = 1;
   
   if (testSockets)
      numSockets = topo.NumSockets();
   
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumNodes(); srcIdx++) { 

         //Host To Host Memory Transfers
         for (int destIdx = 0; destIdx < topo.NumNodes(); destIdx++) { 
            // HtoH Ranged Transfer - Pageable Memory
            std::cout << "Test " << testNum++ << " HtoH, Pageable Memory, Repeated Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY, REPEATED, destIdx, srcIdx, params.numCopiesPerStepHD); 
            if (params.runAllPatternsHD) {
               std::cout << "Test " << testNum++ << " HtoH, Pageable Memory, Random\t\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY, RANDOM, destIdx, srcIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " HtoH, Pageable Memory, Linear Inc Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY, LINEAR_INC, destIdx, srcIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " HtoH, Pageable Memory, Linear Dec Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY, LINEAR_DEC, destIdx, srcIdx, params.numCopiesPerStepHD);
            }

            //HtoH Ranged Transfer - Pinned Memory Src Host
            std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Src, Repeated Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_HOST_COPY, REPEATED, destIdx, srcIdx, params.numCopiesPerStepHD);
            if (params.runAllPatternsHD){ 
               std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Src, Random Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_HOST_COPY, RANDOM, destIdx, srcIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Src, Linear Inc Addr \tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_HOST_COPY, LINEAR_INC, destIdx, srcIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Src, Linear Dec Addr \tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_HOST_COPY, LINEAR_DEC, destIdx, srcIdx, params.numCopiesPerStepHD); 
            }

            //HtoH Ranged Transfer - Pinned Memory Dest Host
            std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Dest, Repeated Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_PINNED_COPY, REPEATED, destIdx, srcIdx, params.numCopiesPerStepHD); 
            if (params.runAllPatternsHD) {
               std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Dest, Random Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_PINNED_COPY, RANDOM, destIdx, srcIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Dest, Linear Inc Addr\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_PINNED_COPY, LINEAR_INC, destIdx, srcIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Dest, Linear Dec Addr\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_PINNED_COPY, LINEAR_DEC, destIdx, srcIdx, params.numCopiesPerStepHD); 
            }

           //HtoH Ranged Transfer - Pinned Memory Both Hosts
            std::cout << "Test " << testNum++ << " HtoH, Both Pinned Memory, Repeated Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, REPEATED, destIdx, srcIdx, params.numCopiesPerStepHD); 
            if (params.runAllPatternsHD) {
               std::cout << "Test " << testNum++ << " HtoH, Both Pinned Memory, Random Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx  << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, RANDOM, destIdx, srcIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " HtoH, Both Pinned Memory, Linear Inc Addr\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, LINEAR_INC, destIdx, srcIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " HtoH, Both Pinned Memory, Linear Dec Addr\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, LINEAR_DEC, destIdx, srcIdx, params.numCopiesPerStepHD); 
            }
         }
      }
   }
}

void TestRangeHDBandwidth(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, bool testSockets, int &testNum) {
   int numSockets = 1;
   
   if (testSockets)
      numSockets = topo.NumSockets();
   
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumNodes(); srcIdx++) { 

         //Host-Device PCIe Memory Transfers
         for (int destIdx = 0; destIdx < params.nDevices; destIdx++) {
             // HtoD Ranged Transfer - Pageable Memory
            std::cout << "Test " << testNum++ << " HtoD, Pageable Memory, Repeated Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, REPEATED, destIdx, srcIdx, params.numCopiesPerStepHD); 
            if (params.runAllPatternsHD) {
               std::cout << "Test " << testNum++ << " HtoD, Pageable Memory, Random Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, RANDOM, destIdx, srcIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " HtoD, Pageable Memory, Linear Inc Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, LINEAR_INC, destIdx, srcIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " HtoD, Pageable Memory, Linear Dec Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, LINEAR_DEC, destIdx, srcIdx, params.numCopiesPerStepHD); 
            }

            // DtoH Ranged Transfer - Pageable Memory
            std::cout << "Test " << testNum++ << " DtoH, Pageable Memory, Repeated Addr\t\tCPU: " << socketIdx << "\t\tDev Src: " << srcIdx << "\tNUMA dest: " << srcIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, REPEATED, srcIdx, destIdx, params.numCopiesPerStepHD); 
            if (params.runAllPatternsHD) {
               std::cout << "Test " << testNum++ << " DtoH, Pageable Memory, Random Addr\t\tCPU: " << socketIdx << "\t\tDev Src: " << destIdx << "\tNUMA dest: " << srcIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, RANDOM, srcIdx, destIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " DtoH, Pageable Memory, Linear Inc Addr\t\tCPU: " << socketIdx << "\t\tDev Src: " << destIdx << "\tNUMA dest: " << srcIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, LINEAR_INC, srcIdx, destIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " DtoH, Pageable Memory, Linear Dec Addr\t\tCPU: " << socketIdx << "\t\tDev Src: " << destIdx << "\tNUMA dest: " << srcIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, LINEAR_DEC, srcIdx, destIdx, params.numCopiesPerStepHD); 
            }
            
            // HtoD Ranged Transfer - Pinned Memory
            std::cout << "Test " << testNum++ << " HtoD, Pinned Memory, Repeated Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_DEVICE_COPY, REPEATED, destIdx, srcIdx, params.numCopiesPerStepHD); 
            if (params.runAllPatternsHD) {
               std::cout << "Test " << testNum++ << " HtoD, Pinned Memory, Random Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_DEVICE_COPY, RANDOM, destIdx, srcIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " HtoD, Pinned Memory, Linear Inc Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_DEVICE_COPY, LINEAR_INC, destIdx, srcIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " HtoD, Pinned Memory, Linear Dec Addr\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_DEVICE_COPY, LINEAR_DEC, destIdx, srcIdx, params.numCopiesPerStepHD); 
            } 

            // DtoH Ranged Transfer - Pinned Memory
            std::cout << "Test " << testNum++ << " DtoH, Pinned Memory, Repeated Addr\t\tCPU: " << socketIdx << "\t\tSrc Dev: " << srcIdx << "\tNUMA Dest: " << srcIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_PINNED_COPY, REPEATED, srcIdx, destIdx, params.numCopiesPerStepHD); 
            if (params.runAllPatternsHD) {
               std::cout << "Test " << testNum++ << " DtoH, Pinned Memory, Random Addr\t\tCPU: " << socketIdx << "\t\tDev Src: " << destIdx << "\tNUMA dest: " << srcIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_PINNED_COPY, RANDOM, srcIdx, destIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " DtoH, Pinned Memory, Linear Inc Addr\t\tCPU: " << socketIdx << "\t\tDev Src: " << destIdx << "\tNUMA dest: " << srcIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_PINNED_COPY, LINEAR_INC, srcIdx, destIdx, params.numCopiesPerStepHD); 
               std::cout << "Test " << testNum++ << " DtoH, Pinned Memory, Linear Dec Addr\t\tCPU: " << socketIdx << "\t\tDev Src: " << destIdx << "\tNUMA dest: " << srcIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_PINNED_COPY, LINEAR_DEC, srcIdx, destIdx, params.numCopiesPerStepHD); 
            }               
         }
      }
   }
}

void MemCopyRun(SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, MEM_OP copyType, MEM_PATTERN patternType, int destIdx, int srcIdx, int numCopiesPerStep) {
   char *destPtr, *srcPtr; 
   long totalSteps = blockSteps.size();
   
   std::vector<float> timedRun(totalSteps, 0.0);
   long long blockSize = blockSteps[totalSteps - 1 ];

   AllocateMemBlock(topo, (void **) &destPtr, (void **) &srcPtr, blockSize, copyType, destIdx, srcIdx);
   SetMemBlockTransfer(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx, -1);
   
   for (long stepNum = 0; stepNum < totalSteps; ++stepNum) { 

      bandwidthData[stepNum].push_back(TimedMemCopyStep((char *) destPtr, (char *) srcPtr, blockSteps[stepNum], blockSize, numCopiesPerStep, copyType, patternType, destIdx, srcIdx));

   }
   
   FreeMemBlock(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx);
}

float BurstMemCopy(SystemTopo &topo, long long blockSize, MEM_OP copyType, int destIdx, int srcIdx, int numSteps) {
   float elapsedTime = 0;
   char *destPtr, *srcPtr;

   AllocateMemBlock(topo, (void **) &destPtr, (void **) &srcPtr, blockSize, copyType, destIdx, srcIdx);
   SetMemBlockTransfer(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx, -1); 

   elapsedTime = TimedMemCopyStep((char *) destPtr, (char *) srcPtr, blockSize, blockSize, numSteps, copyType, REPEATED, destIdx, srcIdx);

   FreeMemBlock(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx);

   return elapsedTime;
}

float TimedMemCopyStep(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, MEM_PATTERN patternType, int destIdx, int srcIdx) {
   long long offset = 0;
   float totalTime = 0; 

   #ifdef USING_CPP
   std::chrono::high_resolution_clock::time_point start_c, stop_c;
   auto total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
   #else
   struct timeval stop_t, start_t, total_t;
   #endif
   
   hipEvent_t start_e, stop_e; 
   checkCudaErrors(hipEventCreate(&start_e));
   checkCudaErrors(hipEventCreate(&stop_e)); 

   if (copyType == HOST_HOST_COPY) {
      #ifdef USING_CPP
      start_c = std::chrono::high_resolution_clock::now();
      #else
      gettimeofday(&start_t, NULL);
      #endif
   } else{
      checkCudaErrors(hipEventRecord(start_e, 0));
   }

   for (int copyIdx = 0; copyIdx < numCopiesPerStep; copyIdx++) {

      MemCopyOp(destPtr + offset, srcPtr + offset, stepSize, blockSize, numCopiesPerStep, copyType, destIdx, srcIdx); 

      //TODO: add options to change 
      if (numCopiesPerStep > 1 && MAX_PATTERN_SIZE) {
         switch (patternType) {
       
           case REPEATED:
               offset = 0;
               break;
            case RANDOM:
               break;
            case PERIODIC:
               break;
            case LINEAR_INC:
               break;
            case LINEAR_DEC:
               break;
            default:
               std::cout << "Error: unrecognized memory access pattern during copy operation" << std::endl; 
               break;
         }
      }
   }

   if (copyType == HOST_HOST_COPY) {
      #ifdef USING_CPP
      stop_c = std::chrono::high_resolution_clock::now(); 
      total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
      totalTime = (float) total_c.count() * NANO_TO_MILLI; 
      #else
      gettimeofday(&stop_t, NULL); 
      timersub(&stop_t, &start_t, &total_t); 
      totalTime = (float) total_t.tv_usec * MICRO_TO_MILLI;
      #endif
   } else{
      checkCudaErrors(hipEventRecord(stop_e, 0));
      checkCudaErrors(hipEventSynchronize(stop_e));   
      checkCudaErrors(hipEventElapsedTime(&totalTime, start_e, stop_e));  
   }

   return totalTime / (float) numCopiesPerStep;
}

void MemCopyOp(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, int destIdx, int srcIdx) {
   switch (copyType) {
      case HOST_HOST_COPY: 
         memcpy((void *) (destPtr), (void *) (srcPtr), stepSize);
         break;
      case HOST_PINNED_HOST_COPY: 
      case HOST_HOST_PINNED_COPY: 
      case HOST_HOST_COPY_PINNED: 
         checkCudaErrors(hipMemcpyAsync((void *)(destPtr), (void *) (srcPtr), stepSize, hipMemcpyHostToHost, 0));
         break;
      case DEVICE_HOST_COPY:
         checkCudaErrors(hipMemcpy((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDeviceToHost));
         break;
      case DEVICE_HOST_PINNED_COPY:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDeviceToHost, 0));
         break;
      case HOST_DEVICE_COPY:
         checkCudaErrors(hipMemcpy((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyHostToDevice));
         break;
      case HOST_PINNED_DEVICE_COPY:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyHostToDevice, 0));
         break;
      case PEER_COPY_NO_UVA:
         checkCudaErrors(hipMemcpyPeerAsync((void *) (destPtr), destIdx, (void *) (srcPtr), srcIdx, 0));
         break;
      case DEVICE_DEVICE_COPY:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDeviceToDevice));
         break;
      case COPY_UVA:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDefault, 0));
         break;
      default:
         std::cout << "Error: unrecognized timed memory copy operation type" << std::endl; 
         break;
   }
}

void FreeMemBlock(SystemTopo &topo, void* destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx, int srcIdx) {
   switch (copyType) {
      case HOST_HOST_COPY: 
         topo.FreeMem((void *) destPtr, numBytes);
         topo.FreeMem((void *) srcPtr, numBytes);
         break;
      case HOST_PINNED_HOST_COPY:  
         checkCudaErrors(hipHostUnregister((void*) srcPtr));
         topo.FreeMem((void *) srcPtr, numBytes);
         topo.FreeMem((void *) destPtr, numBytes);
         break;
     case HOST_HOST_PINNED_COPY:  
         topo.FreeMem((void *) srcPtr, numBytes);
         checkCudaErrors(hipHostUnregister((void*) destPtr));
         topo.FreeMem((void *) destPtr, numBytes);
         break;
     case HOST_HOST_COPY_PINNED:  
         checkCudaErrors(hipHostUnregister((void*) srcPtr));
         topo.FreeMem((void *) srcPtr, numBytes);
         checkCudaErrors(hipHostUnregister((void*) destPtr));
         topo.FreeMem((void *) destPtr, numBytes);
         break;
      case DEVICE_HOST_COPY:
         topo.FreeDeviceMem(srcPtr, srcIdx);
         topo.FreeMem((void *) destPtr, numBytes);
         break;
      case DEVICE_HOST_PINNED_COPY:
         topo.FreeDeviceMem(srcPtr, srcIdx);
         checkCudaErrors(hipHostUnregister((void*) destPtr));
         topo.FreeMem((void *) destPtr, numBytes);
         break;
      case HOST_DEVICE_COPY:
         topo.FreeMem((void *) srcPtr, numBytes);
         topo.FreeDeviceMem(destPtr, destIdx);
         break;
      case HOST_PINNED_DEVICE_COPY:
         checkCudaErrors(hipHostUnregister((void *) srcPtr));
         topo.FreeMem((void *) srcPtr, numBytes);
         topo.FreeDeviceMem(destPtr, destIdx);
         break;
      case PEER_COPY_NO_UVA: 
      case DEVICE_DEVICE_COPY:
      case COPY_UVA:
         topo.FreeDeviceMem(srcPtr, srcIdx);
         topo.FreeDeviceMem(destPtr, destIdx);
         break;
      default:
         std::cout << "Error: unrecognized memory copy operation type for deallocation" << std::endl; 
         break;
   }
}

void SetMemBlockTransfer(SystemTopo &topo, void *destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx, int srcIdx, long long value) {
   switch (copyType) {
      case HOST_HOST_COPY: 
      case HOST_PINNED_HOST_COPY: 
      case HOST_HOST_PINNED_COPY: 
      case HOST_HOST_COPY_PINNED: 
         topo.SetHostMem(srcPtr, value, numBytes);
         topo.SetHostMem(destPtr, value, numBytes);
         break;
      case DEVICE_HOST_COPY:
      case DEVICE_HOST_PINNED_COPY:
         topo.SetDeviceMem(srcPtr, value, numBytes, srcIdx);
         topo.SetHostMem(destPtr, value, numBytes);
         break;
      case HOST_DEVICE_COPY:
      case HOST_PINNED_DEVICE_COPY:
         topo.SetHostMem(srcPtr, value, numBytes);
         topo.SetDeviceMem(destPtr, value, numBytes, destIdx);
         break;
      case PEER_COPY_NO_UVA: 
      case DEVICE_DEVICE_COPY:
      case COPY_UVA:
         topo.SetDeviceMem(srcPtr, value, numBytes, srcIdx);
         topo.SetDeviceMem(destPtr, value, numBytes, destIdx);
         break;
      default:
         std::cout << "Error: unrecognized memory copy operation type for mem set" << std::endl; 
         break;
   }
}

void AllocateMemBlock(SystemTopo &topo, void **destPtr, void **srcPtr,long  long numBytes, MEM_OP copyType, int destIdx, int srcIdx) {
   switch (copyType) {
      case HOST_HOST_COPY: 
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         *srcPtr =topo.AllocMemByNode(srcIdx, numBytes);
         break;
      case HOST_PINNED_HOST_COPY: 
         *srcPtr =topo.AllocMemByNode(srcIdx, numBytes);
         checkCudaErrors(hipHostRegister(*srcPtr, numBytes, hipHostRegisterPortable));
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         break;
      case HOST_HOST_PINNED_COPY: 
         *srcPtr =topo.AllocMemByNode(srcIdx, numBytes);
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         checkCudaErrors(hipHostRegister(*destPtr, numBytes, hipHostRegisterPortable));
         break;
      case HOST_HOST_COPY_PINNED: 
         *srcPtr =topo.AllocMemByNode(srcIdx, numBytes);
         checkCudaErrors(hipHostRegister(*srcPtr, numBytes, hipHostRegisterPortable));
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         checkCudaErrors(hipHostRegister(*destPtr, numBytes, hipHostRegisterPortable));
         break;
      case DEVICE_HOST_COPY:
         topo.AllocDeviceMem(srcPtr, numBytes, srcIdx);
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         break;
      case DEVICE_HOST_PINNED_COPY:
         topo.AllocDeviceMem(srcPtr, numBytes, srcIdx);
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         checkCudaErrors(hipHostRegister(*destPtr, numBytes, hipHostRegisterPortable));
         break;
      case HOST_DEVICE_COPY:
         *srcPtr = topo.AllocMemByNode(srcIdx, numBytes);
         topo.AllocDeviceMem(destPtr, numBytes, destIdx);
         break;
      case HOST_PINNED_DEVICE_COPY:
         *srcPtr = topo.AllocMemByNode(srcIdx, numBytes);
         checkCudaErrors(hipHostRegister(*srcPtr, numBytes, hipHostRegisterPortable));
         topo.AllocDeviceMem(destPtr, numBytes, destIdx);
         break;
      case PEER_COPY_NO_UVA: 
      case DEVICE_DEVICE_COPY:
      case COPY_UVA:
         topo.AllocDeviceMem(srcPtr, numBytes, srcIdx);
         topo.AllocDeviceMem(destPtr, numBytes, destIdx);
         break;
      default:
         std::cout << "Error: unrecognized memory copy operation type for allocation" << std::endl; 
         break;
   }
}

float TimedMemOp(void **MemBlk, long long NumBytes, MEM_OP TimedOp) {
   #ifdef USING_CPP
   std::chrono::high_resolution_clock::time_point start_c, stop_c;
   auto total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
   #else
   struct timeval stop_t, start_t, total_t;
   #endif
   
   hipEvent_t start_e, stop_e; 
   checkCudaErrors(hipEventCreate(&start_e));
   checkCudaErrors(hipEventCreate(&stop_e));
   float OpTime = 0;
  
   switch (TimedOp) {
      case HOST_MALLOC:
         #ifdef USING_CPP
         start_c = std::chrono::high_resolution_clock::now();
         *MemBlk = malloc(NumBytes);
         stop_c = std::chrono::high_resolution_clock::now();
         total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);      
         OpTime = (float) total_c.count() * NANO_TO_MILLI;
         #else
         gettimeofday(&start_t, NULL);
         *MemBlk = malloc(NumBytes); 
         gettimeofday(&stop_t, NULL);
         timersub(&stop_t, &start_t, &total_t);
         OpTime = (float) total_t.tv_usec * MICRO_TO_MILLI;
         #endif
         break;
      case HOST_PINNED_MALLOC:
         checkCudaErrors(hipEventRecord(start_e, 0));      
         checkCudaErrors(hipHostMalloc(MemBlk, NumBytes));
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e));
         break;
      case DEVICE_MALLOC:
         checkCudaErrors(hipEventRecord(start_e, 0));
         checkCudaErrors(hipMalloc(MemBlk, NumBytes));
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e)); 
         break;
      case HOST_FREE:
         #ifdef USING_CPP
         start_c = std::chrono::high_resolution_clock::now();
         free(*MemBlk);
         stop_c = std::chrono::high_resolution_clock::now(); 
         total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
         OpTime = (float) total_c.count() * NANO_TO_MILLI;
         #else
         gettimeofday(&start_t, NULL);
         free(*MemBlk); 
         gettimeofday(&stop_t, NULL); 
         timersub(&stop_t, &start_t, &total_t); 
         OpTime = (float) total_t.tv_usec * MICRO_TO_MILLI;
         #endif
         break;
      case HOST_PINNED_FREE:
         checkCudaErrors(hipEventRecord(start_e, 0));
         checkCudaErrors(hipHostFree(*MemBlk));
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e));
         break;
      case DEVICE_FREE:
         checkCudaErrors(hipEventRecord(start_e, 0));
         checkCudaErrors(hipFree(*MemBlk)); 
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));   
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e));  
         break;
      default:
         std::cout << "Error: unrecognized timed memory operation type" << std::endl; 
         break;
   }
   checkCudaErrors(hipEventDestroy(start_e));
   checkCudaErrors(hipEventDestroy(stop_e));

   return OpTime;
}

int CalcRunSteps(std::vector<long long> &blockSteps, long long startStep, long long stopStep, long long numSteps) {
   int magStart = max((int)log10(startStep), 1);
   int magStop = log10(stopStep);

   long long start = pow(10, magStart);
   double stepSize = 10 * start / numSteps;
   int extra = (stopStep - pow(10, magStop)) / pow(10, magStop) * numSteps;
   long long stop = pow(10, magStop - 1) * (10 + extra); 
   int rangeSkip = numSteps / start;
   int totalSteps = (magStop - magStart) * (numSteps - rangeSkip) + extra + 1;  
   double step = start;

   for (long stepNum = 0; stepNum < totalSteps; ++stepNum) { 
      blockSteps.push_back(step);
      
      if ((stepNum) && (stepNum) % (numSteps - rangeSkip) == 0 && (stepSize * numSteps * 10) <= stop) {
         stepSize *= 10.0;
      } 
      
      step += stepSize; 
   }

   return totalSteps;
}

void PrintP2PBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData, bool testSockets) {
   long long blockSize = BURST_BLOCK_SIZE;
   int dataIdx = 0;
   int numSockets = 1;
   if (testSockets)
      numSockets = topo.NumSockets();

   //TODO: fix patterns
   int numPatterns = 1;
   if (false)
      numPatterns = NUM_PATTERNS;
   int matrixWidth = params.nDevices;
   int matrixHeight = params.nDevices * 4;
   
   //topo.DeviceGroupUVA(srcIdx, destIdx))   
   //if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {

   for (int i = 0; i < burstData.size(); i++) {
      for (int j = 0; j < burstData[i].size(); j++) {
         std::cout << burstData[i][j] << ",";
      }
      std::cout << std::endl;
   }

   std::cout << "\nDevice-To-Device Unidirectional Memory Transfers" << std::endl;
   std::cout << "Transfer Block Size: " << blockSize << std::endl;

   std::cout << "-------------------------------------------------------------------------------------------------" << std::endl;
   std::cout << "|\t\t\t\t|------------------------- Destination -------------------------|" << std::endl;
   std::cout << "|\t|---------------------------------------------------------------------------------------|" << std::endl;

   std::cout << "|\t| GPU   | Transfer\t|";
   for (int i = 0; i < matrixWidth; i++)
      std::cout << "\t\t" << i << "\t\t|";
   std::cout << "" << std::endl;
   
   std::cout << "|\t|   #   | Type\t\t|---------------------------------------------------------------|" << std::endl;
   std::cout << "|-------|-----------------------|---------------------------------------------------------------|" << std::endl;
   std::vector<int> deviceIdxs(params.nDevices, 0);
   for (int i = 0; i < matrixHeight; ++i) {

      std::cout << "|\t|  " << i  / 4 <<  "\t|";
      std::cout << std::setprecision(2) << std::fixed;          
      if (i % 4 == 0) {
         std::cout << " Standard D2D\t|    ";
      } else if (i % 4 == 1) {
         std::cout << " Peer, No UVA\t|    ";
      } else if (i % 4 == 2) {
         std::cout << " No Peer, UVA\t|    ";
      } else { 
         std::cout << " Peer, UVA\t|    ";
      }
      int dataIdx = 0;
      if (i % 4 == 0)
         deviceIdxs = std::vector<int>(matrixWidth, 0);
      
      for (int j = 0; j < matrixWidth; ++j) {
         std::cout << dataIdx + deviceIdxs[j];
         if (i % 4 == 0) {
            std::cout << "\t\t" << burstData[i / 4][dataIdx + deviceIdxs[j]] << "\t\t|";
            deviceIdxs[j]++;
         } else if ((i % 4 == 1) && topo.DeviceGroupCanP2P(i / 4, j)) {
            std::cout << "\t\t" << burstData[i / 4][dataIdx + deviceIdxs[j]] << "\t\t|";
            deviceIdxs[j]++;
         } else if ((i % 4 == 2) && topo.DeviceGroupUVA(i / 4, j)) {
            std::cout << "\t\t" << burstData[i / 4][dataIdx + deviceIdxs[j]] << "\t\t|";
            deviceIdxs[j]++;
         } else if ((i % 4 == 3) && topo.DeviceGroupUVA(i / 4, j) && topo.DeviceGroupCanP2P(i / 4, j)) { 
            std::cout << "\t\t" << burstData[i / 4][dataIdx + deviceIdxs[j]] << "\t\t|";
            deviceIdxs[j]++;
         } else { 
            std::cout << "\t\t  -\t\t|";
         }

         dataIdx++;
         if (topo.DeviceGroupCanP2P(i / 4, j))
            dataIdx++;
         if (topo.DeviceGroupUVA(i / 4, j)) {
            dataIdx++;
            if (topo.DeviceGroupCanP2P(i / 4, j)) 
               dataIdx++;
         }
      }
      
      std::cout << std::setprecision(5);     
      std::cout << std::endl;
      
      if (i + 1 < matrixHeight && (i + 1 != ((float) matrixHeight / 2.0)))
         std::cout << "|\t|-----------------------|---------------------------------------------------------------|" << std::endl;
      else if (i + 1 < matrixHeight)
         std::cout << "|Source |-----------------------|---------------------------------------------------------------|" << std::endl;
   }
   std::cout << "-------------------------------------------------------------------------------------------------" << std::endl;

}

void PrintHDBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData, bool testSockets) {
   long long blockSize = BURST_BLOCK_SIZE;

   int numSockets = 1;
   if (testSockets)
      numSockets = topo.NumSockets();

   //TODO: fix patterns
   int numPatterns = 1;
   if (false)
      numPatterns = NUM_PATTERNS;

/*   std::cout << "\nHost-To-Device and Device-To-Host Unidirectional Memory Transfers" << std::endl;
   std::cout << "Transfer Block Size: " << blockSize << std::endl;

   std::cout << "-------------------------------------------------------------------------------------------------" << std::endl;
   std::cout << "|\t\t|---------------|-------------------------- Destination ------------------------|" << std::endl;
   std::cout << "|   Transfer \t|---------------|---------------------------------------------------------------|" << std::endl;
   std::cout << "|   Point\t| NUMA \t\t|";
   for (int i = 0; i < topo.NumNodes(); i++)
      std::cout << "\t\t" << i << "\t\t|";
   std::cout << "" << std::endl;
   
   std::cout << "|\t\t| Node \t\t|---------------------------------------------------------------|" << std::endl;
   std::cout << "|\t\t| #     Mem Type";
   for (int i = 0; i < matrixWidth; i++){
      if (i % 2)
         std::cout << "|    Pinned\t";
      else
         std::cout << "|    Pageable\t";
   }
   std::cout << "|"<< std::endl;
   std::cout << "|---------------|-------|-------|---------------------------------------------------------------|" << std::endl;

   for (int i = 0; i < matrixHeight; ++i) {
      std::cout << "|\t\t|\t|";//<< std::endl;
      for (int j = 0; j < matrixWidth + 1; ++j)
         std::cout << "\t|\t";
      std::cout << std::endl; 

      std::cout << "|\t\t| " << i / (matrixHeight / topo.NumNodes()) <<  "\t|";
      if (i % 2)
         std::cout << " Pin\t|    ";
      else
         std::cout << " Page\t|    ";
 
      for (int j = 0; j < matrixWidth; ++j) {
         std::cout << burstData[i][j] << "\t|    ";
      }
          
      std::cout << "\n|\t\t|\t|";
      for (int j = 0; j < matrixWidth + 1; ++j)
         std::cout << "\t|\t";
      std::cout << std::endl;
      
      if (i + 1 < matrixHeight && (i + 1 != ((float) matrixHeight / 2.0)))
         std::cout << "|\t\t|-------|-----------------------------------------------------------------------|" << std::endl;
      else if (i + 1 < matrixHeight)
         std::cout << "|    Source     |-------|-----------------------------------------------------------------------|" << std::endl;
   }
   std::cout << "-------------------------------------------------------------------------------------------------" << std::endl;
*/
}

void PrintHHBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData, bool testSockets) {
   long long blockSize = BURST_BLOCK_SIZE;

   int numSockets = 1;
   if (testSockets)
      numSockets = topo.NumSockets();

   //TODO: fix patterns
   int numPatterns = 1;
   if (false)
      numPatterns = NUM_PATTERNS;

   int matrixWidth = HOST_MEM_TYPES * numSockets * topo.NumNodes();
   int matrixHeight = numPatterns * matrixWidth;
   
   std::cout << "Host-Host Multi-Numa Unidirectional Memory Transfers:" << std::endl;
   std::cout << "Transfer Block Size: " << blockSize << " (Bytes)"<< std::endl;
   //std::cout << "Num Patterns: " << numPatterns << std::endl;

   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      std::cout << "\nInitiating Sockets: " << socketIdx << std::endl;

      std::cout << "-------------------------------------------------------------------------------------------------" << std::endl;
      std::cout << "|\t\t|---------------|-------------------------- Destination ------------------------|" << std::endl;
      std::cout << "|   Transfer \t|---------------|---------------------------------------------------------------|" << std::endl;
      std::cout << "|   Point\t| NUMA \t\t|";
      for (int i = 0; i < topo.NumNodes(); i++)
         std::cout << "\t\t" << i << "\t\t|";
      std::cout << "" << std::endl;
      
      std::cout << "|\t\t| Node \t\t|---------------------------------------------------------------|" << std::endl;
      std::cout << "|\t\t| #     Mem Type";
      for (int i = 0; i < matrixWidth; i++){
         if (i % 2)
            std::cout << "|    Pinned\t";
         else
            std::cout << "|    Pageable\t";
      }
      std::cout << "|"<< std::endl;
      std::cout << "|---------------|-------|-------|---------------------------------------------------------------|" << std::endl;

      for (int i = 0; i < matrixHeight; ++i) {
         std::cout << "|\t\t|\t|";//<< std::endl;
         for (int j = 0; j < matrixWidth + 1; ++j)
            std::cout << "\t|\t";
         std::cout << std::endl; 

         std::cout << "|\t\t| " << i / (matrixHeight / topo.NumNodes()) <<  "\t|";
         if (i % 2)
            std::cout << " Pin\t|    ";
         else
            std::cout << " Page\t|    ";
    
         for (int j = 0; j < matrixWidth; ++j) {
            std::cout << burstData[i][j] << "\t|    ";
         }
             
         std::cout << "\n|\t\t|\t|";
         for (int j = 0; j < matrixWidth + 1; ++j)
            std::cout << "\t|\t";
         std::cout << std::endl;
         
         if (i + 1 < matrixHeight && (i + 1 != ((float) matrixHeight / 2.0)))
            std::cout << "|\t\t|-------|-----------------------------------------------------------------------|" << std::endl;
         else if (i + 1 < matrixHeight)
            std::cout << "|    Source     |-------|-----------------------------------------------------------------------|" << std::endl;
      }
      std::cout << "-------------------------------------------------------------------------------------------------" << std::endl;
   }
}

void PrintResults(std::ofstream &outFile, std::vector<long long> &steps, std::vector<std::vector<float> > &results, BenchParams &params) {
   
   if (!outFile.is_open()) {
      std::cout << "Failed to open file to print results" << std::endl;
      return;
   }
   std::vector<std::vector<float> >::iterator iter_o;
   std::vector<float>::iterator iter_i;
   std::vector<long long>::iterator iter_l = steps.begin();
   
   for (iter_o = results.begin(); iter_o != results.end(); ++iter_o) {
      outFile << std::fixed << *iter_l++ << ",";
      for (iter_i = (*iter_o).begin(); iter_i != (*iter_o).end(); ++iter_i) {
         outFile << std::fixed << *iter_i;
         if (iter_i + 1 != (*iter_o).end())
            outFile << ",";
      }
      outFile << std::endl;
   }
}


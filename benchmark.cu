#include "hip/hip_runtime.h"

// Benchmark includes and defines
#ifndef BENCH_HEADER_INC
#define BENCH_HEADER_INC
#include "benchmark.h"
#endif

// BenchParams class definition
#ifndef PARAM_CLASS_INC
#include "parameters.h"
#define PARAM_CLASS_INC
#endif

// SystemTopo class definition
#ifndef TOPOLOGY_CLASS_INC
#include "topology.h"
#define TOPOLOGY_CLASS_INC
#endif

// SystemTopo class definition
#ifndef TIMER_CLASS_INC
#include "timer.h"
#define TIMER_CLASS_INC
#endif

// Benchmark Tests
void RunBenchmarkSuite(BenchParams &params, SystemTopo &topo);
void TestMemoryOverhead(BenchParams &params, SystemTopo &topo);
void HHBurstTransferTest(BenchParams &params, SystemTopo &topo);
void HDBurstTransferTest(BenchParams &params, SystemTopo &topo);
void P2PBurstTransferTest(BenchParams &params, SystemTopo &topo);
void HHRangeTransferTest(BenchParams &params, SystemTopo &topo);
void HDRangeTransferTest(BenchParams &params, SystemTopo &topo);
void P2PRangeTransferTest(BenchParams &params, SystemTopo &topo);
void TestCongestion(BenchParams &params, SystemTopo &topo);
void TestMemoryUsage(BenchParams &params, SystemTopo &topo);

void ContentionSubTestMemAccess(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps);
void ContentionSubTestQPI(BenchParams &params, SystemTopo &topo);
void ContentionSubTestQPI(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps);
void ContentionSubTestPCIe(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps);
void ContentionSubTestP2P(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps);
void ContentionSubTestComplex(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps);

// Test Subfunctions
void MemCopyRun(SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, MEM_OP copyType, MEM_PATTERN patternType, int destIdx, int srcIdx, int numCopiesPerStep); 
float TimedMemOp(void **MemBlk, long long NumBytes, MEM_OP TimedOp); 
float TimedMemCopyStep(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, MEM_PATTERN patternType, int destIdx = 0, int srcIdx = 0);
float BurstMemCopy(SystemTopo &topo, long long blockSize, MEM_OP copyType, int destIdx, int srcIdx, int numSteps, MEM_PATTERN pattern = REPEATED); 
void MemCopyOp(char * destPtr, char *srcPtr, long stepSize, MEM_OP copyType, int destIdx = 0, int srcIdx = 0, hipStream_t stream = 0);

void RangeHDBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData); 
void RangeHHBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData); 
void RangeP2PBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData);

void BurstHDBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData); 
void BurstHHBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData); 
void BurstP2PBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData);  

// Support functions
void AllocMemBlocks(SystemTopo &topo, void **destPtr, void **srcPtr, long long numBytes, MEM_OP copyType, int destIdx = 0, int srcIdx = 0);
void AllocMemBlock(SystemTopo &topo, void **blkPtr, long long numBytes, MEM_TYPE blockType, int srcIdx, int extIdx = 0);
void FreeMemBlocks(SystemTopo &topo, void* destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx = 0, int srcIdx = 0);
void SetMemBlocks(SystemTopo &topo, void *destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx, int srcIdx, long long value); 
void SetMemBlock(SystemTopo &topo, void *blkPtr, long long numBytes, long long value, MEM_TYPE memType, int devIdx = 0);
int CalcRunSteps(std::vector<long long> &blockSteps, long long startStep, long long stopStep, long long numSteps);

// Results output
void PrintRangedHeader(BenchParams &params, SystemTopo &topo, std::ofstream &fileStream, BW_RANGED_TYPE testType); 
void PrintResults(std::ofstream &outFile, std::vector<long long> &steps, std::vector<std::vector<float> > &results);
void PrintHHBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData);
void PrintHDBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData);
void PrintP2PBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData);

std::vector<std::string> PatternNames{"Repeated","Random", "Linear Increasing","Linear Decreasing"};

 
/* Benchmark main()
 * 
 * 
 */
int main (int argc, char **argv) {
   BenchParams benchParams;  
   SystemTopo sysTopo;
   
   std::cout << "\nStarting Multi-GPU, Multi-NUMA Performance Test Suite...\n" << std::endl; 
   
   // Determine the number of recognized CUDA enabled devices
   checkCudaErrors(hipGetDeviceCount(&(benchParams.nDevices)));

   // Exit if system contains no devices
   if (benchParams.nDevices <= 0) {
      std::cout << "No devices found...aborting benchmarks." << std::endl;
      exit(-1);
   }

   // Setup benchmark parameters
   if (argc == 1) { 
      // No input file, use default parameters
      benchParams.SetDefault();
   
   } else if (argc == 2) {       
      // Parse input file and set parameter class local variables
      benchParams.ParseParamFile(std::string(argv[1]));

   } else {
      // Unknown input parameter list, abort test
      std::cout << "Aborting test: Incorrect number of input parameters" << std::endl;
      exit(-1);
   }

   // Print HWLOC topology info
   // Class constructor parses system topology from device files (linux)
   std::string topoFileName ="./results/" + benchParams.topoFile;
   std::ofstream topoFile(topoFileName.c_str());

   // Output system topology to file and screen
   sysTopo.PrintTopology(topoFile);

   // Output device properties for each CUDA enabled GPU to file
   sysTopo.PrintDeviceProps(benchParams);

   // Check parameters and fix parameters associated with boolean flags
   if (benchParams.runSustainedTests == false)
      benchParams.numStepRepeats = 1;

   if (benchParams.runSocketTests)
      benchParams.nSockets = sysTopo.NumSockets();

   if (!benchParams.runAllDevices)
      benchParams.nDevices = 1;

   // Print actual benchmark parameters for user/script parsing
   benchParams.PrintParams();

   // Run the benchmark per parameters defined in params
   RunBenchmarkSuite(benchParams, sysTopo);

   std::cout << "\nBenchmarks complete!\n" << std::endl;
  
   return 0;
}

void RunBenchmarkSuite(BenchParams &params, SystemTopo &topo) {

   // Memory overhead tests (allocation and deallocations)
   if (params.runMemoryOverheadTest)
      TestMemoryOverhead(params, topo);

   //Burst bandwidth tests (matricies printed)
   if (params.runBandwidthTestHH && params.runBurstTests)
      HHBurstTransferTest(params, topo);
   
   if (params.runBandwidthTestHD && params.runBurstTests)
      HDBurstTransferTest(params, topo);

   if (params.runBandwidthTestP2P && params.runBurstTests && params.runAllDevices)
      P2PBurstTransferTest(params, topo);

   //Ranged bandwidth tests (cvs files printed)
   if (params.runBandwidthTestHH && params.runRangeTests)
      HHRangeTransferTest(params, topo);
   
   if (params.runBandwidthTestHD && params.runRangeTests)
      HDRangeTransferTest(params, topo);

   if (params.runBandwidthTestP2P && params.runRangeTests && params.runAllDevices)
      P2PRangeTransferTest(params, topo);

   // Congestion benchmark tests
   if (params.runCongestionTest)
      TestCongestion(params, topo);

   if (params.runUsageTest) 
      TestMemoryUsage(params, topo);

}

void TestMemoryOverhead(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Ranged Memory Overhead Test...\n" << std::endl;
   
   char *deviceMem = NULL, * managedMem = NULL, * mappedMem = NULL; 
   char *hostMem = NULL, *hostPinnedMem = NULL, * hostCombinedMem = NULL;
   std::vector<long long> steps;
   std::vector<std::vector<float> > overheadData;
   int testNum = 0;
  
   CalcRunSteps(steps, params.rangeMemOverhead[0], params.rangeMemOverhead[1], params.rangeMemOverhead[2]);  
   overheadData.resize(steps.size());
   
   // Memory overhead test will run for each device utilizing the hipMalloc and hipFree functions
   // on the first iteration of the look, assuming there is atleast one device, the host will run the 
   // pinned and un-pinned memory tests
   for (int socketIdx = 0; socketIdx < params.nSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int numaIdx = 0; numaIdx < topo.NumNodes(); numaIdx++) { 
         topo.PinNumaNode(numaIdx);
        
         std::cout << "Test " << testNum++ << " Host Alloc/Free, Pinned/Pageable/Write-Combined\t" << "NUMA node: " << numaIdx << " CPU " << socketIdx << std::endl;            
         // Host based management for CASE 1 & 2
         for (long stepIdx = 0; stepIdx < steps.size(); stepIdx++) {
            long long chunkSize = steps[stepIdx];
            
            float hostAllocTime = 0, pinAllocTime = 0, combAllocTime = 0, managedAllocTime = 0, mappedAllocTime = 0;
            float hostFreeTime = 0, pinFreeTime = 0, combFreeTime = 0, managedFreeTime = 0, mappedFreeTime = 0; 
            // repeat same block run and average times
            for (int reIdx = 0; reIdx < params.numStepRepeats; reIdx++) {
               hostFreeTime += TimedMemOp((void **) &hostMem, chunkSize, HOST_MALLOC);
               hostAllocTime += TimedMemOp((void **) &hostMem, 0, HOST_FREE);

               if (params.testAllMemTypes) {
                  pinAllocTime += TimedMemOp((void **) &hostPinnedMem, chunkSize, HOST_PINNED_MALLOC);
                  pinFreeTime += TimedMemOp((void **) &hostPinnedMem, 0, HOST_PINNED_FREE); 
               
                  combAllocTime += TimedMemOp((void **) &hostCombinedMem, chunkSize, HOST_COMBINED_MALLOC);
                  combFreeTime += TimedMemOp((void **) &hostCombinedMem, chunkSize, HOST_COMBINED_FREE);

                  managedAllocTime += TimedMemOp((void **) &managedMem, chunkSize, MANAGED_MALLOC);
                  managedFreeTime += TimedMemOp((void **) &managedMem, chunkSize, MANAGED_FREE);

                  mappedAllocTime += TimedMemOp((void **) &mappedMem, chunkSize, MAPPED_MALLOC);
                  mappedFreeTime += TimedMemOp((void **) &mappedMem, chunkSize, MAPPED_FREE);
               }
            }
            overheadData[stepIdx].push_back(hostAllocTime / (float) params.numStepRepeats);
            overheadData[stepIdx].push_back(hostFreeTime / (float) params.numStepRepeats);

            overheadData[stepIdx].push_back(pinAllocTime / (float) params.numStepRepeats);
            overheadData[stepIdx].push_back(pinFreeTime / (float) params.numStepRepeats);

            overheadData[stepIdx].push_back(combAllocTime / (float) params.numStepRepeats);
            overheadData[stepIdx].push_back(combFreeTime / (float) params.numStepRepeats);

            overheadData[stepIdx].push_back(managedAllocTime / (float) params.numStepRepeats);
            overheadData[stepIdx].push_back(managedFreeTime / (float) params.numStepRepeats);
            
            overheadData[stepIdx].push_back(mappedFreeTime / (float) params.numStepRepeats);
            overheadData[stepIdx].push_back(mappedFreeTime / (float) params.numStepRepeats);

         }   
      }

      topo.PinNumaNode(0);
      // Device based memory management for CASE 3 & 4
      for (int currDev = 0; currDev < params.nDevices; currDev++) {
         checkCudaErrors(hipSetDevice(currDev)); 
         std::cout << "Test " << testNum++ << " Device Alloc/Free \t\t\t\t" << "CPU " << socketIdx << " Dev:" << currDev << std::endl;            
         
         for (long stepIdx = 0; stepIdx < steps.size(); stepIdx++) {
            long long chunkSize = steps[stepIdx];
            float devAllocTime = 0, devFreeTime = 0;

            // repeat same block run and average times
            for (int reIdx = 0; reIdx < params.numStepRepeats; reIdx++) {
               // CASE 3: Allocation of device memory  
               devAllocTime += TimedMemOp((void **) &deviceMem, chunkSize, DEVICE_MALLOC);
               // CASE 4: DeAllocation of device memory 
               devFreeTime += TimedMemOp((void **) &deviceMem, 0, DEVICE_FREE);
            }

            overheadData[stepIdx].push_back(devAllocTime / (float) params.numStepRepeats);
            overheadData[stepIdx].push_back(devFreeTime / (float) params.numStepRepeats);
         }
      }
   }
   
   std::string dataFileName = "./results/" + params.runTag + "_overhead.csv";
   std::ofstream overheadResultsFile(dataFileName.c_str());
   overheadResultsFile << params.nSockets << ",";
   overheadResultsFile << topo.NumNodes() << ",";
   overheadResultsFile << params.nDevices;
   if (params.testAllMemTypes)
      overheadResultsFile << ",t";
   else 
      overheadResultsFile << ",f";

   for (int i = 0; i < params.nDevices; i++)
      overheadResultsFile << "," << topo.GetDeviceName(i);
   overheadResultsFile << std::endl;
   PrintResults(overheadResultsFile, steps, overheadData);

   std::cout << "\nMemory Overhead Test Complete!" << std::endl;
   
}

void HHBurstTransferTest(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Host-Host Burst Bandwidth Tests...\n" << std::endl;

   std::vector<std::vector<float> > burstData;

   BurstHHBandwidthRun(params, topo, burstData); 
   PrintHHBurstMatrix(params, topo, burstData);
}

void HDBurstTransferTest(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Host-Device Burst Bandwidth Tests...\n" << std::endl;

   std::vector<std::vector<float> > burstData;

   BurstHDBandwidthRun(params, topo, burstData);  
   
   PrintHDBurstMatrix(params, topo, burstData);
   
}

void P2PBurstTransferTest(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Device-Device Burst Bandwidth Tests...\n" << std::endl;
   
   std::vector<std::vector<float> > burstData;

   BurstP2PBandwidthRun(params, topo, burstData);
 
   PrintP2PBurstMatrix(params, topo, burstData);
}

void HHRangeTransferTest(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Ranged Host-Host Bandwidth Tests...\n" << std::endl;
   
   std::vector<std::vector<float> > rangeData;
   std::vector<long long> steps;
  
   CalcRunSteps(steps, params.rangeHostHostBW[0], params.rangeHostHostBW[1], params.rangeHostHostBW[2]); 
   rangeData.resize(steps.size());
   
   RangeHHBandwidthRun(params, topo, steps, rangeData);

   // tt == Transfer Time
   std::string dataFileName = "./results/" + params.runTag + "_ranged_hh_tt.csv";
   std::ofstream ttResultsFileHH(dataFileName.c_str());
   PrintRangedHeader(params, topo, ttResultsFileHH, HH); 
   PrintResults(ttResultsFileHH, steps, rangeData);

   // Output throughput (GB/S) and block size
   for (int blkIdx = 0; blkIdx < steps.size(); ++blkIdx) {
      for (int runIdx = 0; runIdx < rangeData[blkIdx].size(); ++runIdx) {
         rangeData[blkIdx][runIdx] = ((double) steps[blkIdx]) / rangeData[blkIdx][runIdx] * 1.0e6;
         rangeData[blkIdx][runIdx] /= pow(2.0, 30.0);
      }
   }

   dataFileName = "./results/" + params.runTag + "_ranged_hh_bw.csv";
   std::ofstream bwResultsFileHH(dataFileName.c_str());
   PrintRangedHeader(params, topo, bwResultsFileHH, HH); 
   PrintResults(bwResultsFileHH, steps, rangeData);

   std::cout << "\nRanged Host-Host Bandwidth Tests complete!" << std::endl;
}

void HDRangeTransferTest(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning Ranged Host-Device Bandwidth Tests...\n" << std::endl;
   
   std::vector<std::vector<float> > rangeData;
   std::vector<long long> steps;

   CalcRunSteps(steps, params.rangeHostDeviceBW[0], params.rangeHostDeviceBW[1], params.rangeHostDeviceBW[2]); 
   rangeData.resize(steps.size());
   
   RangeHDBandwidthRun(params, topo, steps, rangeData);
   
   // tt == Transfer Time
   std::string dataFileName = "./results/" + params.runTag + "_ranged_hd_tt.csv";
   std::ofstream ttResultsFileHD(dataFileName.c_str());
   PrintRangedHeader(params, topo, ttResultsFileHD, HD); 
   PrintResults(ttResultsFileHD, steps, rangeData);

   // Output throughput (GB/S) and block size
   for (int blkIdx = 0; blkIdx < steps.size(); ++blkIdx) {
      for (int runIdx = 0; runIdx < rangeData[blkIdx].size(); ++runIdx) {
         rangeData[blkIdx][runIdx] = ((double) steps[blkIdx]) / rangeData[blkIdx][runIdx] * 1.0e6;
         rangeData[blkIdx][runIdx] /= pow(2.0, 30.0);
      }
   }

   dataFileName = "./results/" + params.runTag + "_ranged_hd_bw.csv";
   std::ofstream bwResultsFileHD(dataFileName.c_str());
   PrintRangedHeader(params, topo, bwResultsFileHD, HD); 
   PrintResults(bwResultsFileHD, steps, rangeData);

   std::cout << "\nHost-Device Ranged Bandwidth Tests complete!" << std::endl;
}

void P2PRangeTransferTest(BenchParams &params, SystemTopo &topo){
   std::cout << "\nRunning P2P Device Ranged Bandwidth test..." << std::endl;

   std::vector<std::vector<float> > rangeData;
   std::vector<long long> steps;

   CalcRunSteps(steps, params.rangeDeviceBW[0], params.rangeDeviceBW[1], params.rangeDeviceBW[2]); 
   rangeData.resize(steps.size());
   
   RangeP2PBandwidthRun(params, topo, steps, rangeData);

   // tt == Transfer Time
   std::string dataFileName = "./results/" + params.runTag + "_ranged_p2p_tt.csv";
   std::ofstream ttResultsFileP2P(dataFileName.c_str());
   PrintRangedHeader(params, topo, ttResultsFileP2P, P2P); 
   PrintResults(ttResultsFileP2P, steps, rangeData);

   // Output throughput (GB/S) and block size
   for (int blkIdx = 0; blkIdx < steps.size(); ++blkIdx) {
      for (int runIdx = 0; runIdx < rangeData[blkIdx].size(); ++runIdx) {
         rangeData[blkIdx][runIdx] = ((double) steps[blkIdx]) / rangeData[blkIdx][runIdx] * 1.0e6;
         rangeData[blkIdx][runIdx] /= pow(2.0, 30.0);
      }
   }

   dataFileName = "./results/" + params.runTag + "_ranged_p2p_bw.csv";
   std::ofstream bwResultsFileP2P(dataFileName.c_str());
   PrintRangedHeader(params, topo, bwResultsFileP2P, P2P); 
   PrintResults(bwResultsFileP2P, steps, rangeData);

   std::cout << "\nP2P Device Ranged Bandwidth Test Complete!" << std::endl;
}

void TestCongestion(BenchParams &params, SystemTopo &topo) {
   std::cout << "Running congestion tests..." << std::endl;

   // No parameters for this test, set default here
   // TODO: migrate relevent parameters to param file input

   params.testCongRange = true;
   params.numCongMemTypes = 2;
   params.numCongRepeats = 500;
   params.rangeCong[0] = 10000000;   // 100 KB
   params.rangeCong[1] = 10000000; // 100 MB
   params.rangeCong[2] = 1;
  
   std::vector<long long> blockSteps;
   //blockSteps.push_back(params.rangeCong[1]);
   CalcRunSteps(blockSteps, params.rangeCong[0], params.rangeCong[1], params.rangeCong[2]);
 
   /* Memory Access: Single Socket, Single Node
    *
    * Host-Host single node memory access
    * Inherently bidirectional transfer (actual bandwidth is double)
    * since destination is same as source.
    */

   //ContentionSubTestMemAccess(params, topo, blockSteps);

   /* QPI Bus Test (Multiple Sockets)
    *
    * Host-to-Host: bidirectional and unidirectional
    * Pin multiple cores on a single 
    */
   if (topo.NumSockets() >= 2)
      ContentionSubTestQPI(params, topo);
   else
      std::cout << "One Socket Detected: No inter-CPU communication bus to test!" << std::endl;

   /* PCIe (Single and Multiple Sockets)
    * 
    * Host-to-Device & Device-to-Host: bidirectional and unidirectional
    * Single socket (avoid QPI effects) to each combination of GPUs
    */
   //ContentionSubTestPCIe(params, topo, blockSteps);

   /* P2P
    * 
    * Host-Host Transfers: bidirectional and unidirectionsal 
    * Every combination up to one per transfer
    * Multiple from one to all devices (if more than one)
    */
   //ContentionSubTestP2P(params, topo, blockSteps);

   /* Complex Contention Test: P2P + Host-Device
    * 
    * 
    * 
    * 
    */
   //ContentionSubTestComplex(params, topo, blockSteps);

   std::cout << "Congestion tests complete!" << std::endl;
}

void TestMemoryUsage(BenchParams &params, SystemTopo &topo) {
   std::cout << "\nRunning memory usage pattern tests..." << std::endl;

   std::cout << "Test not yet implemented!" << std::endl;

   std::cout << "\nMemory usage patterns tests complete!" << std::endl;
}

void ContentionSubTestMemAccess(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps) {
   //int numThreads = 1;
   //int maxThreads = topo.NumPUsPerCore() * topo.NumCoresPerSocket();//topo.NumCores();//topo.NumCoresPerSocket();
   int PUsPerSocket = topo.NumCoresPerSocket() * topo.NumPUsPerCore();
   //float threadBW[topo.NumPUs()];
   float aggBW = 0;
   MEM_TYPE memType;
   //MEM_OP copyType = HOST_PINNED_HOST_COPY;
   float conv = 1.0e-6; 
   int NumOps = 2; 
   std::vector<std::vector<float> > data;
   data.resize(blockSteps.size());
   std::cout << PUsPerSocket << std::endl;
   std::cout << "Main Memory Contention Test" << std::endl;

   long long blockSize = blockSteps[blockSteps.size() - 1] / sizeof(double);
   //static double srcBlk[blockSize];
   //static double destBlk[blockSize];

   for (int socketCount = 0; socketCount < topo.NumSockets(); socketCount++) {
      for (int memIdx = 0; memIdx < params.numCongMemTypes; memIdx++) {
         for (int opIdx = 0; opIdx < NumOps; opIdx++) {
            if (memIdx == 0)
               memType = PAGE;
            else
               memType = PINNED;

            std::cout << "- " << socketCount << " - " << memIdx << " - " << opIdx << std::endl;
            int maxThreads = (socketCount + 1) * topo.NumCoresPerSocket() * topo.NumPUsPerCore();
            int numThreads = 1;
            do {
               omp_set_num_threads(numThreads);

               #pragma omp parallel
               {
                  // Get local thread ID
                  int threadIdx = omp_get_thread_num();
                  double * srcBlk, * destBlk;
                  /*
                  // Set the memory type and timers as indicated by memIdx

                  // pin threads to execution space (socket)
                  // TODO: Check to see pinning per core works then change this
                  //topo.PinPUBySocket(threadIdx / PUsPerSocket, threadIdx % PUsPerSocket);
                  topo.PinSocket(threadIdx / PUsPerSocket);
                  //topo.PinPU(threadIdx * (socketIdx + 1));
                  //todo.PinCoreBySocket();
    
                  // allocate src and dest blocks to NUMA nodes
                  AllocMemBlock(topo, (void **) &srcBlk, blockSteps[blockSteps.size() - 1], memType, threadIdx / PUsPerSocket);
                  AllocMemBlock(topo, (void **) &destBlk, blockSteps[blockSteps.size() - 1], memType, threadIdx / PUsPerSocket);
                  SetMemBlock(topo, srcBlk, blockSize, 1, memType);
                  SetMemBlock(topo, destBlk, blockSize, 0, memType);
                  

                  // Run ranged test for each thread, sync between steps
                  for (int stepIdx = 0; stepIdx < blockSteps.size(); stepIdx++) {
                     double totalTime = 0;
                     #pragma omp barrier

                     #pragma omp master
                     {
                     static Timer threadTimer(true);//useHostTimer);
                     threadTimer.StartTimer();
                     }
                     #pragma omp for
                     for (int repCount = 0; repCount < params.numCongRepeats; repCount++) {
                        
                        #pragma omp for 
                        for (register long long i = 0; i < blockSize; ++i)
                           destBlk[i] = srcBlk[i];   
                        
                        //threadTimer.StopTimer();                     
                        //totalTime += (double) threadTimer.ElapsedTime();
                     }
                     
                     #pragma omp barrier*/
                     // initiate transfers on each thread simultaneously 
       
                     /*for (int repCount = 0; repCount < params.numCongRepeats; repCount++) {
                        if (opIdx == 0)
                           MemCopyOp((char *) destBlk, (char *) srcBlk, blockSteps[stepIdx], copyType);
                        else
                           SetMemBlock(topo, srcBlk, blockSteps[blockSteps.size() - 1], repCount % 2, memType);
                           //SetMemBlock(topo, destBlk, blockSteps[blockSteps.size() - 1], repCount % 2, memType);
                     }*/
                     /*#pragma omp master
                     {
                     totalTime = totalTime / (double) params.numCongRepeats;
                     long long totalBytes = blockSteps[blockSteps.size() - 1];
                     double bandwidth = ((double) totalBytes / (double) pow(2.0, 30.0)) / (totalTime * conv);
                     //threadBW[threadIdx] = bandwidth; 

                     // sum aggragite bandwidths
                     //#pragma omp atomic

                     //#pragma omp barrier

                     //#pragma omp single
                     //{
                        data[0].push_back(aggBW);
                        aggBW = bandwidth;
                        //for (int i = 0; i < omp_get_num_threads(); ++i) {
                        //   aggBW += threadBW[i];
                        //   data[stepIdx].push_back(threadBW[i]);
                        //}
                        std::cout << numThreads << "|" << blockSize << ": " << aggBW << std::endl;
                        aggBW = 0;
                     }
                  }
                  */                  
                  if (memType == PAGE) {
                     topo.FreeHostMem(srcBlk, params.rangeCong[1]);
                     topo.FreeHostMem(destBlk, params.rangeCong[1]);
                  } else {
                     topo.FreePinMem(srcBlk, params.rangeCong[1]);
                     topo.FreePinMem(destBlk, params.rangeCong[1]);
                  }
               }

               if (numThreads == 1)
                  numThreads++;
               else 
                  numThreads *= 2;

            } while (numThreads <= maxThreads);
         }
      }
   }

   // Output results
   // Header: sockets, memtypes, max thread count, test range
   std::string dataFileName = "./results/congestion/" + params.runTag + "_congestion_host_mem_.csv";
   std::ofstream resultsFile(dataFileName.c_str());
   resultsFile << topo.NumSockets() << ","; 
   resultsFile << params.numCongMemTypes << ","; 
   resultsFile << topo.NumCoresPerSocket() * topo.NumPUsPerCore() << ","; 
   if (params.testCongRange) 
      resultsFile << "t" << std::endl;
   else  
      resultsFile << "f" << std::endl;
   PrintResults(resultsFile, blockSteps, data);
}

void ContentionSubTestQPI(BenchParams &params, SystemTopo &topo) {
   int NumDirs = 2; // Copy Directions: 0->1 unidirectional, bidirectional
   MEM_TYPE memType;
   MEM_OP copyType = HOST_PINNED_HOST_COPY;

   float aggBW = 0;
   float conv = 1.0e-6; 
   long long blockSize = params.rangeCong[1];
   float threadBW[topo.NumPUs()];
   std::vector<long long> blockSteps;
   blockSteps.push_back(blockSize); 
   std::vector<std::vector<float> > data;
   data.resize(1);

   std::cout << "Socket-Socket Communication Contention" << std::endl;
   for (int copyDir = 0; copyDir < NumDirs; copyDir++) {
      for (int memIdx = 0; memIdx < params.numCongMemTypes; memIdx++) {
         std::cout << "- " << copyDir << " - " << memIdx << " - " << std::endl;

         if (memIdx == 0)
            memType = PAGE;
         else
            memType = PINNED;
          
         int numThreads = 1;
         int MaxThreads = topo.NumPUs();
         if (copyDir == 0)
            MaxThreads /= topo.NumSockets();
         
         do {
            omp_set_num_threads(numThreads);
            #pragma omp parallel
            {
               // Get local thread ID
               int threadIdx = omp_get_thread_num();
               void * srcBlk, * destBlk;
               int srcNode = 0, destNode = 0, coreIdx = 0;
               Timer threadTimer(true);

               // Set the memory type and timers as indicated by memIdx
               coreIdx = threadIdx % topo.NumCoresPerSocket();
               if (copyDir == 0) { // unidirectional, only testing one direction; should be equivalent 
                  srcNode = 0;
                  destNode = 1;
               } else if (copyDir == 1) { // bidirectional
                  srcNode = (threadIdx / topo.NumPUsPerSocket()) % 2;
                  destNode = (threadIdx / topo.NumPUsPerSocket() + 1) % 2;
               } 
               
               // allocate memory and pin threads to execution space
               // TODO: Check to see pinning per core works then change this
               //topo.PinCore(coreIdx); //
               topo.PinCoreBySocket(srcNode, coreIdx);
               AllocMemBlock(topo, &srcBlk, blockSize, memType, srcNode);
               AllocMemBlock(topo, &destBlk, blockSize, memType, destNode);
               SetMemBlock(topo, srcBlk, blockSize, 0x0, memType);
               SetMemBlock(topo, destBlk, blockSize, 0x0, memType);
  
               // initiate transfers on each thread simultaneously 
               #pragma omp barrier
               threadTimer.StartTimer();
               
               for (register int repCount = 0; repCount < params.numCongRepeats; repCount++) 
                  MemCopyOp((char *) destBlk, (char *) srcBlk, blockSize, copyType);
               
               //#pragma omp barrier
               threadTimer.StopTimer();     
                  
               // calculate thread local bandwidth
               double time = (double) threadTimer.ElapsedTime() / (double) params.numCongRepeats;
               double bandwidth = ((double) blockSize / (double) pow(2.0, 30.0)) / (time * conv);
              
               threadBW[threadIdx] = bandwidth; 

               // sum aggragite bandwidths
               #pragma omp atomic
               aggBW += bandwidth;

               #pragma omp barrier

               #pragma omp single
               {
                  for (int i = 0; i < omp_get_num_threads(); ++i)
                     data[0].push_back(threadBW[i]);
                  data[0].push_back(aggBW);
                  
                  std::cout << numThreads << ": " << aggBW << std::endl;
                  aggBW = 0;
               }
            
               if (memType == PAGE) {
                  topo.FreeHostMem(srcBlk, blockSize);
                  topo.FreeHostMem(destBlk, blockSize);
               } else {
                  topo.FreePinMem(srcBlk, blockSize);
                  topo.FreePinMem(destBlk, blockSize);
               }

            }
            if (numThreads == 1)
               numThreads++;
            else 
               //numThreads*=2;
               numThreads+=2;
         } while (numThreads <= MaxThreads);
      }
   }

   // Output results
   std::string dataFileName = "./results/congestion/" + params.runTag + "_congestion_inter_socket_.csv";
   std::ofstream resultsFile(dataFileName.c_str());

   //TODO Confirm these are correct header values
   resultsFile << topo.NumSockets() << ","; 
   resultsFile << params.numCongMemTypes << ","; 
   resultsFile << topo.NumPUsPerCore() * topo.NumCoresPerSocket() << ","; 
   if (params.testCongRange) 
      resultsFile << "t" << std::endl;
   else  
      resultsFile << "f" << std::endl;
   PrintResults(resultsFile, blockSteps, data);

}

void ContentionSubTestPCIe(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps) {
   int maxThreads = 1;   

   // Output results
   std::string dataFileName = "./results/congestion/" + params.runTag + "_congestion_pcie_.csv";
   std::ofstream resultsFile(dataFileName.c_str());

   //TODO Confirm these are correct header values
   resultsFile << topo.NumSockets() << ","; 
   resultsFile << params.numCongMemTypes << ","; 
   resultsFile << maxThreads << ","; 
   if (params.testCongRange) 
      resultsFile << "t" << std::endl;
   else  
      resultsFile << "f" << std::endl;
   //PrintResults(resultsFile, blockSteps, data);
}

void ContentionSubTestP2P(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps) {
   int maxThreads = 1;

   std::string dataFileName = "./results/congestion/" + params.runTag + "_congestion_p2p_.csv";
   std::ofstream resultsFile(dataFileName.c_str());

   //TODO Confirm these are correct header values
   resultsFile << topo.NumSockets() << ","; 
   resultsFile << params.numCongMemTypes << ","; 
   resultsFile << maxThreads << ","; 
   if (params.testCongRange) 
      resultsFile << "t" << std::endl;
   else  
      resultsFile << "f" << std::endl;
   //PrintResults(resultsFile, blockSteps, data);
}

void ContentionSubTestComplex(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps) {

   int maxThreads = 1;
   std::string dataFileName = "./results/congestion/" + params.runTag + "_congestion_complex_.csv";
   std::ofstream resultsFile(dataFileName.c_str());

   //TODO Confirm these are correct header values
   resultsFile << topo.NumSockets() << ","; 
   resultsFile << params.numCongMemTypes << ","; 
   resultsFile << maxThreads << ","; 
   if (params.testCongRange) 
      resultsFile << "t" << std::endl;
   else  
      resultsFile << "f" << std::endl;
   //PrintResults(resultsFile, blockSteps, data);
}

void BurstHHBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData) { 
   long long blockSize = params.burstBlockSize;
   int numNodes = topo.NumNodes();
   int numSockets = params.nSockets;
   int numPatterns = 1;

   if (params.runPatternsHD)
      numPatterns = NUM_PATTERNS;

   burstData.resize(numPatterns * numSockets);
   double convConst = (double) blockSize / (double) pow(2.0, 30.0) * (double) 1.0e6; 

   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
      
      for (int patternNum = 0; patternNum < numPatterns; patternNum ++) {
   
         MEM_PATTERN pattern = REPEATED;
         if (patternNum == 1)
            pattern = LINEAR_INC;
         if (patternNum == 2)
            pattern = LINEAR_DEC;
      
         for (int srcIdx = 0; srcIdx < numNodes; srcIdx++) { 

            for (int destIdx = 0; destIdx < numNodes; destIdx++) { 
               // HtoH Ranged Transfer - Pageable Memory
               int rowIdx = socketIdx * numPatterns + patternNum;
               burstData[rowIdx].push_back(convConst / BurstMemCopy(topo, blockSize, HOST_HOST_COPY, destIdx, srcIdx, params.numStepRepeats, pattern));        
              
               if (params.testAllMemTypes) {
                  // HtoH Ranged Transfer - Pinned Memory Src
                  burstData[rowIdx].push_back(convConst / BurstMemCopy(topo, blockSize, HOST_PINNED_HOST_COPY, destIdx, srcIdx, params.numStepRepeats, pattern)); 
                  // HtoH Ranged Transfer - Pinned Memory Dest
                  burstData[rowIdx].push_back(convConst / BurstMemCopy(topo, blockSize, HOST_HOST_PINNED_COPY, destIdx, srcIdx, params.numStepRepeats, pattern));        
                  // HtoH Ranged Transfer - Pinned Memory Both
                  burstData[rowIdx].push_back(convConst / BurstMemCopy(topo, blockSize, HOST_HOST_COPY_PINNED, destIdx, srcIdx, params.numStepRepeats, pattern));

                  // HtoH Ranged Transfer - WC Memory Src
                  burstData[rowIdx].push_back(convConst / BurstMemCopy(topo, blockSize, HOST_COMBINED_HOST_COPY, destIdx, srcIdx, params.numStepRepeats, pattern));        
                  // HtoH Ranged Transfer - WC Memory Dest
                  burstData[rowIdx].push_back(convConst / BurstMemCopy(topo, blockSize, HOST_HOST_COMBINED_COPY, destIdx, srcIdx, params.numStepRepeats, pattern));
                  // HtoH Ranged Transfer - WC Memory Both 
                  burstData[rowIdx].push_back(convConst / BurstMemCopy(topo, blockSize, HOST_HOST_COPY_COMBINED, destIdx, srcIdx, params.numStepRepeats, pattern));
               }       
            }
         }
      }
   }
}

void BurstHDBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData) { 
   long long blockSize = params.burstBlockSize;
   double convConst = (double) blockSize / (double) pow(2.0, 30.0) * (double) 1.0e6; 

   int numSockets = params.nSockets;
   int numPatterns = 1;
   if (params.runPatternsHD)
      numPatterns = NUM_PATTERNS;
   
   burstData.resize(numPatterns * numSockets);
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
      
      for (int patternNum = 0; patternNum < numPatterns; patternNum++) {
      
         MEM_PATTERN pattern = REPEATED;
         if (patternNum == 1)
            pattern = LINEAR_INC;
         if (patternNum == 2)
            pattern = LINEAR_DEC;
    
         for (int srcIdx = 0; srcIdx < topo.NumNodes(); srcIdx++) { 

            //Host-Device Memory Transfers
            for (int destIdx = 0; destIdx < params.nDevices; destIdx++) {
               topo.SetActiveDevice(destIdx); 
               int rowIdx = socketIdx * numPatterns + patternNum; 

               // HtoD Ranged Transfer - Pageable Memory
               burstData[rowIdx].push_back( convConst / BurstMemCopy(topo, blockSize, HOST_DEVICE_COPY, destIdx, srcIdx, params.numStepRepeats, pattern));        
               
               // DtoH Ranged Transfer - Pageable Memory
               burstData[rowIdx].push_back( convConst / BurstMemCopy(topo, blockSize, DEVICE_HOST_COPY, srcIdx, destIdx, params.numStepRepeats, pattern));        
               
               if ( params.testAllMemTypes) {      
                  // HtoD Ranged Transfer - Pinned Memory
                  burstData[rowIdx].push_back( convConst / BurstMemCopy(topo, blockSize, HOST_PINNED_DEVICE_COPY, destIdx, srcIdx, params.numStepRepeats, pattern));
                  // DtoH Ranged Transfer - Pinned Memory
                  burstData[rowIdx].push_back( convConst / BurstMemCopy(topo, blockSize, DEVICE_HOST_PINNED_COPY, srcIdx, destIdx, params.numStepRepeats, pattern)); 
                  // HtoD Ranged Transfer - WC Memory
                  burstData[rowIdx].push_back( convConst / BurstMemCopy(topo, blockSize, HOST_COMBINED_DEVICE_COPY, destIdx, srcIdx, params.numStepRepeats, pattern));
                  // DtoH Ranged Transfer - WC Memory
                  burstData[rowIdx].push_back( convConst / BurstMemCopy(topo, blockSize, DEVICE_HOST_COMBINED_COPY, srcIdx, destIdx, params.numStepRepeats, pattern)); 
               }
            }
         }
      }
   }
}

void BurstP2PBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData) { 
   long long blockSize = params.burstBlockSize;
   double convConst = (double) blockSize / (double) pow(2.0, 30.0) * (double) 1.0e-6; 
   
   burstData.resize(topo.NumGPUs() * params.nSockets);
   for (int socketIdx = 0; socketIdx < params.nSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumGPUs(); srcIdx++) { 
         //topo.SetActiveDevice(srcIdx); 
         for (int destIdx = 0; destIdx < topo.NumGPUs(); destIdx++) { 
            // DtoD Burst Transfer - No Peer, No UVA
            burstData[socketIdx * topo.NumGPUs() + srcIdx].push_back(convConst / BurstMemCopy(topo, blockSize, DEVICE_DEVICE_COPY, destIdx, srcIdx, params.numStepRepeats)); 
            // DtoD Burst Transfer - Peer, No UVA
            if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {
               topo.DeviceGroupSetP2P(srcIdx, destIdx, true);
               burstData[socketIdx * topo.NumGPUs() + srcIdx].push_back(convConst / BurstMemCopy(topo, blockSize, PEER_COPY_NO_UVA, destIdx, srcIdx, params.numStepRepeats)); 
               topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
            }

            if (topo.DeviceGroupUVA(srcIdx, destIdx)) {  
               // DtoD Burst Transfer - No Peer, UVA
               burstData[socketIdx * topo.NumGPUs() + srcIdx].push_back(convConst / BurstMemCopy(topo, blockSize, COPY_UVA, destIdx, srcIdx, params.numStepRepeats)); 
               
               // DtoD Burst Transfer - Peer, UVA
               if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {
                  topo.DeviceGroupSetP2P(srcIdx, destIdx, true);
                  burstData[socketIdx * topo.NumGPUs() + srcIdx].push_back( convConst / BurstMemCopy(topo, blockSize, COPY_UVA, destIdx, srcIdx, params.numStepRepeats));        
                  topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
               }
            }
         }
      }
   }
}

void RangeHHBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData) {
   int testNum = 0;
   long numRepeats = params.numStepRepeats;
   
   for (int socketIdx = 0; socketIdx < params.nSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumNodes(); srcIdx++) { 

         //Host To Host Memory Transfers
         for (int destIdx = 0; destIdx < topo.NumNodes(); destIdx++) { 
            // HtoH Ranged Transfer - Pageable Memory
            std::cout << "Test " << testNum++ << " HtoH, Pageable Memory\t\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY, REPEATED, destIdx, srcIdx, numRepeats); 
            if (params.runPatternsHD) {
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats);
            }

            if (params.testAllMemTypes) {
               // HtoH Ranged Transfer - Pinned Memory Src Host
               std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Src  \t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_HOST_COPY, REPEATED, destIdx, srcIdx, numRepeats);
               if (params.runPatternsHD){ 
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_HOST_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_HOST_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats); 
               }

               // HtoH Ranged Transfer - Pinned Memory Dest Host
               std::cout << "Test " << testNum++ << " HtoH, Pinned Memory Dest \t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_PINNED_COPY, REPEATED, destIdx, srcIdx, numRepeats); 
               if (params.runPatternsHD) {
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_PINNED_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_PINNED_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats); 
               }

              // HtoH Ranged Transfer - Pinned Memory Both Hosts
               std::cout << "Test " << testNum++ << " HtoH, Both Pinned Memory \t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, REPEATED, destIdx, srcIdx, numRepeats); 
               if (params.runPatternsHD) {
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, LINEAR_INC, destIdx, srcIdx, numRepeats); 
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, LINEAR_DEC, destIdx, srcIdx, numRepeats);
               } 

               // HtoH Ranged Transfer - Write-Combined Memory Src Host
               std::cout << "Test " << testNum++ << " HtoH, Write-Combined Memory Src    \tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_HOST_COPY, REPEATED, destIdx, srcIdx, numRepeats);
               if (params.runPatternsHD){ 
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_COMBINED_HOST_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_COMBINED_HOST_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats); 
               }

               // HtoH Ranged Transfer - Write-Combined Memory Dest Host
               std::cout << "Test " << testNum++ << " HtoH, Write-Combined Memory Dest  \tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COMBINED_COPY, REPEATED, destIdx, srcIdx, numRepeats); 
               if (params.runPatternsHD) {
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COMBINED_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COMBINED_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats); 
               }

               // HtoH Ranged Transfer - Write-Combined Memory Both Hosts
               std::cout << "Test " << testNum++ << " HtoH, Both Write-Combined Memory\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest NUMA: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_COMBINED, REPEATED, destIdx, srcIdx, numRepeats); 
               if (params.runPatternsHD) {
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_COMBINED, LINEAR_INC, destIdx, srcIdx, numRepeats); 
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_HOST_COPY_COMBINED, LINEAR_DEC, destIdx, srcIdx, numRepeats);
               }
            }
         }
      }
   }
}

void RangeHDBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData) {
   int testNum = 0;
   long numRepeats = params.numStepRepeats;  
 
   for (int socketIdx = 0; socketIdx < params.nSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumNodes(); srcIdx++) { 

         //Host-Device PCIe Memory Transfers
         for (int destIdx = 0; destIdx < params.nDevices; destIdx++) {
             // HtoD Ranged Transfer - Pageable Memory
            std::cout << "Test " << testNum++ << " HtoD, Pageable Memory\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, REPEATED, destIdx, srcIdx, numRepeats); 
            if (params.runPatternsHD) {
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats); 
            }

            // DtoH Ranged Transfer - Pageable Memory
            std::cout << "Test " << testNum++ << " DtoH, Pageable Memory\t\tCPU: " << socketIdx << "\t\tDev Src: " << srcIdx << "\tNUMA dest: " << srcIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, REPEATED, srcIdx, destIdx, numRepeats); 
            if (params.runPatternsHD) {
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, LINEAR_INC, srcIdx, destIdx, numRepeats); 
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, LINEAR_DEC, srcIdx, destIdx, numRepeats); 
            }
            
            if (params.testAllMemTypes) {
               // HtoD Ranged Transfer - Pinned Memory
               std::cout << "Test " << testNum++ << " HtoD, Pinned Memory\t\tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_DEVICE_COPY, REPEATED, destIdx, srcIdx, numRepeats); 
               if (params.runPatternsHD) {
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_DEVICE_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_PINNED_DEVICE_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats); 
               } 

               // DtoH Ranged Transfer - Pinned Memory
               std::cout << "Test " << testNum++ << " DtoH, Pinned Memory\t\tCPU: " << socketIdx << "\t\tSrc Dev: " << srcIdx << "\tNUMA Dest: " << srcIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_PINNED_COPY, REPEATED, srcIdx, destIdx, numRepeats); 
               if (params.runPatternsHD) {
                  MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_PINNED_COPY, LINEAR_INC, srcIdx, destIdx, numRepeats); 
                  MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_PINNED_COPY, LINEAR_DEC, srcIdx, destIdx, numRepeats);
               } 

               // HtoD Ranged Transfer - Write-Combined Memory
               std::cout << "Test " << testNum++ << " HtoD, Write-Combined Memory \tCPU: " << socketIdx << "\t\tNUMA Src: " << srcIdx << "\tDest Dev: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, HOST_COMBINED_DEVICE_COPY, REPEATED, destIdx, srcIdx, numRepeats); 
               if (params.runPatternsHD) {
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_COMBINED_DEVICE_COPY, LINEAR_INC, destIdx, srcIdx, numRepeats); 
                  MemCopyRun(topo, blockSteps, bandwidthData, HOST_COMBINED_DEVICE_COPY, LINEAR_DEC, destIdx, srcIdx, numRepeats); 
               } 

               // DtoH Ranged Transfer - Write-Combined Memory
               std::cout << "Test " << testNum++ << " DtoH, Write-Combined Memory\t\tCPU: " << socketIdx << "\t\tSrc Dev: " << srcIdx << "\tNUMA Dest: " << srcIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COMBINED_COPY, REPEATED, srcIdx, destIdx, numRepeats); 
               if (params.runPatternsHD) {
                  MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COMBINED_COPY, LINEAR_INC, srcIdx, destIdx, numRepeats); 
                  MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_HOST_COMBINED_COPY, LINEAR_DEC, srcIdx, destIdx, numRepeats);
               }
            }               
         }
      }
   }
}

void RangeP2PBandwidthRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData) {
   int testNum = 0;
   long numRepeats = params.numStepRepeats;  
   
   for (int socketIdx = 0; socketIdx < params.nSockets; socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int srcIdx = 0; srcIdx < topo.NumGPUs(); srcIdx++) { 

         for (int destIdx = 0; destIdx < topo.NumGPUs(); destIdx++) { 
            // DtoD Ranged Transfer - No Peer, No UVA
            std::cout << "Test " << testNum++ << " Device-To-Device, No Peer, No UVA\tCPU: " << socketIdx << "\tSrc Device: " << srcIdx << "\tDest Device: " << destIdx << std::endl;
            MemCopyRun(topo, blockSteps, bandwidthData, DEVICE_DEVICE_COPY, REPEATED, destIdx, srcIdx, numRepeats); 

            // DtoD Ranged Transfer - Peer, No UVA
            if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {
            std::cout << "Test " << testNum++ << " Device-To-Device, Peer Enabled, No UVA\tCPU: " << socketIdx << "\tSrc Device: " << srcIdx << "\tDest Device: " << destIdx << std::endl;
               topo.DeviceGroupSetP2P(srcIdx, destIdx, true);
               MemCopyRun(topo, blockSteps, bandwidthData, PEER_COPY_NO_UVA, REPEATED, destIdx, srcIdx, numRepeats);  
               topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
            }
            
            if (topo.DeviceGroupUVA(srcIdx, destIdx)) {  
               // DtoD Ranged Transfer - No Peer, UVA
               std::cout << "Test " << testNum++ << " Device-To-Device, No Peer, UVA\t\tCPU: " << socketIdx << "\tSrc Device: " << srcIdx << "\tDest Device: " << destIdx << std::endl;
               MemCopyRun(topo, blockSteps, bandwidthData, COPY_UVA, REPEATED, destIdx, srcIdx, numRepeats); 
 
               // DtoD Ranged Transfer - Peer, UVA
               if (topo.DeviceGroupCanP2P(srcIdx, destIdx)) {
                  std::cout << "Test " << testNum++ << " Device-To-Device, Peer Enabled, No UVA\tCPU: " << socketIdx << "\tSrc Device: " << srcIdx << "\tDest Device: " << destIdx << std::endl;
                  topo.DeviceGroupSetP2P(srcIdx, destIdx, true);
                  MemCopyRun(topo, blockSteps, bandwidthData, COPY_UVA, REPEATED, destIdx, srcIdx, numRepeats); 
                  topo.DeviceGroupSetP2P(srcIdx, destIdx, false);
               }
            }
         }
      }
   }
}

void MemCopyRun(SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &data, MEM_OP copyType, MEM_PATTERN pattern, int destIdx, int srcIdx, int numCopies) {
   char *destPtr, *srcPtr; 
   long totalSteps = blockSteps.size();
   
   std::vector<float> timedRun(totalSteps, 0.0);
   long long blockSize = blockSteps[totalSteps - 1 ];

   AllocMemBlocks(topo, (void **) &destPtr, (void **) &srcPtr, blockSize, copyType, destIdx, srcIdx);
   SetMemBlocks(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx, -1);
  
   for (long stepNum = 0; stepNum < totalSteps; ++stepNum) { 
      data[stepNum].push_back(TimedMemCopyStep((char *) destPtr, (char *) srcPtr, blockSteps[stepNum], blockSize, numCopies, copyType, pattern, destIdx, srcIdx));
   }
  
   FreeMemBlocks(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx);
}

float BurstMemCopy(SystemTopo &topo, long long blockSize, MEM_OP copyType, int destIdx, int srcIdx, int numSteps, MEM_PATTERN pattern) {  
   float elapsedTime = 0;
   char *destPtr, *srcPtr;

   AllocMemBlocks(topo, (void **) &destPtr, (void **) &srcPtr, blockSize, copyType, destIdx, srcIdx);
   SetMemBlocks(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx, -1); 

   elapsedTime = TimedMemCopyStep((char *) destPtr, (char *) srcPtr, blockSize, blockSize, numSteps, copyType, pattern, destIdx, srcIdx);

   FreeMemBlocks(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx);

   return elapsedTime;
}

float TimedMemCopyStep(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, MEM_PATTERN patternType, int destIdx, int srcIdx) {
   long long offset = 0;
   float totalTime = 0; 
   long long maxFrameSize = pow(2, 27);
   long long gap = maxFrameSize - stepSize;

   bool usingPattern = false;
   if (blockSize < maxFrameSize) {
      numCopiesPerStep *= 5;
      switch (patternType) {
         case LINEAR_INC:
            usingPattern = true;
            offset = 0;
            break;
         case LINEAR_DEC:
            usingPattern = true;
            offset = blockSize - stepSize;
            break;
         default:
            usingPattern = false;
            break;
      }
   }

   #ifdef USING_CPP
   std::chrono::high_resolution_clock::time_point start_c, stop_c;
   #else
   struct timeval stop_t, start_t, total_t;
   #endif
   
   hipEvent_t start_e, stop_e; 
   checkCudaErrors(hipEventCreate(&start_e));
   checkCudaErrors(hipEventCreate(&stop_e)); 

   if (copyType == HOST_HOST_COPY) {
      #ifdef USING_CPP
      start_c = std::chrono::high_resolution_clock::now();
      #else
      gettimeofday(&start_t, NULL);
      #endif
   } else{
      checkCudaErrors(hipEventRecord(start_e, 0));
   }

   for (int copyIdx = 0; copyIdx < numCopiesPerStep; copyIdx++) {

      MemCopyOp(destPtr + offset, srcPtr + offset, stepSize, copyType, destIdx, srcIdx); 

      if (usingPattern) {
         switch (patternType) {
       
           case REPEATED:
               offset = 0;
               break;
            case LINEAR_INC:
               offset += gap;
               if (offset > blockSize)
                  offset = 0;
               break;
            case LINEAR_DEC:
               offset -= gap;
               if (offset < 0)
                  offset = blockSize - stepSize;
               break;
            default:
               offset = 0;
               std::cout << "Error: unrecognized memory access pattern during copy operation" << std::endl; 
               break;
         }
      }
   }

   if (copyType == HOST_HOST_COPY) {
      #ifdef USING_CPP
      stop_c = std::chrono::high_resolution_clock::now(); 
      auto total_c = std::chrono::duration_cast<std::chrono::microseconds>(stop_c - start_c);
      totalTime = (float) total_c.count(); 
      #else
      gettimeofday(&stop_t, NULL); 
      timersub(&stop_t, &start_t, &total_t); 
      totalTime = (float) total_t.tv_usec + (float) total_t.tv_sec * 1.0e6;
      #endif
   } else{
      checkCudaErrors(hipEventRecord(stop_e, 0));
      checkCudaErrors(hipEventSynchronize(stop_e));   
      checkCudaErrors(hipEventElapsedTime(&totalTime, start_e, stop_e));  
      totalTime = totalTime * 1.0e-3;
   }

   return totalTime / (double) numCopiesPerStep;
}

void SetMemBlock(SystemTopo &topo, void *blkPtr, long long numBytes, long long value, MEM_TYPE memType, int devIdx) {
   switch (memType) {
      case PAGE:
      case PINNED:
      case WRITE_COMBINED:
      case MANAGED:
      case MAPPED:
         topo.SetHostMem(blkPtr, value, numBytes);
         break;
      case DEVICE:
         topo.SetDeviceMem(blkPtr, value, numBytes, devIdx);
         break;
      default:
         std::cout << "Error: unrecognized memory set operation type for block set!" << std::endl; 
         break;
   }
}

void SetMemBlocks(SystemTopo &topo, void *destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx, int srcIdx, long long value) {
   switch (copyType) {
      case HOST_HOST_COPY: 
      case HOST_PINNED_HOST_COPY: 
      case HOST_HOST_PINNED_COPY: 
      case HOST_HOST_COPY_PINNED: 
      case HOST_COMBINED_HOST_COPY:
      case HOST_HOST_COMBINED_COPY:
      case HOST_HOST_COPY_COMBINED:
         topo.SetHostMem(srcPtr, value, numBytes);
         topo.SetHostMem(destPtr, value, numBytes);
         break;
      case DEVICE_HOST_COPY:
      case DEVICE_HOST_PINNED_COPY:
      case DEVICE_HOST_COMBINED_COPY:
         topo.SetDeviceMem(srcPtr, value, numBytes, srcIdx);
         topo.SetHostMem(destPtr, value, numBytes);
         break;
      case HOST_DEVICE_COPY:
      case HOST_PINNED_DEVICE_COPY:
      case HOST_COMBINED_DEVICE_COPY:
         topo.SetHostMem(srcPtr, value, numBytes);
         topo.SetDeviceMem(destPtr, value, numBytes, destIdx);
         break;
      case PEER_COPY_NO_UVA: 
      case DEVICE_DEVICE_COPY:
      case COPY_UVA:
         topo.SetDeviceMem(srcPtr, value, numBytes, srcIdx);
         topo.SetDeviceMem(destPtr, value, numBytes, destIdx);
         break;
      default:
         std::cout << "Error: unrecognized memory set operation type for setting blocks!" << std::endl; 
         break;
   }
}

void AllocMemBlock(SystemTopo &topo, void **blkPtr, long long numBytes, MEM_TYPE blockType, int srcIdx, int extIdx) {
   switch (blockType) {
      case PAGE:
         *blkPtr = topo.AllocMemByNode(srcIdx, numBytes);
         break;
      case PINNED:
         *blkPtr = topo.AllocPinMemByNode(srcIdx, numBytes);
         break;
      case WRITE_COMBINED:
         *blkPtr = topo.AllocWCMemByNode(srcIdx, numBytes);
         break;
      case MANAGED:
         *blkPtr = topo.AllocManagedMemByNode(srcIdx, extIdx, numBytes);
         break;
      case MAPPED:
         *blkPtr = topo.AllocMappedMemByNode(srcIdx, extIdx, numBytes);
         break;
      case DEVICE:
         *blkPtr = topo.AllocDeviceMem(srcIdx, numBytes);
         break;
      default:
         std::cout << "Error: unrecognized memory type for allocation!" << std::endl; 
         break;
   }
}

void AllocMemBlocks(SystemTopo &topo, void **destPtr, void **srcPtr, long  long numBytes, MEM_OP copyType, int destIdx, int srcIdx) {
   switch (copyType) {
      case HOST_HOST_COPY: 
         *srcPtr = topo.AllocMemByNode(srcIdx, numBytes);
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         break;
      case HOST_PINNED_HOST_COPY: 
         *srcPtr = topo.AllocPinMemByNode(srcIdx, numBytes);
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         break;
      case HOST_HOST_PINNED_COPY: 
         *srcPtr = topo.AllocMemByNode(srcIdx, numBytes);
         *destPtr = topo.AllocPinMemByNode(destIdx, numBytes);
         break;
      case HOST_HOST_COPY_PINNED: 
         *srcPtr = topo.AllocPinMemByNode(srcIdx, numBytes);
         *destPtr = topo.AllocPinMemByNode(destIdx, numBytes);
         break;
      case HOST_COMBINED_HOST_COPY:
         *srcPtr = topo.AllocWCMemByNode(srcIdx, numBytes);
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         break;
      case HOST_HOST_COMBINED_COPY:
         *srcPtr =topo.AllocMemByNode(srcIdx, numBytes);
         *destPtr = topo.AllocWCMemByNode(destIdx, numBytes);
          break;
      case HOST_HOST_COPY_COMBINED:
         *srcPtr =topo.AllocWCMemByNode(srcIdx, numBytes);
         *destPtr = topo.AllocWCMemByNode(destIdx, numBytes);
         break;
      case DEVICE_HOST_COPY:
         *srcPtr = topo.AllocDeviceMem(srcIdx, numBytes);
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         break;
      case DEVICE_HOST_PINNED_COPY:
         *srcPtr = topo.AllocDeviceMem(srcIdx, numBytes);
         *destPtr = topo.AllocPinMemByNode(destIdx, numBytes);
         break;
      case DEVICE_HOST_COMBINED_COPY:
         *srcPtr = topo.AllocDeviceMem(srcIdx, numBytes);
         *destPtr = topo.AllocWCMemByNode(destIdx, numBytes);
         break;
      case HOST_DEVICE_COPY:
         *srcPtr = topo.AllocMemByNode(srcIdx, numBytes);
         *destPtr = topo.AllocDeviceMem(destIdx, numBytes);
         break;
      case HOST_PINNED_DEVICE_COPY:
         *srcPtr = topo.AllocPinMemByNode(srcIdx, numBytes);
         *destPtr = topo.AllocDeviceMem(destIdx, numBytes);
         break;
      case HOST_COMBINED_DEVICE_COPY:
         *srcPtr = topo.AllocWCMemByNode(srcIdx, numBytes);
         *destPtr = topo.AllocDeviceMem(destIdx, numBytes);
         break;
      case PEER_COPY_NO_UVA: 
      case DEVICE_DEVICE_COPY:
      case COPY_UVA:
         *srcPtr = topo.AllocDeviceMem(srcIdx, numBytes);
         *destPtr = topo.AllocDeviceMem(destIdx, numBytes);
         break;
      default:
         std::cout << "Error: unrecognized memory copy operation type for allocation!" << std::endl;
         break;
   }
}

void MemCopyOp(char * destPtr, char *srcPtr, long stepSize, MEM_OP copyType, int destIdx, int srcIdx, hipStream_t stream) {
   switch (copyType) {
      case HOST_HOST_COPY: 
         memcpy((void *) (destPtr), (void *) (srcPtr), stepSize);
         break;
      case HOST_PINNED_HOST_COPY: 
      case HOST_HOST_PINNED_COPY:
      case HOST_COMBINED_HOST_COPY:
      case HOST_HOST_COMBINED_COPY: 
      case HOST_HOST_COPY_PINNED: 
      case HOST_HOST_COPY_COMBINED:
         checkCudaErrors(hipMemcpy((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyHostToHost));
         break;
      case DEVICE_HOST_COPY:
         checkCudaErrors(hipMemcpy((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDeviceToHost));
         break;
      case DEVICE_HOST_PINNED_COPY:
      case DEVICE_HOST_COMBINED_COPY:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDeviceToHost, stream));
         break;
      case HOST_DEVICE_COPY:
         checkCudaErrors(hipMemcpy((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyHostToDevice));
         break;
      case HOST_PINNED_DEVICE_COPY:
      case HOST_COMBINED_DEVICE_COPY:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyHostToDevice, stream));
         break;
      case PEER_COPY_NO_UVA:
         checkCudaErrors(hipMemcpyPeerAsync((void *) (destPtr), destIdx, (void *) (srcPtr), srcIdx, 0));
         break;
      case DEVICE_DEVICE_COPY:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDeviceToDevice, 0));
         break;
      case COPY_UVA:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDefault, stream));
         break;
      default:
         std::cout << "Error: unrecognized timed memory copy operation type" << std::endl; 
         break;
   }
}

void FreeMemBlocks(SystemTopo &topo, void* destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx, int srcIdx) {
   switch (copyType) {
      case HOST_HOST_COPY: 
         topo.FreeHostMem((void *) destPtr, numBytes);
         topo.FreeHostMem((void *) srcPtr, numBytes);
         break;
      case HOST_PINNED_HOST_COPY:  
         topo.FreePinMem((void *) srcPtr, numBytes);
         topo.FreeHostMem((void *) destPtr, numBytes);
         break;
      case HOST_HOST_PINNED_COPY:  
         topo.FreeHostMem((void *) srcPtr, numBytes);
         topo.FreePinMem((void *) destPtr, numBytes);
         break;
      case HOST_HOST_COPY_PINNED:  
         topo.FreePinMem((void *) srcPtr, numBytes);
         topo.FreePinMem((void *) destPtr, numBytes);
         break;
      case HOST_COMBINED_HOST_COPY:
         topo.FreeWCMem((void *) srcPtr);
         topo.FreeHostMem((void *) destPtr, numBytes);
         break;
      case HOST_HOST_COMBINED_COPY:
         topo.FreeHostMem((void *) srcPtr, numBytes);
         topo.FreeWCMem((void *) destPtr);
         break;
      case HOST_HOST_COPY_COMBINED:
         topo.FreeWCMem((void *) srcPtr);
         topo.FreeWCMem((void *) destPtr);
         break;
      case DEVICE_HOST_COPY:
         topo.FreeDeviceMem(srcPtr, srcIdx);
         topo.FreeHostMem((void *) destPtr, numBytes);
         break;
      case DEVICE_HOST_PINNED_COPY:
         topo.FreeDeviceMem(srcPtr, srcIdx);
         topo.FreePinMem((void *) destPtr, numBytes);
         break;
      case DEVICE_HOST_COMBINED_COPY:
         topo.FreeDeviceMem(srcPtr, srcIdx);
         topo.FreeWCMem((void *) destPtr);
         break;
      case HOST_DEVICE_COPY:
         topo.FreeHostMem((void *) srcPtr, numBytes);
         topo.FreeDeviceMem(destPtr, destIdx);
         break;
      case HOST_PINNED_DEVICE_COPY:
         topo.FreePinMem((void *) srcPtr, numBytes);
         topo.FreeDeviceMem(destPtr, destIdx);
         break;
      case HOST_COMBINED_DEVICE_COPY:
         topo.FreeWCMem((void *) srcPtr);
         topo.FreeDeviceMem(destPtr, destIdx);
         break;
      case PEER_COPY_NO_UVA: 
      case DEVICE_DEVICE_COPY:
      case COPY_UVA:
         topo.FreeDeviceMem(srcPtr, srcIdx);
         topo.FreeDeviceMem(destPtr, destIdx);
         break;
      default:
         std::cout << "Error: unrecognized memory copy operation type for deallocation!" << std::endl; 
         break;
   }
}

float TimedMemOp(void **MemBlk, long long NumBytes, MEM_OP TimedOp) {
   #ifdef USING_CPP
   std::chrono::high_resolution_clock::time_point start_c, stop_c;
   #else
   struct timeval stop_t, start_t, total_t;
   #endif
   
   float OpTime = 0;
   
   #ifdef USING_CPP
   start_c = std::chrono::high_resolution_clock::now();
   #else
   gettimeofday(&start_t, NULL);
   #endif

   switch (TimedOp) {
      case HOST_MALLOC:
         *MemBlk = malloc(NumBytes); 
         break;
      case HOST_PINNED_MALLOC:
         checkCudaErrors(hipHostAlloc(MemBlk, NumBytes, hipHostMallocPortable));
         break;
      case HOST_COMBINED_MALLOC:
         checkCudaErrors(hipHostAlloc(MemBlk, NumBytes, hipHostMallocPortable | hipHostMallocWriteCombined));
         break;
      case MANAGED_MALLOC:
         checkCudaErrors(hipMallocManaged(MemBlk, NumBytes));
         break;
      case MAPPED_MALLOC: 
         checkCudaErrors(hipHostAlloc(MemBlk, NumBytes, hipHostMallocPortable | hipHostMallocMapped));
         break;
      case DEVICE_MALLOC:
         checkCudaErrors(hipMalloc(MemBlk, NumBytes));
         break;
      case HOST_FREE:
         free(*MemBlk);
         break;
      case HOST_PINNED_FREE:
         checkCudaErrors(hipHostFree(*MemBlk));
         break;
      case HOST_COMBINED_FREE:
         checkCudaErrors(hipHostFree(*MemBlk));
         break;
      case MANAGED_FREE:
         checkCudaErrors(hipFree(*MemBlk));
         break;
      case MAPPED_FREE:
         checkCudaErrors(hipHostFree(*MemBlk));
         break;
      case DEVICE_FREE:
         checkCudaErrors(hipFree(*MemBlk)); 
         break;
      default:
         std::cout << "Error: unrecognized timed memory operation type!" << std::endl; 
         break;
   }

   #ifdef USING_CPP
   stop_c = std::chrono::high_resolution_clock::now();
   auto total_c = std::chrono::duration_cast<std::chrono::microseconds>(stop_c - start_c);      
   OpTime = (float) total_c.count();
   #else
   gettimeofday(&stop_t, NULL);
   timersub(&stop_t, &start_t, &total_t);
   OpTime = (float) total_t.tv_usec + (float) total_t.tv_sec * 1.0e6;
   #endif

   return OpTime;
}

int CalcRunSteps(std::vector<long long> &blockSteps, long long startStep, long long stopStep, long long numSteps) {
   int magStart = max((int) log10(startStep), 1);
   int magStop = log10(stopStep);
   long totalSteps = (magStop - magStart) * numSteps;
   long long start = pow(10, magStart);
   long long stop = pow(10, magStop); 
   long long step = start;

   double expStep = ((double) (magStop  - magStart)) / (double) totalSteps;
   double exp = 1.0;

   if (stop == step) {
      blockSteps.push_back(start);      
      totalSteps = 1;
   }

   while (step < stop) {
      step = pow(10, exp);
      blockSteps.push_back(step); 
      exp += expStep;
   }

/*   int magStart = max((int)log10(startStep), 1);
   int magStop = log10(stopStep);

   long long start = pow(10, magStart);
   double stepSize = 10 * start / numSteps;
   long long extra = (stopStep - pow(10, magStop)) / pow(10, magStop) * numSteps;
   long long stop = pow(10, magStop - 1) * (10 + extra); 
   long long rangeSkip = numSteps / start;
   long long totalSteps = (magStop - magStart) * (numSteps - rangeSkip) + extra + 1;  
   double step = start;

   for (long stepNum = 0; stepNum < totalSteps; ++stepNum) { 
      blockSteps.push_back(step);
      
      if ((stepNum) && (stepNum) % (numSteps - rangeSkip) == 0 && (stepSize * numSteps * 10) <= stop) {
         stepSize *= 10.0;
      } 
      
      step += stepSize; 
   }
*/
   return blockSteps.size();
}

void PrintP2PBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData) {
   long long blockSize = params.burstBlockSize;
   int numSockets = params.nSockets;
   std::vector<int> deviceIdxs;
   deviceIdxs.resize(params.nDevices, 0);
   int dataIdx = 0;
   
   int matrixWidth = params.nDevices;
   int matrixHeight = params.nDevices * 4;
   std::cout << "\nDevice-To-Device Unidirectional Memory Transfers:" << std::endl;
   std::cout << "Transfer Block Size: " << blockSize / BYTES_TO_MEGA << " (MB)"<< std::endl;
  
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      std::cout << "\nInitiating Socket: " << socketIdx << std::endl;
      
      std::cout << "-----------------------------------------"; 
      for (int i = 0; i < matrixWidth; i++)
         std::cout << "----------------";
      std::cout << std::endl;

      std::cout << "|\t\t|-----------------------|"; 
      for (int i = 0; i < matrixWidth * 8 - 7; i++)
         std::cout << "-";

      std::cout << " Destination ";
      for (int i = 0; i < matrixWidth * 8 - 7; i++)
         std::cout << "-";
      std::cout << "|" << std::endl;
      
      std::cout << "|\t\t| GPU   | Transfer\t";
      for (int i = 0; i < matrixWidth; i++)
         std::cout << "|---------------";
      std::cout << "|" << std::endl;

      std::cout << "|\t\t|   #   | Type\t\t|";
      for (int i = 0; i < matrixWidth; i++)
         std::cout << "\t" << i << "\t|";
      std::cout << std::endl;

      std::cout << "|---------------|-----------------------"; 
      for (int i = 0; i < matrixWidth; i++)
         std::cout << "|---------------";
      std::cout << "|" << std::endl;


      std::cout << std::setprecision(2) << std::fixed;          
      
      std::fill(deviceIdxs.begin(), deviceIdxs.end(), 0);
      for (int i = 0; i < matrixHeight; ++i) {

         std::cout << "|\t\t|  " << i  / 4 <<  "\t|";
         if (i % 4 == 0) {
            std::cout << " Standard D2D\t|";
         } else if (i % 4 == 1) {
            std::cout << " Peer, No UVA\t|";
         } else if (i % 4 == 2) {
            std::cout << " No Peer, UVA\t|";
         } else { 
            std::cout << " Peer, UVA\t|";
         }
         
         if (i % 4 == 0) {
            //deviceIdxs.resize(matrixWidth, 0);
            //deviceIdxs.assign(deviceIdxs.begin(), deviceIdxs.end(), 0);
            std::fill(deviceIdxs.begin(), deviceIdxs.end(), 0);
         }
         dataIdx = 0;
         for (int j = 0; j < matrixWidth; ++j) {
            if (i % 4 == 0) {
               std::cout << "      " << burstData[socketIdx * matrixWidth + i / 4][dataIdx + deviceIdxs[j]] << "\t|";
               deviceIdxs[j]++;
            } else if ((i % 4 == 1) && topo.DeviceGroupCanP2P(i / 4, j)) {
               std::cout << "      " << burstData[socketIdx * matrixWidth + i / 4][dataIdx + deviceIdxs[j]] << "\t|";
               deviceIdxs[j]++;
            } else if ((i % 4 == 2) && topo.DeviceGroupUVA(i / 4, j)) {
               std::cout << "      " << burstData[socketIdx * matrixWidth + i / 4][dataIdx + deviceIdxs[j]] << "\t|";
               deviceIdxs[j]++;
            } else if ((i % 4 == 3) && topo.DeviceGroupUVA(i / 4, j) && topo.DeviceGroupCanP2P(i / 4, j)) { 
               std::cout << "      " << burstData[socketIdx * matrixWidth + i / 4][dataIdx + deviceIdxs[j]] << "\t|";
               deviceIdxs[j]++;
            } else { 
               std::cout << "\t-\t|";
            }

            dataIdx++;
            if (topo.DeviceGroupCanP2P(i / 4, j))
               dataIdx++;
            if (topo.DeviceGroupUVA(i / 4, j)) {
               dataIdx++;
               if (topo.DeviceGroupCanP2P(i / 4, j)) 
                  dataIdx++;
            }
         }
         
         std::cout << std::endl;
         
         if (i + 1 < matrixHeight && (i + 1 == ((float) matrixHeight / 2.0))) {
            std::cout << "|   Source\t|-----------------------";
            for (int i = 0; i < matrixWidth; i++)
               std::cout << "|---------------";
            std::cout << "|" << std::endl;
         } else if (i + 1 < matrixHeight && (i + 1) % 4  ==  0) {
            std::cout << "|\t\t|-----------------------";
            for (int i = 0; i < matrixWidth; i++)
               std::cout << "|---------------";
            std::cout << "|" << std::endl;
         }
      }
      std::cout << std::setprecision(4) << std::fixed;          
      
      std::cout << "-----------------------------------------"; 
      for (int i = 0; i < matrixWidth; i++)
         std::cout << "----------------";
      std::cout << std::endl;
   }
}

void PrintHDBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData) {
   long long blockSize = params.burstBlockSize;
   int numSockets = params.nSockets;
   
   int numPatterns = 1;
   if (params.runPatternsHD)
      numPatterns = NUM_PATTERNS;
   
   int matrixWidth = topo.NumNodes();
   int matrixHeight = params.nDevices;
   
   std::cout << "\nHost/Device Unidirectional Memory Transfers:" << std::endl;
   std::cout << "Transfer Block Size: " << blockSize / BYTES_TO_MEGA << " (MB)"<< std::endl;
   std::cout << "Num Patterns: " << numPatterns << std::endl;

   std::cout << std::setprecision(2) << std::fixed;          
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      std::cout << "\nInitiating Socket: " << socketIdx << std::endl;
      
      for (int patternNum = 0; patternNum < numPatterns; patternNum++) {
         std::cout << "Memory Access Pattern: " <<  PatternNames[patternNum] << std::endl;   
   
         std::cout << "-------------------------"; 
         for (int i = 0; i < matrixWidth * 2; i++)
            std::cout << "----------------";
         std::cout << std::endl;

         std::cout << "|\t\t\t|"; 
         for (int i = 0; i < matrixWidth * 16 - 6; i++)
            std::cout << "-";

         std::cout << " Host CPU ";
         for (int i = 0; i < matrixWidth * 16 - 5; i++)
            std::cout << "-";
         std::cout << "|" << std::endl;

         std::cout << "|\t\t\t|";
         for (int i = 0; i < matrixWidth; i++)
            std::cout << "\t\t" << i << "\t\t|";
         std::cout << std::endl;

         std::cout << "|\t\t\t|"; 
         for (int i = 0; i < matrixWidth * 2; i++){
            if (i + 1 < matrixWidth * 2)
               std::cout << "----------------";
            else 
               std::cout << "---------------";
         }
         std::cout << "|" << std::endl;
       
         std::cout << "|\t\t\t";
         for (int i = 0; i < matrixWidth; i++)
            std::cout << "| Host-2-Device | Device-2-Host ";
         std::cout << "|" << std::endl;
           
         std::cout << "|\t       Transfer\t|";
         for (int i = 0; i < matrixWidth * 2; i++){
            if (i + 1 < matrixWidth * 2)
               std::cout << "----------------";
            else 
               std::cout << "---------------";
         }
         std::cout << "|" << std::endl;

         std::cout << "|\t\t  Type\t";
         for (int i = 0; i < matrixWidth * 2; i++) {
            std::cout << "| Page\t";
            std::cout << "|  Pin\t";
         }
         std::cout << "|" << std::endl;

         std::cout << "|-----------------------"; 
         for (int i = 0; i < matrixWidth * 2; i++)
            std::cout << "----------------";
         std::cout << "|" << std::endl;
         
         std::cout << std::setprecision(2) << std::fixed;          
         for (int i = 0; i < matrixHeight; ++i) {

            std::cout << "|\t\t|  " << i <<  "\t|";
            int rowIdx = socketIdx * numPatterns + patternNum;
            for (int j = 0; j < matrixWidth; ++j) {
                  int colIdx = j * topo.NumGPUs() * 4 + i * 4;
                  std::cout << " " << burstData[rowIdx][colIdx + 0] << "\t|";
                  std::cout << " " << burstData[rowIdx][colIdx + 2] << "\t|";
                  std::cout << " " << burstData[rowIdx][colIdx + 1] << "\t|";
                  std::cout << " " << burstData[rowIdx][colIdx + 3] << "\t|";
            }
            std::cout << std::endl;
            
            if (i + 1 < matrixHeight && (i + 1 == ((float) matrixHeight / 2.0))) {
               std::cout << "|     Device\t|-------";
               for (int i = 0; i < matrixWidth * 2; i++)
                  std::cout << "----------------";
               std::cout << "|" << std::endl;
            } else if (i + 1 < matrixHeight) {
               std::cout << "|\t\t|-------";
               for (int i = 0; i < matrixWidth * 2; i++)
                  std::cout << "----------------";
               std::cout << "|" << std::endl;
            }
         }
         std::cout << std::setprecision(4) << std::fixed;          

         std::cout << "-------------------------"; 
         for (int i = 0; i < matrixWidth * 2; i++)
            std::cout << "----------------";
         std::cout << std::endl;
      }
   }
}

void PrintHHBurstMatrix(BenchParams &params, SystemTopo &topo, std::vector<std::vector<float> > &burstData) {
   long long blockSize = params.burstBlockSize;
   int numSockets = params.nSockets;

   int numPatterns = 1;
   if (params.runPatternsHH)
      numPatterns = NUM_PATTERNS;
   int nodeWidth = pow(HOST_MEM_TYPES * topo.NumNodes(), 2) / topo.NumNodes();

   int matrixWidth = HOST_MEM_TYPES * topo.NumNodes();
   int matrixHeight = HOST_MEM_TYPES * topo.NumNodes();
   
   std::cout << "\nHost-Host Multi-NUMA Unidirectional Memory Transfers:" << std::endl;
   std::cout << "Transfer Block Size: " << blockSize / BYTES_TO_MEGA << " (MB)"<< std::endl;
   std::cout << "Num Patterns: " << numPatterns << std::endl;

   std::cout << std::setprecision(2) << std::fixed;          
   for (int socketIdx = 0; socketIdx < numSockets; socketIdx++) {
      std::cout << "\nInitiating Socket: " << socketIdx << std::endl;
      
      for (int patternNum = 0; patternNum < numPatterns; patternNum++) {
         std::cout << "Memory Access Pattern: " <<  PatternNames[patternNum] << std::endl;   
         
         std::cout << "---------------------------------"; 
         for (int i = 0; i < matrixWidth; i++)
            std::cout << "----------------";
         std::cout << std::endl;

         std::cout << "|\t\t|----------------"; 
         for (int i = 0; i < matrixWidth * 8 - 7; i++)
            std::cout << "-";

         std::cout << " Destination ";
         for (int i = 0; i < matrixWidth * 8 - 7; i++)
            std::cout << "-";
         std::cout << "|" << std::endl;

         std::cout << "|   Transfer \t|---------------";// << std::endl;
         for (int i = 0; i < matrixWidth; i++)
            std::cout << "----------------";
         std::cout << "|" << std::endl;

         std::cout << "|   Point\t| NUMA \t\t|";
         for (int i = 0; i < topo.NumNodes(); i++)
            std::cout << "\t\t" << i << "\t\t|";
         std::cout << "" << std::endl;
         
         std::cout << "|\t\t| Node \t\t|";
         for (int i = 0; i < matrixWidth; i++) {
            if (i + 1 < matrixWidth)
               std::cout << "----------------";
            else 
               std::cout << "---------------";
         }
         std::cout << "|" << std::endl;
    
         std::cout << "|\t\t| #     Mem Type";
         for (int i = 0; i < matrixWidth; i++){
            if (i % 2)
               std::cout << "|    Pinned\t";
            else
               std::cout << "|    Pageable\t";
         }
         std::cout << "|"<< std::endl;
    
         std::cout << "|-------------------------------"; 
         for (int i = 0; i < matrixWidth; i++)
            std::cout << "----------------";
         std::cout << "|" << std::endl;
        
         for (int i = 0; i < matrixHeight; ++i) {
            std::cout << "|\t\t|\t|";//<< std::endl;
            for (int j = 0; j < matrixWidth + 1; ++j)
               std::cout << "\t|\t";
            std::cout << std::endl; 

            std::cout << "|\t\t| " << i / (matrixHeight / topo.NumNodes()) <<  "\t|";
            if (i % 2)
               std::cout << " Pin\t|    ";
            else
               std::cout << " Page\t|    ";
       
            int rowIdx = socketIdx * numPatterns + patternNum;
            for (int j = 0; j < matrixWidth; ++j) {
               int colIdx = (i / HOST_MEM_TYPES * nodeWidth) + j * HOST_MEM_TYPES + i % HOST_MEM_TYPES;
               std::cout << burstData[rowIdx][colIdx] << "\t|    ";
            }
                
            std::cout << "\n|\t\t|\t|";
            for (int j = 0; j < matrixWidth + 1; ++j)
               std::cout << "\t|\t";
            std::cout << std::endl;
            
            if (i + 1 < matrixHeight && (i + 1 != ((float) matrixHeight / 2.0))) {
               std::cout << "|\t\t|-------|-------|";
               for (int i = 0; i < matrixWidth; i++) {
                  if (i + 1 < matrixWidth)
                     std::cout << "----------------";
                  else 
                     std::cout << "---------------";
               }
               std::cout << "|" << std::endl; 
            } else if (i + 1 < matrixHeight) {
               std::cout << "|    Source     |-------|-------|";
               for (int i = 0; i < matrixWidth; i++) {
                  if (i + 1 < matrixWidth)
                     std::cout << "----------------";
                  else 
                     std::cout << "---------------";
               }
               std::cout << "|" << std::endl; 
            }
         }

         std::cout << "---------------------------------"; 
         for (int i = 0; i < matrixWidth; i++)
            std::cout << "----------------";
         std::cout << std::endl;
      }
      std::cout << std::setprecision(2) << std::fixed;          
   }
}

void PrintRangedHeader(BenchParams &params, SystemTopo &topo, std::ofstream &fileStream, BW_RANGED_TYPE testType) {

   std::vector<std::vector<int> > peerGroups;// = topo.GetPeerGroups();
   switch (testType) {
      case HH: 
         if (!params.runSocketTests) 
            fileStream << "0,";
         else
            fileStream << topo.NumSockets() << ",";

         fileStream << topo.NumNodes();
         if (params.testAllMemTypes)
            fileStream << ",t";
         else 
           fileStream  << ",f";

         if (params.runSocketTests) 
            fileStream << ",t";
         else
            fileStream << ",f";

         fileStream << ",Repeated";
         if (params.runPatternsHD) {
            fileStream << ",Linear Inc";
            fileStream << ",Linear Dec";
         }
         fileStream << std::endl;
         break;
      case HD:
         if (!params.runSocketTests) 
            fileStream << "0,";
         else
            fileStream << topo.NumSockets() << ",";

         fileStream << topo.NumNodes() << ",";
         fileStream << params.nDevices;
         if (params.testAllMemTypes)
            fileStream << ",t";
         else 
           fileStream  << ",f";

         if (params.runSocketTests) 
            fileStream << ",t";
         else
            fileStream << ",f";

         for (int i = 0; i < params.nDevices; i++) {
            fileStream << "," << topo.GetDeviceName(i);
         }

         fileStream << ",Repeated";
         if (params.runPatternsHD) {
            fileStream << ",Linear Inc";
            fileStream << ",Linear Dec";
         }

         fileStream << std::endl;
         break;
      case P2P:
         if (!params.runSocketTests) 
            fileStream << "0,";
         else
            fileStream << topo.NumSockets() << ",";

         fileStream << params.nDevices;
         fileStream << "," << topo.NumPeerGroups();
         
         if (params.runSocketTests) 
            fileStream << ",t";
         else
            fileStream << ",f";

         for (int i = 0; i < params.nDevices; i++) {
            fileStream << "," << topo.GetDeviceName(i);
         }
 
         for (int i = 0; i < params.nDevices; i++) {
            fileStream << "," << std::boolalpha << topo.DeviceUVA(i) << std::noboolalpha;
         }
         
         fileStream << std::endl;
         peerGroups = topo.GetPeerGroups();
         for (int i = 0; i < peerGroups.size(); i++) {
            for (int j = 0; j < peerGroups[i].size(); j++) {
               fileStream << peerGroups[i][j];
               if (j + 1 < peerGroups[i].size()) {
                  fileStream << ",";
               }
            }
            fileStream << std::endl;
         }
         break;
      default:
         std::cout << "Error: unrecognized ranged transfer test type!" << std::endl; 
         break;

   }
}

void PrintResults(std::ofstream &outFile, std::vector<long long> &steps, std::vector<std::vector<float> > &results) {
   
   if (!outFile.is_open()) {
      std::cout << "Failed to open file to print results" << std::endl;
      return;
   }
   std::vector<std::vector<float> >::iterator iter_o;
   std::vector<float>::iterator iter_i;
   std::vector<long long>::iterator iter_l = steps.begin();
   
   for (iter_o = results.begin(); iter_o != results.end(); ++iter_o) {
      outFile << std::fixed << *iter_l++ << ",";
      for (iter_i = (*iter_o).begin(); iter_i != (*iter_o).end(); ++iter_i) {
         outFile << std::fixed << *iter_i;
         if (iter_i + 1 != (*iter_o).end())
            outFile << ",";
      }
      outFile << std::endl;
   }
}


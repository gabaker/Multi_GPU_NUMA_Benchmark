#include "hip/hip_runtime.h"

// Benchmark includes and defines
#ifndef BENCH_HEADER_INC
#define BENCH_HEADER_INC
#include "benchmark.h"
#endif

// BenchParams class definition
#ifndef PARAM_CLASS_INC
#include "parameters.h"
#define PARAM_CLASS_INC
#endif

// SystemTopo class definition
#ifndef TOPOLOGY_CLASS_INC
#include "topology.h"
#define TOPOLOGY_CLASS_INC
#endif

// Benchmark Tests
void RunTestSuite(BenchParams &params, SystemTopo &topo);
void TestMemoryOverhead(hipDeviceProp_t *props, BenchParams &params, SystemTopo &topo);
void TestHostDeviceBandwidth(hipDeviceProp_t *props, BenchParams &params, SystemTopo &topo);
void TestP2PDeviceBandwidth(hipDeviceProp_t *props, BenchParams &params, SystemTopo &topo);
void TestPCIeCongestion(hipDeviceProp_t *props, BenchParams &params, SystemTopo &topo);
void TestTaskScalability(hipDeviceProp_t *props, BenchParams &params, SystemTopo &topo);

// Test Subfunctions
void MemCopyRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, MEM_OP copyType, MEM_PATTERN patternType, int destIdx, int srcIdx); 
//void MemCopyRun(BenchParams &params, char *destPtr, char *srcPtr, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, MEM_OP copyType, MEM_PATTERN patternType, int destIdx = 0, int srcIdx = 0);
float TimedMemOp(void **MemBlk, long long NumBytes, MEM_OP TimedOp); 
float TimedMemCopyStep(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, MEM_PATTERN patternType, int destIdx = 0, int srcIdx = 0);
void MemCopyOp(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, int destIdx = 0, int srcIdx = 0);

// Support functions
void AllocateMemBlock(SystemTopo &topo, void **destPtr, void **srcPtr, long long numBytes, MEM_OP copyType, int destIdx = 0, int srcIdx = 0);
void FreeMemBlock(SystemTopo &topo, void* destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx = 0, int srcIdx = 0);
int CalcRunSteps(std::vector<long long> &blockSteps, long long startStep, long long stopStep, long long numSteps); 

// Device Properties
void GetAllDeviceProps(hipDeviceProp_t *props, int dCount);
void PrintDeviceProps(hipDeviceProp_t *props, BenchParams &params);
void ResetDevices(int numToReset);

// Results output
void PrintResults(std::ofstream &outFile, std::vector<long long> &steps, std::vector<std::vector<float> > &results, BenchParams &params);

/* Benchmark main()
 * 
 * 
 */

int main (int argc, char **argv) {
   BenchParams params;  
   SystemTopo topo;
   
   std::cout << "Starting Multi-GPU Performance Test Suite...\n" << std::endl; 
   
   // Determine the number of recognized CUDA enabled devices
   checkCudaErrors(hipGetDeviceCount(&(params.nDevices)));

   // Exit if system contains no devices
   if (params.nDevices <= 0) {
      std::cout << "No devices found...aborting benchmarks." << std::endl;
      exit(-1);
   }

   // Setup benchmark parameters
   if (argc == 1) { 
      // No input file, use default parameters
      params.SetDefault();
   
   } else if (argc == 2) {       
      // Parse input file and set parameter class local variables
      params.ParseParamFile(std::string(argv[1]));

   } else {
      // Unknown input parameter list, abort test
      std::cout << "Aborting test: Incorrect number of input parameters" << std::endl;
      exit(-1);
   }

   // Print HWLOC topology info
   // Class constructor parses system topology from device files (linux)
   std::string topoFileName ="./results/topology.out";
   std::ofstream topoFile(topoFileName.c_str());
   topo.PrintTopology(topoFile);

   // Print device parameters for user/script parsing
   params.PrintParams();

   // Run the benchmark per parameters defines in params
   RunTestSuite(params, topo);

   return 0;
}

void RunTestSuite(BenchParams &params, SystemTopo &topo) {
   hipDeviceProp_t *props = (hipDeviceProp_t *) calloc (sizeof(hipDeviceProp_t), params.nDevices);

   // Aquire device properties for each CUDA enabled GPU
   GetAllDeviceProps(props, params.nDevices);

   if (params.runMemoryOverheadTest) {
      
      TestMemoryOverhead(props, params, topo);
   
   }

   if (params.runHDBandwidthTest) {

      TestHostDeviceBandwidth(props, params, topo);

   }

   if (params.runP2PBandwidthTest) {  
      
      TestP2PDeviceBandwidth(props, params, topo);
   
   }

   if (params.runPCIeCongestionTest) {

      TestPCIeCongestion(props, params, topo);

   }

   if (params.runTaskScalabilityTest) { 

      TestTaskScalability(props, params, topo);

   }

   // Output device properties for each CUDA enabled GPU
   if (params.printDevProps) {
      PrintDeviceProps(props, params);
   }

   std::cout << "\nBenchmarks complete!\n" << std::endl;

   free(props);

   return;
}

void TestMemoryOverhead(hipDeviceProp_t *props, BenchParams &params, SystemTopo &topo) {
   char *deviceMem = NULL;
   char *hostMem = NULL;
   char *hostPinnedMem = NULL;
   int nDevices = params.nDevices;
   long long chunkSize = 0;

   // Only run overhead device cases on a single device
   // default to device 0
   if (!params.runAllDevices)
      nDevices = 1;

   std::vector<long long> blockSteps;
   CalcRunSteps(blockSteps, params.rangeMemOverhead[0], params.rangeMemOverhead[1], params.rangeMemOverhead[2]);  
   std::vector<std::vector<float> > overheadData;
   overheadData.resize(blockSteps.size());
   
   // Memory overhead test will run for each device utilizing the hipMalloc and hipFree functions
   // on the first iteration of the look, assuming there is atleast one device, the host will run the 
   // pinned and un-pinned memory tests
   for (int numaIdx = 0; numaIdx < topo.NumNodes(); numaIdx++) { 
      topo.PinNumaNode(numaIdx);
      
      for (int socketIdx = 0; socketIdx < topo.NumSockets(); socketIdx++) {
         topo.PinSocket(socketIdx);
         
         // Host based management for CASE 1 & 2
         for (long stepIdx = 0; stepIdx < blockSteps.size(); stepIdx++) {
            chunkSize = blockSteps[stepIdx];
            float pinAllocTime = 0, pinFreeTime = 0, hostAllocTime = 0, hostFreeTime = 0;
            
            // repeat same block run and average times
            for (int reIdx = 0; reIdx < params.numStepRepeatsOH; reIdx++) {
               if (params.usePinnedMem) {
                  //CASE 1: Host Pinned Memory Overhead
                  pinAllocTime += TimedMemOp((void **) &hostPinnedMem, chunkSize, HOST_PINNED_MALLOC);
                  pinFreeTime += TimedMemOp((void **) &hostPinnedMem, 0, HOST_PINNED_FREE); 
               }
               //CASE 2: Host UnPinned Memory Overhead
               hostAllocTime += TimedMemOp((void **) &hostMem, 0, HOST_FREE);
               hostFreeTime += TimedMemOp((void **) &hostMem, chunkSize, HOST_MALLOC);
            }

            overheadData[stepIdx].push_back(pinAllocTime / (float) params.numStepRepeatsOH);
            overheadData[stepIdx].push_back(pinFreeTime / (float) params.numStepRepeatsOH);
            overheadData[stepIdx].push_back(hostAllocTime / (float) params.numStepRepeatsOH);
            overheadData[stepIdx].push_back(hostFreeTime / (float) params.numStepRepeatsOH);
         }
         
         // Device based memory management for CASE 3 & 4
         for (int currDev = 0; currDev < nDevices; currDev++) {
            checkCudaErrors(hipSetDevice(currDev)); 

            for (long stepIdx = 0; stepIdx < blockSteps.size(); stepIdx++) {
               chunkSize = blockSteps[stepIdx];
               float devAllocTime = 0, devFreeTime = 0;

               // repeat same block run and average times
               for (int reIdx = 0; reIdx < params.numStepRepeatsOH; reIdx++) {
                  // CASE 3: Allocation of device memory  
                  devAllocTime += TimedMemOp((void **) &deviceMem, chunkSize, DEVICE_MALLOC);
                  // CASE 4: DeAllocation of device memory 
                  devFreeTime += TimedMemOp((void **) &deviceMem, 0, DEVICE_FREE);
               }

               overheadData[stepIdx].push_back(devAllocTime / (float) params.numStepRepeatsOH);
               overheadData[stepIdx].push_back(devFreeTime / (float) params.numStepRepeatsOH);
            }
         }
      }
   }
   std::string dataFileName = "./results/" + params.resultsFile + "_overhead.csv";
   std::ofstream overheadResultsFile(dataFileName.c_str());
   PrintResults(overheadResultsFile, blockSteps, overheadData, params);

   return;
}

void TestHostDeviceBandwidth(hipDeviceProp_t *props, BenchParams &params, SystemTopo &topo) {
   std::cout << "Running host-device bandwidth test" << std::endl;

   params.numCopiesPerStepHD = 20;
   
   if (params.runSustainedHD == false) {
      params.numCopiesPerStepHD = 1;
   }

   std::vector<std::vector<float> > bandwidthData;
   std::vector<long long> blockSteps;
   CalcRunSteps(blockSteps, params.rangeHostDeviceBW[0], params.rangeHostDeviceBW[1], params.rangeHostDeviceBW[2]); 
   bandwidthData.resize(blockSteps.size());

   for (int socketIdx = 0; socketIdx < topo.NumSockets(); socketIdx++) {
      topo.PinSocket(socketIdx);
 
      for (int numaSrc = 0; numaSrc < topo.NumNodes(); numaSrc++) { 
         topo.PinNumaNode(numaSrc);

         //Host To Host Memory Transfers
         for (int numaDest = 0; numaDest < topo.NumNodes(); numaDest++) { 
            // HtoH Ranged Transfer - Pageable Memory
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_HOST_COPY, REPEATED, numaDest, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_HOST_COPY, RANDOM, numaDest, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_HOST_COPY, LINEAR_INC, numaDest, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_HOST_COPY, LINEAR_DEC, numaDest, numaSrc); 

            //HtoH Ranged Transfer - Pinned Memory
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, REPEATED, numaDest, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, RANDOM, numaDest, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, LINEAR_INC, numaDest, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_HOST_COPY_PINNED, LINEAR_DEC, numaDest, numaSrc); 

         }

         //Host-Device PCIe Memory Transfers
/*         for (int currDev = 0; currDev < params.nDevices; currDev++) {
            checkCudaErrors(hipSetDevice(currDev));

            // HtoD Ranged Transfer - Pageable Memory
            MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, REPEATED, currDev, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, RANDOM, currDev, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, LINEAR_INC, currDev, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_DEVICE_COPY, LINEAR_DEC, currDev, numaSrc); 

            // DtoH Ranged Transfer - Pageable Memory
            MemCopyRun(params, topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, REPEATED, currDev, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, RANDOM, currDev, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, LINEAR_INC, currDev, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, DEVICE_HOST_COPY, LINEAR_DEC, currDev, numaSrc); 

            // HtoD Ranged Transfer - Pinned Memory
            MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_DEVICE_COPY_PINNED, REPEATED, currDev, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_DEVICE_COPY_PINNED, RANDOM, currDev, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_DEVICE_COPY_PINNED, LINEAR_INC, currDev, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, HOST_DEVICE_COPY_PINNED, LINEAR_DEC, currDev, numaSrc); 

            // DtoH Ranged Transfer - Pinned Memory
            MemCopyRun(params, topo, blockSteps, bandwidthData, DEVICE_HOST_COPY_PINNED, REPEATED, currDev, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, DEVICE_HOST_COPY_PINNED, RANDOM, currDev, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, DEVICE_HOST_COPY_PINNED, LINEAR_INC, currDev, numaSrc); 
            //MemCopyRun(params, topo, blockSteps, bandwidthData, DEVICE_HOST_COPY_PINNED, LINEAR_DEC, currDev, numaSrc); 
         }
*/
      }
   }

   std::string dataFileName = "./results/" + params.resultsFile + "_bandwidth.csv";
   std::ofstream bandwidthResultsFile(dataFileName.c_str());
   //PrintResults(bandwidthResultsFile, blockSteps, bandwidthData, params);

   return;
}

void TestP2PDeviceBandwidth(hipDeviceProp_t *props, BenchParams &params, SystemTopo &topo){
   std::cout << "Running P2P device bandwidth test" << std::endl;

   //Device to Device transfers
   /*for (int srcDev = 0; currDev < params.nDevices; currDev++) {
      checkCudaErrors(hipSetDevice(currDev));
      for (int destDev = 0; currDev < nDevices; currDev++) {
         checkCudaErrors(hipSetDevice(currDev));  
      
         //must support p2p to allow direct transfer
         if (srcDev != destDev) {
            
         }
      }
   } */     
   return;
}

void TestPCIeCongestion(hipDeviceProp_t *props, BenchParams &params, SystemTopo &topo) {
   std::cout << "Running PCIe congestion test" << std::endl;
   return;
}

void TestTaskScalability(hipDeviceProp_t *props, BenchParams &params, SystemTopo &topo) {
   std::cout << "Running task scalability test" << std::endl;
   return;
}

void MemCopyRun(BenchParams &params, SystemTopo &topo, std::vector<long long> &blockSteps, std::vector<std::vector<float> > &bandwidthData, MEM_OP copyType, MEM_PATTERN patternType, int destIdx, int srcIdx) {
   char *destPtr, *srcPtr; 
   long totalSteps = blockSteps.size();
   
   std::vector<float> timedRun(totalSteps, 0.0);
   long long blockSize = blockSteps[totalSteps - 1 ];

   AllocateMemBlock(topo, (void **) &destPtr, (void **) &srcPtr, blockSize, copyType, destIdx, srcIdx);
   FreeMemBlock(topo, (void *) destPtr, (void *) srcPtr, blockSize, copyType, destIdx, srcIdx);

  
   for (long stepNum = 0; stepNum < totalSteps; ++stepNum) { 

      bandwidthData[stepNum].push_back(TimedMemCopyStep((char *) destPtr, (char *) srcPtr, blockSteps[stepNum], blockSize, params.numCopiesPerStepHD, copyType, patternType, destIdx, srcIdx));

   }
}

float TimedMemCopyStep(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, MEM_PATTERN patternType, int destIdx, int srcIdx) {
   long long offset = 0;
   float totalTime = 0; 

   #ifdef USING_CPP
   std::chrono::high_resolution_clock::time_point start_c, stop_c;
   auto total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
   #else
   struct timeval stop_t, start_t, total_t;
   #endif
   
   hipEvent_t start_e, stop_e; 
   checkCudaErrors(hipEventCreate(&start_e));
   checkCudaErrors(hipEventCreate(&stop_e)); 

   if (HOST_HOST_COPY) {
      #ifdef USING_CPP
      start_c = std::chrono::high_resolution_clock::now();
      #else
      gettimeofday(&start_t, NULL);
      #endif
   } else{
      checkCudaErrors(hipEventRecord(start_e, 0));
   }

   for (int copyIdx = 0; copyIdx < numCopiesPerStep; copyIdx++) {

      MemCopyOp(destPtr + offset, srcPtr + offset, stepSize, blockSize, numCopiesPerStep, copyType, destIdx, srcIdx); 

      //TODO: add options to change 
      if (numCopiesPerStep > 1 && MAX_PATTERN_SIZE) {
         switch (patternType) {
       
           case REPEATED:
               offset = 0;
               break;
            case RANDOM:
               break;
            case PERIODIC:
               break;
            case LINEAR_INC:
               break;
            case LINEAR_DEC:
               break;
            default: //BURST
               break;
         }
      }
   }

   if (HOST_HOST_COPY) {
      #ifdef USING_CPP
      stop_c = std::chrono::high_resolution_clock::now(); 
      total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
      totalTime = (float) total_c.count() * NANO_TO_MILLI; 
      #else
      gettimeofday(&stop_t, NULL); 
      timersub(&stop_t, &start_t, &total_t); 
      totalTime = (float) total_t.tv_usec * MICRO_TO_MILLI;
      #endif
   } else{
      checkCudaErrors(hipEventRecord(stop_e, 0));
      checkCudaErrors(hipEventSynchronize(stop_e));   
      checkCudaErrors(hipEventElapsedTime(&totalTime, start_e, stop_e));  
   }

   return totalTime;
}

void MemCopyOp(char * destPtr, char *srcPtr, long stepSize, long long blockSize, int numCopiesPerStep, MEM_OP copyType, int destIdx, int srcIdx) {
   switch (copyType) {
      case HOST_HOST_COPY:  
         memcpy((void *) (destPtr), (void *) (srcPtr), stepSize);
         break;
      case HOST_HOST_COPY_PINNED:  
         checkCudaErrors(hipMemcpyAsync((void *)(destPtr), (void *) (srcPtr), stepSize, hipMemcpyHostToHost, 0));
         break;
      case DEVICE_HOST_COPY:
         checkCudaErrors(hipMemcpy((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDeviceToHost));
         break;
      case DEVICE_HOST_COPY_PINNED:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDeviceToHost, 0));
         break;
      case HOST_DEVICE_COPY:
         checkCudaErrors(hipMemcpy((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyHostToDevice));
         break;
      case HOST_DEVICE_COPY_PINNED:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyHostToDevice, 0));
         break;
      case PEER_COPY_NO_UVA:
         checkCudaErrors(hipMemcpyPeerAsync((void *) (destPtr), destIdx, (void *) (srcPtr), srcIdx, 0));
         break;
      case DEVICE_DEVICE_COPY:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDeviceToDevice));
         break;
      case COPY_UVA:
         checkCudaErrors(hipMemcpyAsync((void *) (destPtr), (void *) (srcPtr), stepSize, hipMemcpyDefault, 0));
         break;
      default:
         std::cout << "Error: unrecognized timed memory copy operation type" << std::endl; 
         break;
   }
}

void FreeMemBlock(SystemTopo &topo, void* destPtr, void *srcPtr, long long numBytes, MEM_OP copyType, int destIdx, int srcIdx) {
   switch (copyType) {
      case HOST_HOST_COPY: 
         topo.FreeMem((void *) destPtr, numBytes);
         topo.FreeMem((void *) srcPtr, numBytes);
         break;
      case HOST_HOST_COPY_PINNED:  
         checkCudaErrors(hipHostUnregister((void*) srcPtr));
         topo.FreeMem((void *) srcPtr, numBytes);
         checkCudaErrors(hipHostUnregister((void*) destPtr));
         topo.FreeMem((void *) destPtr, numBytes);
         break;
      case DEVICE_HOST_COPY:
         checkCudaErrors(hipSetDevice(srcIdx));
         checkCudaErrors(hipFree((void *) srcPtr));
         topo.FreeMem((void *) destPtr, numBytes);
         break;
      case DEVICE_HOST_COPY_PINNED:
         checkCudaErrors(hipSetDevice(srcIdx));
         checkCudaErrors(hipFree((void *) srcPtr));
         checkCudaErrors(hipHostUnregister((void*) destPtr));
         topo.FreeMem((void *) destPtr, numBytes);
         break;
      case HOST_DEVICE_COPY:
         topo.FreeMem((void *) srcPtr, numBytes);
         checkCudaErrors(hipSetDevice(destIdx));
         checkCudaErrors(hipFree((void *) destPtr));
         break;
      case HOST_DEVICE_COPY_PINNED:
         checkCudaErrors(hipHostUnregister((void *) srcPtr));
         topo.FreeMem((void *) srcPtr, numBytes);
         checkCudaErrors(hipSetDevice(destIdx));
         checkCudaErrors(hipFree((void *) destPtr));
         break;
      case PEER_COPY_NO_UVA: 
         checkCudaErrors(hipSetDevice(srcIdx));
         checkCudaErrors(hipFree((void *) srcPtr));
         checkCudaErrors(hipSetDevice(destIdx));
         checkCudaErrors(hipFree((void *) destPtr));
         break;
      case DEVICE_DEVICE_COPY:
         checkCudaErrors(hipSetDevice(srcIdx));
         checkCudaErrors(hipFree((void *) srcPtr));
         checkCudaErrors(hipSetDevice(destIdx));
         checkCudaErrors(hipFree((void *) destPtr));
         break;
      case COPY_UVA:
         checkCudaErrors(hipSetDevice(srcIdx));
         checkCudaErrors(hipFree((void *) srcPtr));
         checkCudaErrors(hipSetDevice(destIdx));
         checkCudaErrors(hipFree((void *) destPtr));
         break;
      default:
         std::cout << "Error: unrecognized memory copy operation type for deallocation" << std::endl; 
         break;
   }
}

void AllocateMemBlock(SystemTopo &topo, void **destPtr, void **srcPtr,long  long numBytes, MEM_OP copyType, int destIdx, int srcIdx) {
   switch (copyType) {

      case HOST_HOST_COPY: 
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         *srcPtr =topo.AllocMemByNode(srcIdx, numBytes);
         break;
      case HOST_HOST_COPY_PINNED:  
         *srcPtr =topo.AllocMemByNode(srcIdx, numBytes);
         checkCudaErrors(hipHostRegister(*srcPtr, numBytes, hipHostRegisterPortable));
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         checkCudaErrors(hipHostRegister(*destPtr, numBytes, hipHostRegisterPortable));
         break;
      case DEVICE_HOST_COPY:
         checkCudaErrors(hipMalloc(srcPtr, numBytes));
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         break;
      case DEVICE_HOST_COPY_PINNED:
         checkCudaErrors(hipSetDevice(srcIdx));
         checkCudaErrors(hipMalloc(srcPtr, numBytes));
         *destPtr = topo.AllocMemByNode(destIdx, numBytes);
         checkCudaErrors(hipHostRegister(*destPtr, numBytes, hipHostRegisterPortable));
         break;
      case HOST_DEVICE_COPY:
         *srcPtr = topo.AllocMemByNode(srcIdx, numBytes);
         checkCudaErrors(hipSetDevice(destIdx));
         checkCudaErrors(hipMalloc(destPtr, numBytes));
         break;
      case HOST_DEVICE_COPY_PINNED:
         *srcPtr = topo.AllocMemByNode(srcIdx, numBytes);
         checkCudaErrors(hipHostRegister(*srcPtr, numBytes, hipHostRegisterPortable));
         checkCudaErrors(hipSetDevice(destIdx));
         checkCudaErrors(hipMalloc(destPtr, numBytes));
         break;
      case PEER_COPY_NO_UVA: 
         checkCudaErrors(hipSetDevice(srcIdx));
         checkCudaErrors(hipMalloc(srcPtr, numBytes));
         checkCudaErrors(hipSetDevice(destIdx));
         checkCudaErrors(hipMalloc(destPtr, numBytes));
         break;
      case DEVICE_DEVICE_COPY:
         checkCudaErrors(hipSetDevice(srcIdx));
         checkCudaErrors(hipMalloc(srcPtr, numBytes));
         checkCudaErrors(hipSetDevice(destIdx));
         checkCudaErrors(hipMalloc(destPtr, numBytes));
         break;
      case COPY_UVA:
         checkCudaErrors(hipSetDevice(srcIdx));
         checkCudaErrors(hipMalloc(srcPtr, numBytes));
         checkCudaErrors(hipSetDevice(destIdx));
         checkCudaErrors(hipMalloc(destPtr, numBytes));
         break;
      default:
         std::cout << "Error: unrecognized memory copy operation type for allocation" << std::endl; 
         break;
   }
}

int CalcRunSteps(std::vector<long long> &blockSteps, long long startStep, long long stopStep, long long numSteps) {
   int magStart = max((int)log10(startStep), 1);
   int magStop = log10(stopStep);

   long long start = pow(10, magStart);
   double stepSize = 10 * start / numSteps;
   int extra = (stopStep - pow(10, magStop)) / pow(10, magStop) * numSteps;
   long long stop = pow(10, magStop - 1) * (10 + extra); 
   int rangeSkip = numSteps / start;
   int totalSteps = (magStop - magStart) * (numSteps - rangeSkip) + extra + 1;  
   double step = start;

   for (long stepNum = 0; stepNum < totalSteps; ++stepNum) { 
      blockSteps.push_back(step);
      
      if ((stepNum) && (stepNum) % (numSteps - rangeSkip) == 0 && (stepSize * numSteps * 10) <= stop) {
         stepSize *= 10.0;
      } 
      
      step += stepSize; 
   }

   return totalSteps;
}

float TimedMemOp(void **MemBlk, long long NumBytes, MEM_OP TimedOp) {
   #ifdef USING_CPP
   std::chrono::high_resolution_clock::time_point start_c, stop_c;
   auto total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
   #else
   struct timeval stop_t, start_t, total_t;
   #endif
   
   hipEvent_t start_e, stop_e; 
   checkCudaErrors(hipEventCreate(&start_e));
   checkCudaErrors(hipEventCreate(&stop_e));
   float OpTime = 0;
  
   switch (TimedOp) {
      case HOST_MALLOC:
         #ifdef USING_CPP
         start_c = std::chrono::high_resolution_clock::now();
         *MemBlk = malloc(NumBytes);
         stop_c = std::chrono::high_resolution_clock::now();
         total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);      
         OpTime = (float) total_c.count() * NANO_TO_MILLI;
         #else
         gettimeofday(&start_t, NULL);
         *MemBlk = malloc(NumBytes); 
         gettimeofday(&stop_t, NULL);
         timersub(&stop_t, &start_t, &total_t);
         OpTime = (float) total_t.tv_usec * MICRO_TO_MILLI;
         #endif
         break;
      case HOST_PINNED_MALLOC:
         checkCudaErrors(hipEventRecord(start_e, 0));      
         checkCudaErrors(hipHostMalloc(MemBlk, NumBytes));
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e));
         break;
      case DEVICE_MALLOC:
         checkCudaErrors(hipEventRecord(start_e, 0));
         checkCudaErrors(hipMalloc(MemBlk, NumBytes));
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e)); 
         break;
      case HOST_FREE:
         #ifdef USING_CPP
         start_c = std::chrono::high_resolution_clock::now();
         free(*MemBlk);
         stop_c = std::chrono::high_resolution_clock::now(); 
         total_c = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_c - start_c);
         OpTime = (float) total_c.count() * NANO_TO_MILLI;
         #else
         gettimeofday(&start_t, NULL);
         free(*MemBlk); 
         gettimeofday(&stop_t, NULL); 
         timersub(&stop_t, &start_t, &total_t); 
         OpTime = (float) total_t.tv_usec * MICRO_TO_MILLI;
         #endif
         break;
      case HOST_PINNED_FREE:
         checkCudaErrors(hipEventRecord(start_e, 0));
         checkCudaErrors(hipHostFree(*MemBlk));
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e));
         break;
      case DEVICE_FREE:
         checkCudaErrors(hipEventRecord(start_e, 0));
         checkCudaErrors(hipFree(*MemBlk)); 
         checkCudaErrors(hipEventRecord(stop_e, 0));
         checkCudaErrors(hipEventSynchronize(stop_e));   
         checkCudaErrors(hipEventElapsedTime(&OpTime, start_e, stop_e));  
         break;
      default:
         std::cout << "Error: unrecognized timed memory operation type" << std::endl; 
         break;
   }
   checkCudaErrors(hipEventDestroy(start_e));
   checkCudaErrors(hipEventDestroy(stop_e));

   return OpTime;
}

// Prints the device properties out to file based named depending on the 
void PrintDeviceProps(hipDeviceProp_t *props, BenchParams &params) {
   std::cout << "\nSee " << params.devPropFile << " for information about your device's properties." << std::endl; 
   std::string devFileName = "./results/" + params.devPropFile;
   std::ofstream deviceProps(devFileName.c_str());

   deviceProps << "-------- Device Properties --------" << std::endl;

   for (int i = 0; i < params.nDevices; i++) {
      deviceProps << props[i].name << std::endl;
      deviceProps << "CUDA Capability: " << props[i].major << "." << props[i].minor << std::endl;
      deviceProps << "PCI Bus/Device/Domain ID: " <<   props[i].pciBusID << ":" <<  props[i].pciDeviceID << ":" <<  props[i].pciDomainID << std::endl; 
      deviceProps << "Clock: " << props[i].clockRate << std::endl; 
      deviceProps << "Memory Clock: " << props[i].memoryClockRate << std::endl; 
      deviceProps << "Memory Bus Width: " << props[i].memoryBusWidth << std::endl; 
      deviceProps << "Theoretical BW: " << props[i].clockRate << std::endl;
      deviceProps << "Global Mem: " << props[i].totalGlobalMem << std::endl;

 
/*        printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
        printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

        SPRINTF(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
        printf("%s", msg);

        printf("  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
               deviceProp.multiProcessorCount,
               _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
               _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
        printf("  GPU Max Clock rate:                            %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

#if CUDART_VERSION >= 5000
        // This is supported in CUDA 5.0 (runtime API device properties)
        printf("  Memory Clock rate:                             %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
        printf("  Memory Bus Width:                              %d-bit\n",   deviceProp.memoryBusWidth);

        if (deviceProp.l2CacheSize)
        {
            printf("  L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);
        }
#else
        // This only available in CUDA 4.0-4.2 (but these were only exposed in the CUDA Driver API)
        int memoryClock;
        getCudaAttribute<int>(&memoryClock, hipDeviceAttributeMemoryClockRate, dev);
        printf("  Memory Clock rate:                             %.0f Mhz\n", memoryClock * 1e-3f);
        int memBusWidth;
        getCudaAttribute<int>(&memBusWidth, hipDeviceAttributeMemoryBusWidth, dev);
        printf("  Memory Bus Width:                              %d-bit\n", memBusWidth);
        int L2CacheSize;
        getCudaAttribute<int>(&L2CacheSize, hipDeviceAttributeL2CacheSize, dev);

        if (L2CacheSize)
        {
            printf("  L2 Cache Size:                                 %d bytes\n", L2CacheSize);
        }
*/
   }
   deviceProps << "-----------------------------------" << std::endl;

   deviceProps.close();
}

void PrintResults(std::ofstream &outFile, std::vector<long long> &steps, std::vector<std::vector<float> > &results, BenchParams &params) {
   
   if (!outFile.is_open()) {
      std::cout << "Failed to open file to print results" << std::endl;
      return;
   }
   std::vector<std::vector<float> >::iterator iter_o;
   std::vector<float>::iterator iter_i;
   std::vector<long long>::iterator iter_l = steps.begin();
   std::cout << results[0].size() << std::endl;
   
   for (iter_o = results.begin(); iter_o != results.end(); ++iter_o) {
      outFile << std::fixed << *iter_l++ << ",";
      for (iter_i = (*iter_o).begin(); iter_i != (*iter_o).end(); ++iter_i) {
         outFile << std::fixed << *iter_i;
         if (iter_i + 1 != (*iter_o).end())
            outFile << ",";
      }
      outFile << std::endl;
   }
}

// Creates an array of hipDeviceProp_t structs with populated data
// located in a pre-allocated section of memory
void GetAllDeviceProps(hipDeviceProp_t *props, int dCount) {
   for (int i = 0; i < dCount; ++i) {
      checkCudaErrors(hipGetDeviceProperties(&props[i], i));
   }
}

// function for cleaning up device state including profile data
// to be used before and after any test in benchmark suite.
void ResetDevices(int numToReset) {
   for (int devNum = 0; devNum < numToReset; ++devNum) {
      checkCudaErrors(hipSetDevice(devNum));
      checkCudaErrors(hipDeviceReset());
   }
}

